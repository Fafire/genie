#include "hip/hip_runtime.h"
/** Name: test_6.cu
 * Description:
 * focus on serialization of table, especially under multiload setting
 *   sift data
 *   data is from csv file
 *   query is from csv file, single range
 *
 *
 */


#undef NDEBUG

#include <cassert>
#include <fstream>

#include <boost/archive/text_oarchive.hpp>
#include <boost/archive/text_iarchive.hpp>

#include <GPUGenie/interface.h>
#include <GPUGenie/inv_table.h>
#include <GPUGenie/inv_compr_table.h>
#include <GPUGenie/serialization.h>

using namespace std;
using namespace GPUGenie;


void testSerialization(GPUGenie::GPUGenie_Config &config)
{
    Logger::log(Logger::INFO, "Preprocessing inverted table...");
    GPUGenie::inv_table * table = nullptr;
    GPUGenie::inv_compr_table * comprTable = nullptr;
    GPUGenie::preprocess_for_knn_csv(config, table); // this returns inv_compr_table if config.compression is set
    assert(table != nullptr);
    assert(table->build_status() == inv_table::builded);

    string inv_filename("/tmp/genie_test_serialization.invtable");
    Logger::log(Logger::INFO, "Saving inverted table to file...");
    {
        std::ofstream ofs(inv_filename.c_str());
        boost::archive::text_oarchive oa(ofs);
        oa << *table;
    }
    
    GPUGenie::inv_table * loaded_table = new inv_table();
    Logger::log(Logger::INFO, "Loading inverted table from file...");
    {
        std::ifstream ifs(inv_filename.c_str());
        boost::archive::text_iarchive ia(ifs);
        ia >> *loaded_table;
    }

    Logger::log(Logger::INFO, "Checking loaded table correctness...");

    // assert(table->table_index == loaded_table->table_index);

    Logger::log(Logger::DEBUG, "Deallocating inverted table...");
    delete[] table;
    delete loaded_table;

}

int main(int argc, char* argv[])
{
    string dataFile = "../static/sift_20.csv";
    
    Logger::log(Logger::INFO, "Reading csv data file %s ...", dataFile.c_str());
    std::vector<std::vector<int>> data;
    GPUGenie::GPUGenie_Config config;
    config.data_points = &data;
    config.data_type = 0;
    GPUGenie::read_file(data, dataFile.c_str(), -1);

    // Test inv_table
    config.compression = NO_COMPRESSION;
    testSerialization(config);

    // Test inv_compr_table
    config.compression = HEAVYWEIGHT_COMPRESSION_TYPE;
    testSerialization(config);

    return 0;
}
