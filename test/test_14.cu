#include "hip/hip_runtime.h"
/**
 * Name: test_14.cu
 * Description:
 *  Test counting of queries on CPU. Similar to CPU-Idx in GENIE paper.
 */

#include <GPUGenie.h>

#include <algorithm>
#include <assert.h>
#include <vector>
#include <iostream>

#include <sstream>
#include <stdio.h>

#include "codecfactory.h"
#include "intersection.h"

using namespace GPUGenie;
using namespace SIMDCompressionLib;

const int MAX_PRINT_LEN = 128;
const std::string DEFAULT_TEST_DATASET = "../static/sift_20.dat";
const std::string DEFAULT_QUERY_DATASET = "../static/sift_20.csv";

void logQueries(std::vector<query> &queries)
{
    for (query &q : queries)
    {
        Logger::log(Logger::DEBUG, "Query idx: %d, topk: %d, count_ranges: %d, selectivity: %f",
                    q.index(), q.topk(), q.count_ranges(), q.selectivity());
        q.print(MAX_PRINT_LEN);

        std::vector<query::dim> dims;
        q.dump(dims);

        for (query::dim &d : dims){
            Logger::log(Logger::DEBUG, "  Dim -- query: %d, order: %d, start_pos: %d, end_pos: %d",
                    d.query, d.order, d.start_pos, d.end_pos);
        }
    }
}

void logResults(std::vector<query> &queries, std::vector<int> &result, std::vector<int> &result_count)
{
    size_t resultsBeginIdx = 0;
    for (query &q : queries)
    {
        Logger::log(Logger::DEBUG, "---");
        Logger::log(Logger::DEBUG, "Query idx: %d, topk: %d, count_ranges: %d, selectivity: %f",
                    q.index(), q.topk(), q.count_ranges(), q.selectivity());

        std::stringstream ss;
        size_t noResultsToPrint = std::min(q.topk(),MAX_PRINT_LEN);
        for (size_t i = 0; i < noResultsToPrint; ++i)
            ss << result[resultsBeginIdx+i] << "(" << result_count[resultsBeginIdx+i] << ") ";
        Logger::log(Logger::DEBUG, "  Results: %s", ss.str().c_str());
        resultsBeginIdx += q.topk();
    }
}

void log_table(GPUGenie::inv_table *table, size_t max_print_len = 256)
{
    if (table->build_status() == GPUGenie::inv_table::not_builded)
    {
        Logger::log(Logger::DEBUG, "Inv table not built.");
        return;
    }

    std::stringstream ss;    
    std::vector<int> *ck = table->ck();
    if (ck)
    {
        auto end = (ck->size() <= max_print_len) ? ck->end() : (ck->begin() + max_print_len); 
        std::copy(ck->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "CK:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }

    std::vector<int> *inv = table->inv();
    if (inv)
    {
        auto end = (inv->size() <= max_print_len) ? inv->end() : (inv->begin() + max_print_len); 
        std::copy(inv->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "INV:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }

    std::vector<int> *inv_index = table->inv_index();
    if (inv_index)
    {
        auto end = (inv_index->size() <= max_print_len) ? inv_index->end() : (inv_index->begin() + max_print_len); 
        std::copy(inv_index->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "INV_INDEX:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }


    std::vector<int> *inv_pos = table->inv_pos();
    if (inv_pos)
    {
        auto end = (inv_pos->size() <= max_print_len) ? inv_pos->end() : (inv_pos->begin() + max_print_len); 
        std::copy(inv_pos->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "INV_POS:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }
}

void log_inv_lists(const std::vector<std::vector<uint32_t>> &rawInvertedLists, size_t max_print_len = 16)
{
    std::stringstream ss;
    auto inv_it_end = (rawInvertedLists.size() <= max_print_len)
                            ? rawInvertedLists.end() : (rawInvertedLists.begin() + max_print_len);
    Logger::log(Logger::DEBUG, "rawInvertedLists.size(): %d", rawInvertedLists.size());
    for (auto inv_it = rawInvertedLists.begin(); inv_it != inv_it_end; inv_it++)
    {
        const std::vector<uint32_t> &invList = *inv_it; 
        auto end = (invList.size() <= max_print_len) ? invList.end() : (invList.begin() + max_print_len);
        std::copy(invList.begin(), end, std::ostream_iterator<uint32_t>(ss, " "));
        Logger::log(Logger::DEBUG, "*** [%s]", ss.str().c_str());
        ss.str(std::string());
        ss.clear();

    }
}


int main(int argc, char* argv[])
{
    Logger::log(Logger::INFO, "Available codecs (SIMDCompressionLib::CODECFactory::scodecmap):");
    for (auto &kv : CODECFactory::scodecmap)
        Logger::log(Logger::INFO, "  %s", kv.first.c_str());
    
    // Logger::log(Logger::INFO, "Available compressions in GENIE (GPUGenie_Config::):");
    // for (auto &kv : GPUGenie_Config::compression_types)
    //     Logger::log(Logger::INFO, "  %s", kv.first);


    string dataFile = DEFAULT_TEST_DATASET;
    if (argc == 2)
        dataFile = std::string(argv[1]);
    string queryFile = DEFAULT_QUERY_DATASET;

    vector<vector<int>> queryPoints;
    inv_table * table = NULL;
    GPUGenie_Config config;

    config.dim = 5;
    config.count_threshold = 14;
    config.num_of_topk = 5;
    config.hashtable_size = 14*config.num_of_topk*1.5;
    config.query_radius = 0;
    config.use_device = 0;
    config.use_adaptive_range = false;
    config.selectivity = 0.0f;

    config.query_points = &queryPoints;
    config.data_points = NULL;

    config.use_load_balance = false;
    config.posting_list_max_length = 6400;
    config.multiplier = 1.5f;
    config.use_multirange = false;

    config.data_type = 1;
    config.search_type = 0;
    config.max_data_size = 0;

    config.num_of_queries = 3;

    config.compression_type = GPUGenie_Config::COMPRESSION_TYPE::NO_COMPRESSION;


    std::cout << "Reading data file " << dataFile << "..." << std::endl;  
    read_file(dataFile.c_str(), &config.data, config.item_num, &config.index, config.row_num);
    assert(config.item_num > 0);
    assert(config.row_num > 0);
    Logger::log(Logger::DEBUG, "config.item_num: %d", config.item_num);
    Logger::log(Logger::DEBUG, "config.row_num: %d", config.row_num);
    std::cout << "Done reading data file!" << std::endl;  


    std::cout << "Preprocessing data (" << config.item_num << " items total)..." << std::endl;  
    preprocess_for_knn_binary(config, table);
    // check how many tables we have
    assert(table != NULL);
    assert(table->get_total_num_of_table() == 1);
    // assert(config.compression_type == GPUGenie_Config::DELTA);
    std::cout << "Done preprocessing data..." << std::endl; 


    std::cout << "Examining inverted lists...";
    std::vector<GPUGenie::inv_list> *inv_lists = table->inv_lists();
    // check inverted index of the tables using inv_list class
    for (size_t attr_index = 0; attr_index < inv_lists->size(); attr_index++)
    {
        GPUGenie::inv_list invertedList = (*inv_lists)[attr_index];
        int posting_list_length = invertedList.size();
        int posting_list_min = invertedList.min();
        int posting_list_max = invertedList.max();
        Logger::log(Logger::DEBUG, "  attr_index %d, posting_list_length: %d, min: %d, max: %d",
                        attr_index, posting_list_length, posting_list_min, posting_list_max);
        Logger::log(Logger::DEBUG, "    table->get_lowerbound_of_list(%d): %d, table->get_upperbound_of_list(%d): %d", attr_index, table->get_lowerbound_of_list(attr_index),
            attr_index, table->get_upperbound_of_list(attr_index));
    }

    log_table(table);

    std::cout << "Done examining inverted lists..." << std::endl;

    std::cout << "Copying inverted lists for compression..." << std::endl;

    std::vector<int> *ck = table->ck();
    std::vector<int> *inv = table->inv();
    std::vector<int> *inv_index = table->inv_index();
    std::vector<int> *inv_pos = table->inv_pos();

    std::vector<std::vector<uint32_t>> rawInvertedLists;
    size_t rawInvertedListsSize = inv_pos->back();

    auto inv_it = inv->begin();
    size_t prev_inv_pos = *(inv_pos->begin());
    for (auto inv_pos_it = (inv_pos->begin()+1); inv_pos_it != inv_pos->end(); inv_pos_it++)
    {
        size_t offset = (*inv_pos_it) - prev_inv_pos;
        prev_inv_pos = (*inv_pos_it);
        
        std::vector<uint32_t> invList(inv_it, inv_it + offset);
        inv_it += offset;
        rawInvertedLists.push_back(invList);
    }

    log_inv_lists(rawInvertedLists);

    std::cout << "Done copying inverted lists for compression!" << std::endl;
    
    double avg_inv_list_length = ((double)rawInvertedListsSize) / ((double)inv_pos->size());
    Logger::log(Logger::DEBUG, "Total inverted lists: %d, Average length of inv list: %f",
        rawInvertedListsSize, avg_inv_list_length);
    Logger::log(Logger::DEBUG, "Uncompressed size of inv: %d bytes", inv->size() * 4);
    Logger::log(Logger::DEBUG, "Uncompressed size of inv_pos: %d bytes", inv_pos->size() * 4);

    std::cout << std::endl;
    std::cout << std::endl;



    std::cout << "Compressing inverted lists..." << std::endl;
    // for (auto &kv : CODECFactory::scodecmap)
    // {
    // string compression_name = "copy";
    string compression_name = "s4-bp128-d1";
    bool manualDelta = false;

    if (compression_name == "for" || compression_name == "frameofreference"
            || compression_name == "simdframeofreference")
        manualDelta = true;

    // std::cout << "Compressing inverted lists using " << compression_name << "..." << std::endl;
    IntegerCODEC &codec = *CODECFactory::getFromName(compression_name);
    
    size_t compressedsize_total = 0;

    std::vector<std::vector<uint32_t>> comprInvertedLists(rawInvertedLists.size());

    // Compress all inverted lists
    for (size_t i = 0; i < rawInvertedLists.size(); i++)
    {
        comprInvertedLists[i].resize(rawInvertedLists[i].size() + 1024);
        size_t compressedsize = comprInvertedLists[i].size();

        if (manualDelta)
            delta<uint32_t>(static_cast<uint32_t>(0), rawInvertedLists[i].data(), rawInvertedLists[i].size());
        codec.encodeArray(
                rawInvertedLists[i].data(), rawInvertedLists[i].size(),
                comprInvertedLists[i].data(),compressedsize);

        comprInvertedLists[i].resize(compressedsize);
        compressedsize_total += compressedsize;
    }

    std::cout << "Done compressing inverted lists..." << std::endl;


    std::cout << "Preprocessing queries..." << std::endl;

    read_file(*config.query_points, queryFile.c_str(), config.num_of_queries);

    std::vector<query> queries;
    std::vector<int> results;
    std::vector<int> results_count;

    load_query(*table, queries, config);

    knn_search(*table, queries, results, results_count, config);

    logResults(queries, results, results_count);

    std::vector<uint32_t> tmpResultCounts(config.row_num), resultCounts;
    std::vector<uint32_t> tmpResultIdxs(config.row_num), resultIdxs;
    resultCounts.reserve(config.num_of_topk * config.num_of_queries);
    resultIdxs.reserve(config.num_of_topk * config.num_of_queries);

    int shifter = table->shifter();
    for (query &q : queries)
    {
        std::vector<int> invListsTocount;
        std::vector<query::range> ranges;
        int queryIndex = q.index();

        q.dump(ranges);
        Logger::log(Logger::DEBUG, "Processing query %d, has %d ranges", queryIndex, ranges.size());

        if (ranges.empty())
        {
            Logger::log(Logger::ALERT, "Query %d has no ranges!", queryIndex);
            continue;
        }

        for (query::range &r : ranges)
        {
            int low = r.low;
            int up = r.up;

            int dimShifted = r.dim << shifter;
            
            Logger::log(Logger::DEBUG, "  range %d, query: %d, dim: %d, low: %d, up: %d", r.order, r.query, 
                r.dim, r.low, r.up);

            if(low > up || low > table->get_upperbound_of_list(r.dim) || up < table->get_lowerbound_of_list(r.dim))
            {
                Logger::log(Logger::DEBUG, "  range %d out of bounds of inverted lists in dim %d", r.order, r.dim); 
                continue;
            }

            low = low < table->get_lowerbound_of_list(r.dim) ? table->get_lowerbound_of_list(r.dim) : low;
            up = up > table->get_upperbound_of_list(r.dim) ? table->get_upperbound_of_list(r.dim) : up;

            int min = dimShifted + low - table->get_lowerbound_of_list(r.dim);
            int max = dimShifted + up - table->get_lowerbound_of_list(r.dim);
            Logger::log(Logger::DEBUG, "  low %d, up: %d, min: %d, max: %d", low, up, min, max);

            // Record ids of inverted lists to be counted
            int invList = (*inv_index)[min];
            do
            {
                Logger::log(Logger::DEBUG, "  adding inverted list %d", invList);
                invListsTocount.push_back(invList++);
            }
            while (invList < (*inv_index)[max+1]);
        }

        Logger::log(Logger::DEBUG, " inverted lists to count for query %d", queryIndex);
        for (int i : invListsTocount)
            Logger::log(Logger::DEBUG, "  inverted list %d", i);

        // Reset temporary count and index vector -- these vectors are used directly for counting
        std::fill(tmpResultCounts.begin(),tmpResultCounts.end(),static_cast<uint32_t>(0));
        std::iota(tmpResultIdxs.begin(), tmpResultIdxs.end(),static_cast<uint32_t>(0));

        for (int invListIndex : invListsTocount)
        {
            size_t decompressedsize = rawInvertedLists[invListIndex].size();

            // Decompress the compressed inverted list with index invListIndex
            codec.decodeArray(
                comprInvertedLists[invListIndex].data(), comprInvertedLists[invListIndex].size(),
                rawInvertedLists[invListIndex].data(),decompressedsize);
            if (manualDelta)
                inverseDelta<uint32_t>(static_cast<uint32_t>(0), rawInvertedLists[invListIndex].data(),
                        rawInvertedLists[invListIndex].size());

            assert(rawInvertedLists[invListIndex].size() == decompressedsize);

            // Count docId from the decompressed list
            for (int docId : rawInvertedLists[invListIndex])
                ++tmpResultCounts[docId];
        }

        // Sort tmpResultIdxs according to tmpResultCount
        std::sort(tmpResultIdxs.begin(), tmpResultIdxs.end(),
           [&tmpResultCounts](uint32_t lhs, uint32_t rhs) {return tmpResultCounts[lhs] < tmpResultCounts[rhs];});

        // Copy the first q.topk() results into the final results vectors resultCounts and resultIdxs
        for (auto it = tmpResultIdxs.begin(); it < tmpResultIdxs.begin() + q.topk(); it++)
        {
            resultCounts.push_back(tmpResultCounts[*it]);
            resultIdxs.push_back(*it);
        }
    }

    logResults(queries, results, results_count);

    return 0;
}

