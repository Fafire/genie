#include "hip/hip_runtime.h"
/**
 * Name: test_14.cu
 * Description:
 *  Test counting of queries on CPU. Similar to CPU-Idx in GENIE paper.
 */

#include <GPUGenie.h>

#include <algorithm>
#include <assert.h>
#include <vector>
#include <iostream>

#include <sstream>
#include <stdio.h>

#include "codecfactory.h"
#include "intersection.h"

using namespace GPUGenie;
using namespace SIMDCompressionLib;

const int MAX_PRINT_LEN = 128;
const std::string DEFAULT_TEST_DATASET = "../static/sift_20.dat";
const std::string DEFAULT_QUERY_DATASET = "../static/sift_20.csv";

void printResults(std::vector<query> &queries, std::vector<int> &result, std::vector<int> &result_count)
{
    size_t resultsBeginIdx = 0;
    for (query &q : queries)
    {
        Logger::log(Logger::DEBUG, "---");
        Logger::log(Logger::DEBUG, "Query idx: %d, topk: %d, count_ranges: %d, selectivity: %f",
                    q.index(), q.topk(), q.count_ranges(), q.selectivity());
        q.print(MAX_PRINT_LEN);

        std::stringstream ss;
        size_t noResultsToPrint = std::min(q.topk(),MAX_PRINT_LEN);
        for (size_t i = 0; i < noResultsToPrint; ++i)
            ss << result[resultsBeginIdx+i] << "~" << result_count[resultsBeginIdx+i] << " ";
        Logger::log(Logger::DEBUG, "Results: %s", ss.str().c_str());
        resultsBeginIdx += q.topk();
    }
}

int main(int argc, char* argv[])
{
    Logger::log(Logger::INFO, "Available codecs (SIMDCompressionLib::CODECFactory::scodecmap):");
    for (auto &kv : CODECFactory::scodecmap)
        Logger::log(Logger::INFO, "  %s", kv.first.c_str());
    
    // Logger::log(Logger::INFO, "Available compressions in GENIE (GPUGenie_Config::):");
    // for (auto &kv : GPUGenie_Config::compression_types)
    //     Logger::log(Logger::INFO, "  %s", kv.first);


    string dataFile = DEFAULT_TEST_DATASET;
    if (argc == 2)
        dataFile = std::string(argv[1]);
    string queryFile = DEFAULT_QUERY_DATASET;

    vector<vector<int>> queryPoints;
    inv_table * table = NULL;
    GPUGenie_Config config;

    config.dim = 5;
    config.count_threshold = 14;
    config.num_of_topk = 5;
    config.hashtable_size = 14*config.num_of_topk*1.5;
    config.query_radius = 0;
    config.use_device = 0;
    config.use_adaptive_range = false;
    config.selectivity = 0.0f;

    config.query_points = &queryPoints;
    config.data_points = NULL;

    config.use_load_balance = false;
    config.posting_list_max_length = 6400;
    config.multiplier = 1.5f;
    config.use_multirange = false;

    config.data_type = 1;
    config.search_type = 0;
    config.max_data_size = 0;

    config.num_of_queries = 3;

    config.compression_type = GPUGenie_Config::COMPRESSION_TYPE::NO_COMPRESSION;


    std::cout << "Reading data file " << dataFile << "..." << std::endl;  
    read_file(dataFile.c_str(), &config.data, config.item_num, &config.index, config.row_num);
    assert(config.item_num > 0);
    assert(config.row_num > 0);
    Logger::log(Logger::DEBUG, "config.item_num: %d", config.item_num);
    Logger::log(Logger::DEBUG, "config.row_num: %d", config.row_num);
    std::cout << "Done reading data file!" << std::endl;  


    std::cout << "Preprocessing data (" << config.item_num << " items total)..." << std::endl;  
    preprocess_for_knn_binary(config, table);
    // check how many tables we have
    assert(table != NULL);
    assert(table->get_total_num_of_table() == 1);
    // assert(config.compression_type == GPUGenie_Config::DELTA);
    std::cout << "Done preprocessing data..." << std::endl; 


    std::cout << "Examining inverted lists...";
    std::vector<GPUGenie::inv_list> *inv_lists = table->inv_lists();
    // check inverted index of the tables using inv_list class
    for (size_t attr_index = 0; attr_index < inv_lists->size(); attr_index++)
    {
        GPUGenie::inv_list invertedList = (*inv_lists)[attr_index];
        int posting_list_length = invertedList.size();
        int posting_list_min = invertedList.min();
        int posting_list_max = invertedList.max();
        Logger::log(Logger::DEBUG, "  attr_index %d, posting_list_length: %d, m in: %d, max: %d",
                        attr_index, posting_list_length, posting_list_min, posting_list_max);
        Logger::log(Logger::DEBUG, "    table->get_lowerbound_of_list(%d): %d, table->get_upperbound_of_list(%d): %d", attr_index, table->get_lowerbound_of_list(attr_index),
            attr_index, table->get_upperbound_of_list(attr_index));
    }
    std::cout << "Done examining inverted lists..." << std::endl;

    std::cout << "Copying inverted lists for compression..." << std::endl;

    std::vector<int> *ck = table->ck();
    std::vector<int> *inv = table->inv();
    std::vector<int> *inv_index = table->inv_index();
    std::vector<int> *inv_pos = table->inv_pos();

    std::vector<std::vector<uint32_t>> rawInvertedLists;
    size_t rawInvertedListsSize = inv_pos->back();
    auto inv_it = inv->begin();
    size_t prev_inv_pos = *(inv_pos->begin());
    for (auto inv_pos_it = (inv_pos->begin()+1); inv_pos_it != inv_pos->end(); inv_pos_it++)
    {
        size_t offset = (*inv_pos_it) - prev_inv_pos;
        prev_inv_pos = (*inv_pos_it);
        
        std::vector<uint32_t> invList(inv_it, inv_it + offset);
        inv_it += offset;
        rawInvertedLists.push_back(invList);
    }
    std::cout << "Done copying inverted lists for compression!" << std::endl;
    
    double avg_inv_list_length = ((double)rawInvertedListsSize) / ((double)inv_pos->size());
    Logger::log(Logger::DEBUG, "Total inverted lists: %d, Average length of inv list: %f",
        rawInvertedListsSize, avg_inv_list_length);
    Logger::log(Logger::DEBUG, "Uncompressed size of inv: %d bytes", inv->size() * 4);
    Logger::log(Logger::DEBUG, "Uncompressed size of inv_pos: %d bytes", inv_pos->size() * 4);

    std::cout << std::endl;
    std::cout << std::endl;



    std::cout << "Compressing inverted lists..." << std::endl;
    // for (auto &kv : CODECFactory::scodecmap)
    // {
    // string compression_name = "copy";
    string compression_name = "s4-bp128-d1";
    bool manualDelta = false;

    if (compression_name == "for" || compression_name == "frameofreference"
            || compression_name == "simdframeofreference")
        manualDelta = true;

    // std::cout << "Compressing inverted lists using " << compression_name << "..." << std::endl;
    IntegerCODEC &codec = *CODECFactory::getFromName(compression_name);
    
    size_t compressedsize_total = 0;

    std::vector<std::vector<uint32_t>> comprInvertedLists(rawInvertedLists.size());

    // Compress all inverted lists
    for (size_t i = 0; i < rawInvertedLists.size(); i++)
    {
        comprInvertedLists[i].resize(rawInvertedLists[i].size() + 1024);
        size_t compressedsize = comprInvertedLists[i].size();

        if (manualDelta)
            delta<uint32_t>(static_cast<uint32_t>(0), rawInvertedLists[i].data(), rawInvertedLists[i].size());
        codec.encodeArray(
                rawInvertedLists[i].data(), rawInvertedLists[i].size(),
                comprInvertedLists[i].data(),compressedsize);

        comprInvertedLists[i].resize(compressedsize);
        compressedsize_total += compressedsize;
    }

    std::cout << "Done compressing inverted lists..." << std::endl;


    std::cout << "Preprocessing queries..." << std::endl;

    read_file(*config.query_points, queryFile.c_str(), config.num_of_queries);

    std::vector<query> queries;
    std::vector<int> results;
    std::vector<int> results_count;

    load_query(*table, queries, config);

    knn_search(*table, queries, results, results_count, config);

    printResults(queries, results, results_count);

    // // Decompress all inverted lists
    // unsigned long long time_decompr_start = getTime(), time_decompr_tight_start, time_decompr_tight_stop;
    // double time_decompr_tight = 0.0;
    // for (size_t i = 0; i < rawInvertedLists.size(); i++)
    // {
    //     size_t decompressedsize = rawInvertedLists[i].size();

    //     time_decompr_tight_start = getTime();
    //     codec.decodeArray(
    //         comprInvertedLists[i].data(), comprInvertedLists[i].size(),
    //         rawInvertedLists[i].data(),decompressedsize);
    //     if (manualDelta)
    //         inverseDelta<uint32_t>(static_cast<uint32_t>(0), rawInvertedLists[i].data(),
    //                 rawInvertedLists[i].size());
    //     time_decompr_tight_stop = getTime();

    //     assert(decompressedsize == inv_lists_orig_sizes[i]);
    //     time_decompr_tight += getInterval(time_decompr_tight_start, time_decompr_tight_stop);
    // }
    // unsigned long long time_decompr_stop = getTime();
    // double time_decompr = getInterval(time_decompr_start, time_decompr_stop);

    // std::cout << std::fixed << std::setprecision(3);
    // std::cout << "File: " << dataFile
    //           << ", Compr: " << compression_name
    //           << ", Ratio: "
    //                 << 32.0 * static_cast<double>(compressedsize_total) / static_cast<double>(rawInvertedListsSize)
    //                 << " bpi "
    //           << ", DTime: " << time_decompr
    //           << ", DXTime: " << time_decompr_tight
    //           << std::endl;
    // // }
    // std::cout << "DONE compressing and decompressing inverted lists..." << std::endl;
    // return 0;


    // take a query 

    // uncompress one block of all relevant inverted lists

    // do counting (naively)


    return 0;
}

