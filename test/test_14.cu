#include "hip/hip_runtime.h"
/**
 * Name: test_14.cu
 * Description:
 *  Test counting of queries on CPU. Similar to CPU-Idx in GENIE paper.
 */

#include <GPUGenie.h>

#include <algorithm>
#include <assert.h>
#include <vector>
#include <iostream>

#include <sstream>
#include <stdio.h>

#include "codecfactory.h"
#include "intersection.h"

using namespace GPUGenie;
using namespace SIMDCompressionLib;

const int MAX_PRINT_LEN = 128;
const std::string DEFAULT_TEST_DATASET = "../static/sift_20.dat";
const std::string DEFAULT_QUERY_DATASET = "../static/sift_20.csv";

void logResults(std::vector<query> &queries, std::vector<int> &result, std::vector<int> &result_count)
{
    size_t resultsBeginIdx = 0;
    for (query &q : queries)
    {
        Logger::log(Logger::DEBUG, "---");
        Logger::log(Logger::DEBUG, "Query idx: %d, topk: %d, count_ranges: %d, selectivity: %f",
                    q.index(), q.topk(), q.count_ranges(), q.selectivity());
        q.print(MAX_PRINT_LEN);

        std::vector<query::dim> dims;
        q.dump(dims);

        for (query::dim &d : dims){
            Logger::log(Logger::DEBUG, "  dim -- query: %d, order: %d, start_pos: %d, end_pos: %d",
                    d.query, d.order, d.start_pos, d.end_pos);
        }

        std::stringstream ss;
        size_t noResultsToPrint = std::min(q.topk(),MAX_PRINT_LEN);
        for (size_t i = 0; i < noResultsToPrint; ++i)
            ss << result[resultsBeginIdx+i] << "(" << result_count[resultsBeginIdx+i] << ") ";
        Logger::log(Logger::DEBUG, "Results: %s", ss.str().c_str());
        resultsBeginIdx += q.topk();
    }
}

void log_table(GPUGenie::inv_table *table, size_t max_print_len = 256)
{
    if (table->build_status() == GPUGenie::inv_table::not_builded)
    {
        Logger::log(Logger::DEBUG, "Inv table not built.");
        return;
    }

    std::stringstream ss;    
    std::vector<int> *ck = table->ck();
    if (ck)
    {
        auto end = (ck->size() <= max_print_len) ? ck->end() : (ck->begin() + max_print_len); 
        std::copy(ck->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "CK:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }

    std::vector<int> *inv = table->inv();
    if (inv)
    {
        auto end = (inv->size() <= max_print_len) ? inv->end() : (inv->begin() + max_print_len); 
        std::copy(inv->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "INV:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }

    std::vector<int> *inv_index = table->inv_index();
    if (inv_index)
    {
        auto end = (inv_index->size() <= max_print_len) ? inv_index->end() : (inv_index->begin() + max_print_len); 
        std::copy(inv_index->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "INV_INDEX:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }


    std::vector<int> *inv_pos = table->inv_pos();
    if (inv_pos)
    {
        auto end = (inv_pos->size() <= max_print_len) ? inv_pos->end() : (inv_pos->begin() + max_print_len); 
        std::copy(inv_pos->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "INV_POS:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }
}

void log_inv_lists(const std::vector<std::vector<uint32_t>> &rawInvertedLists, size_t max_print_len = 16)
{
    std::stringstream ss;
    auto inv_it_end = (rawInvertedLists.size() <= max_print_len)
                            ? rawInvertedLists.end() : (rawInvertedLists.begin() + max_print_len);
    Logger::log(Logger::DEBUG, "rawInvertedLists.size(): %d", rawInvertedLists.size());
    for (auto inv_it = rawInvertedLists.begin(); inv_it != inv_it_end; inv_it++)
    {
        const std::vector<uint32_t> &invList = *inv_it; 
        auto end = (invList.size() <= max_print_len) ? invList.end() : (invList.begin() + max_print_len);
        std::copy(invList.begin(), end, std::ostream_iterator<uint32_t>(ss, " "));
        Logger::log(Logger::DEBUG, "*** [%s]", ss.str().c_str());
        ss.str(std::string());
        ss.clear();

    }
}


int main(int argc, char* argv[])
{
    Logger::log(Logger::INFO, "Available codecs (SIMDCompressionLib::CODECFactory::scodecmap):");
    for (auto &kv : CODECFactory::scodecmap)
        Logger::log(Logger::INFO, "  %s", kv.first.c_str());
    
    // Logger::log(Logger::INFO, "Available compressions in GENIE (GPUGenie_Config::):");
    // for (auto &kv : GPUGenie_Config::compression_types)
    //     Logger::log(Logger::INFO, "  %s", kv.first);


    string dataFile = DEFAULT_TEST_DATASET;
    if (argc == 2)
        dataFile = std::string(argv[1]);
    string queryFile = DEFAULT_QUERY_DATASET;

    vector<vector<int>> queryPoints;
    inv_table * table = NULL;
    GPUGenie_Config config;

    config.dim = 5;
    config.count_threshold = 14;
    config.num_of_topk = 5;
    config.hashtable_size = 14*config.num_of_topk*1.5;
    config.query_radius = 0;
    config.use_device = 0;
    config.use_adaptive_range = false;
    config.selectivity = 0.0f;

    config.query_points = &queryPoints;
    config.data_points = NULL;

    config.use_load_balance = false;
    config.posting_list_max_length = 6400;
    config.multiplier = 1.5f;
    config.use_multirange = false;

    config.data_type = 1;
    config.search_type = 0;
    config.max_data_size = 0;

    config.num_of_queries = 3;

    config.compression_type = GPUGenie_Config::COMPRESSION_TYPE::NO_COMPRESSION;


    std::cout << "Reading data file " << dataFile << "..." << std::endl;  
    read_file(dataFile.c_str(), &config.data, config.item_num, &config.index, config.row_num);
    assert(config.item_num > 0);
    assert(config.row_num > 0);
    Logger::log(Logger::DEBUG, "config.item_num: %d", config.item_num);
    Logger::log(Logger::DEBUG, "config.row_num: %d", config.row_num);
    std::cout << "Done reading data file!" << std::endl;  


    std::cout << "Preprocessing data (" << config.item_num << " items total)..." << std::endl;  
    preprocess_for_knn_binary(config, table);
    // check how many tables we have
    assert(table != NULL);
    assert(table->get_total_num_of_table() == 1);
    // assert(config.compression_type == GPUGenie_Config::DELTA);
    std::cout << "Done preprocessing data..." << std::endl; 


    std::cout << "Examining inverted lists...";
    std::vector<GPUGenie::inv_list> *inv_lists = table->inv_lists();
    // check inverted index of the tables using inv_list class
    for (size_t attr_index = 0; attr_index < inv_lists->size(); attr_index++)
    {
        GPUGenie::inv_list invertedList = (*inv_lists)[attr_index];
        int posting_list_length = invertedList.size();
        int posting_list_min = invertedList.min();
        int posting_list_max = invertedList.max();
        Logger::log(Logger::DEBUG, "  attr_index %d, posting_list_length: %d, min: %d, max: %d",
                        attr_index, posting_list_length, posting_list_min, posting_list_max);
        Logger::log(Logger::DEBUG, "    table->get_lowerbound_of_list(%d): %d, table->get_upperbound_of_list(%d): %d", attr_index, table->get_lowerbound_of_list(attr_index),
            attr_index, table->get_upperbound_of_list(attr_index));
    }

    log_table(table);

    std::cout << "Done examining inverted lists..." << std::endl;

    std::cout << "Copying inverted lists for compression..." << std::endl;

    std::vector<int> *ck = table->ck();
    std::vector<int> *inv = table->inv();
    std::vector<int> *inv_index = table->inv_index();
    std::vector<int> *inv_pos = table->inv_pos();

    std::vector<std::vector<uint32_t>> rawInvertedLists;
    size_t rawInvertedListsSize = inv_pos->back();

    auto inv_it = inv->begin();
    size_t prev_inv_pos = *(inv_pos->begin());
    for (auto inv_pos_it = (inv_pos->begin()+1); inv_pos_it != inv_pos->end(); inv_pos_it++)
    {
        size_t offset = (*inv_pos_it) - prev_inv_pos;
        prev_inv_pos = (*inv_pos_it);
        
        std::vector<uint32_t> invList(inv_it, inv_it + offset);
        inv_it += offset;
        rawInvertedLists.push_back(invList);
    }

    log_inv_lists(rawInvertedLists);

    std::cout << "Done copying inverted lists for compression!" << std::endl;
    
    double avg_inv_list_length = ((double)rawInvertedListsSize) / ((double)inv_pos->size());
    Logger::log(Logger::DEBUG, "Total inverted lists: %d, Average length of inv list: %f",
        rawInvertedListsSize, avg_inv_list_length);
    Logger::log(Logger::DEBUG, "Uncompressed size of inv: %d bytes", inv->size() * 4);
    Logger::log(Logger::DEBUG, "Uncompressed size of inv_pos: %d bytes", inv_pos->size() * 4);

    std::cout << std::endl;
    std::cout << std::endl;



    std::cout << "Compressing inverted lists..." << std::endl;
    // for (auto &kv : CODECFactory::scodecmap)
    // {
    // string compression_name = "copy";
    string compression_name = "s4-bp128-d1";
    bool manualDelta = false;

    if (compression_name == "for" || compression_name == "frameofreference"
            || compression_name == "simdframeofreference")
        manualDelta = true;

    // std::cout << "Compressing inverted lists using " << compression_name << "..." << std::endl;
    IntegerCODEC &codec = *CODECFactory::getFromName(compression_name);
    
    size_t compressedsize_total = 0;

    std::vector<std::vector<uint32_t>> comprInvertedLists(rawInvertedLists.size());

    // Compress all inverted lists
    for (size_t i = 0; i < rawInvertedLists.size(); i++)
    {
        comprInvertedLists[i].resize(rawInvertedLists[i].size() + 1024);
        size_t compressedsize = comprInvertedLists[i].size();

        if (manualDelta)
            delta<uint32_t>(static_cast<uint32_t>(0), rawInvertedLists[i].data(), rawInvertedLists[i].size());
        codec.encodeArray(
                rawInvertedLists[i].data(), rawInvertedLists[i].size(),
                comprInvertedLists[i].data(),compressedsize);

        comprInvertedLists[i].resize(compressedsize);
        compressedsize_total += compressedsize;
    }

    std::cout << "Done compressing inverted lists..." << std::endl;


    std::cout << "Preprocessing queries..." << std::endl;

    read_file(*config.query_points, queryFile.c_str(), config.num_of_queries);

    std::vector<query> queries;
    std::vector<int> results;
    std::vector<int> results_count;

    load_query(*table, queries, config);

    knn_search(*table, queries, results, results_count, config);

    logResults(queries, results, results_count);


    void GPUGenie::query::build()
{
    int low, up, min, max, dimShifted;
    int shifter = table->shifter();

    for (query &q : queries)
    {
        std::vector<int> invListsTocount;
        std::vector<query::range> ranges;
        int index = q.index();
        int dimShifted = index << shifter;
        Logger::log(Logger::DEBUG, "Processing query %d", index);

        q.dump(ranges);
        Logger::log(Logger::DEBUG, "  query %d has %d ranges, dimShifted: %d", index, ranges.size(), dimShifted);

        if (ranges.empty())
        {
            Logger::log(Logger::ALERT, "Query %d has no ranges!", index);
            continue;
        }

        for (query::range &r : ranges)
        {
            int low = r.low;
            int up = return.up;
            
            Logger::log(Logger::DEBUG, "  range %d, query: %d, dim: %d, low: %d, up: %d", r.order, r.query, 
                r.dim, r.low, r.up);

            if(low > up || low > table.get_upperbound_of_list(index) ||
                up < table.get_lowerbound_of_list(index))
                continue;

            low = low < table.get_lowerbound_of_list(index) ? table.get_lowerbound_of_list(index) : low;
            up = up > table.get_upperbound_of_list(index) ? table.get_upperbound_of_list(index) : up;

            int min = dimShifted + low - table.get_lowerbound_of_list(index);
            int max = dimShifted + up - table.get_lowerbound_of_list(index);

            // Record ids of inverted lists to be counted
            for (int i = inv_index[min], i < inv_index[max+1]; i++)
                invListsTocount.push_back(i);
        }

    }

}


    // // Decompress all inverted lists
    // unsigned long long time_decompr_start = getTime(), time_decompr_tight_start, time_decompr_tight_stop;
    // double time_decompr_tight = 0.0;
    // for (size_t i = 0; i < rawInvertedLists.size(); i++)
    // {
    //     size_t decompressedsize = rawInvertedLists[i].size();

    //     time_decompr_tight_start = getTime();
    //     codec.decodeArray(
    //         comprInvertedLists[i].data(), comprInvertedLists[i].size(),
    //         rawInvertedLists[i].data(),decompressedsize);
    //     if (manualDelta)
    //         inverseDelta<uint32_t>(static_cast<uint32_t>(0), rawInvertedLists[i].data(),
    //                 rawInvertedLists[i].size());
    //     time_decompr_tight_stop = getTime();

    //     assert(decompressedsize == inv_lists_orig_sizes[i]);
    //     time_decompr_tight += getInterval(time_decompr_tight_start, time_decompr_tight_stop);
    // }
    // unsigned long long time_decompr_stop = getTime();
    // double time_decompr = getInterval(time_decompr_start, time_decompr_stop);

    // std::cout << std::fixed << std::setprecision(3);
    // std::cout << "File: " << dataFile
    //           << ", Compr: " << compression_name
    //           << ", Ratio: "
    //                 << 32.0 * static_cast<double>(compressedsize_total) / static_cast<double>(rawInvertedListsSize)
    //                 << " bpi "
    //           << ", DTime: " << time_decompr
    //           << ", DXTime: " << time_decompr_tight
    //           << std::endl;
    // // }
    // std::cout << "DONE compressing and decompressing inverted lists..." << std::endl;
    // return 0;


    // take a query 

    // uncompress one block of all relevant inverted lists

    // do counting (naively)


    return 0;
}

