#include "hip/hip_runtime.h"
/**
 * Name: test_14.cu
 * Description:
 *  Test counting of queries on CPU. Similar to CPU-Idx in GENIE paper.
 */

#include <GPUGenie.h>

#include <algorithm>
#include <assert.h>
#include <vector>
#include <iostream>

#include <sstream>
#include <stdio.h>

#include "codecfactory.h"
#include "intersection.h"

using namespace GPUGenie;
using namespace SIMDCompressionLib;

const size_t MAX_PRINT_LEN = 128;
const std::string DEFAULT_TEST_DATASET = "../static/sift_20.dat";
const std::string DEFAULT_QUERY_DATASET = "../static/sift_20.csv";

int main(int argc, char* argv[])
{
    Logger::log(Logger::INFO, "Available codecs (SIMDCompressionLib::CODECFactory::scodecmap):");
    for (auto &kv : CODECFactory::scodecmap)
        Logger::log(Logger::INFO, "  %s", kv.first.c_str());
    
    // Logger::log(Logger::INFO, "Available compressions in GENIE (GPUGenie_Config::):");
    // for (auto &kv : GPUGenie_Config::compression_types)
    //     Logger::log(Logger::INFO, "  %s", kv.first);


    string dataFile = DEFAULT_TEST_DATASET;
    if (argc == 2)
        dataFile = std::string(argv[1]);
    string queryFile = DEFAULT_QUERY_DATASET;

    vector<vector<int>> queryPoints;
    inv_table * table = NULL;
    GPUGenie_Config config;

    config.dim = 5;
    config.count_threshold = 14;
    config.num_of_topk = 5;
    config.hashtable_size = 14*config.num_of_topk*1.5;
    config.query_radius = 0;
    config.use_device = 0;
    config.use_adaptive_range = false;
    config.selectivity = 0.0f;

    config.query_points = &queryPoints;
    config.data_points = NULL;

    config.use_load_balance = false;
    config.posting_list_max_length = 6400;
    config.multiplier = 1.5f;
    config.use_multirange = false;

    config.data_type = 1;
    config.search_type = 0;
    config.max_data_size = 0;

    config.num_of_queries = 3;

    config.compression_type = GPUGenie_Config::COMPRESSION_TYPE::NO_COMPRESSION;


    std::cout << "Reading data file " << dataFile << "..." << std::endl;  
    read_file(dataFile.c_str(), &config.data, config.item_num, &config.index, config.row_num);
    assert(config.item_num > 0);
    assert(config.row_num > 0);
    Logger::log(Logger::DEBUG, "config.item_num: %d", config.item_num);
    Logger::log(Logger::DEBUG, "config.row_num: %d", config.row_num);
    std::cout << "Done reading data file!" << std::endl;  


    std::cout << "Preprocessing data (" << config.item_num << " items total)..." << std::endl;  
    preprocess_for_knn_binary(config, table);
    // check how many tables we have
    assert(table != NULL);
    assert(table->get_total_num_of_table() == 1);
    // assert(config.compression_type == GPUGenie_Config::DELTA);
    std::cout << "Done preprocessing data..." << std::endl; 


    std::cout << "Examining inverted lists...";
    std::vector<GPUGenie::inv_list> *inv_lists = table->inv_lists();
    // check inverted index of the tables using inv_list class
    for (size_t attr_index = 0; attr_index < inv_lists->size(); attr_index++)
    {
        GPUGenie::inv_list invertedList = (*inv_lists)[attr_index];
        int posting_list_length = invertedList.size();
        int posting_list_min = invertedList.min();
        int posting_list_max = invertedList.max();
        Logger::log(Logger::DEBUG, "  attr_index %d, posting_list_length: %d, m in: %d, max: %d",
                        attr_index, posting_list_length, posting_list_min, posting_list_max);
        Logger::log(Logger::DEBUG, "    table->get_lowerbound_of_list(%d): %d, table->get_upperbound_of_list(%d): %d", attr_index, table->get_lowerbound_of_list(attr_index),
            attr_index, table->get_upperbound_of_list(attr_index));
    }
    std::cout << "Done examining inverted lists..." << std::endl;

    std::cout << "Copying inverted lists for compression..." << std::endl;

    std::vector<int> *ck = table->ck();
    std::vector<int> *inv = table->inv();
    std::vector<int> *inv_index = table->inv_index();
    std::vector<int> *inv_pos = table->inv_pos();

    std::vector<std::vector<uint32_t>> rawInvertedLists;
    size_t rawInvertedListsSize = inv_pos->back();
    auto inv_it = inv->begin();
    size_t prev_inv_pos = *(inv_pos->begin());
    for (auto inv_pos_it = (inv_pos->begin()+1); inv_pos_it != inv_pos->end(); inv_pos_it++)
    {
        size_t offset = (*inv_pos_it) - prev_inv_pos;
        prev_inv_pos = (*inv_pos_it);
        
        std::vector<uint32_t> invList(inv_it, inv_it + offset);
        inv_it += offset;
        rawInvertedLists.push_back(invList);
    }
    std::cout << "Done copying inverted lists for compression!" << std::endl;
    
    double avg_inv_list_length = ((double)rawInvertedListsSize) / ((double)inv_pos->size());
    Logger::log(Logger::DEBUG, "Total inverted lists: %d, Average length of inv list: %f",
        rawInvertedListsSize, avg_inv_list_length);
    Logger::log(Logger::DEBUG, "Uncompressed size of inv: %d bytes", inv->size() * 4);
    Logger::log(Logger::DEBUG, "Uncompressed size of inv_pos: %d bytes", inv_pos->size() * 4);

    std::cout << std::endl;
    std::cout << std::endl;



    std::cout << "Compressing inverted lists..." << std::endl;
    // for (auto &kv : CODECFactory::scodecmap)
    // {
    // string compression_name = "copy";
    string compression_name = "s4-bp128-d1";
    bool manualDelta = false;

    if (compression_name == "for" || compression_name == "frameofreference"
            || compression_name == "simdframeofreference")
        manualDelta = true;

    // std::cout << "Compressing inverted lists using " << compression_name << "..." << std::endl;
    IntegerCODEC &codec = *CODECFactory::getFromName(compression_name);
    
    size_t compressedsize_total = 0;

    std::vector<std::vector<uint32_t>> comprInvertedLists(rawInvertedLists.size());

    // Compress all inverted lists
    for (size_t i = 0; i < rawInvertedLists.size(); i++)
    {
        comprInvertedLists[i].resize(rawInvertedLists[i].size() + 1024);
        size_t compressedsize = comprInvertedLists[i].size();

        if (manualDelta)
            delta<uint32_t>(static_cast<uint32_t>(0), rawInvertedLists[i].data(), rawInvertedLists[i].size());
        codec.encodeArray(
                rawInvertedLists[i].data(), rawInvertedLists[i].size(),
                comprInvertedLists[i].data(),compressedsize);

        comprInvertedLists[i].resize(compressedsize);
        compressedsize_total += compressedsize;
    }

    std::cout << "Done compressing inverted lists..." << std::endl;


    std::cout << "Preprocessing queries..." << std::endl;

    read_file(*config.query_points, queryFile.c_str(), config.num_of_queries);

    std::vector<query> queries;
    std::vector<int> result;
    std::vector<int> result_count;

    load_query(*table, queries, config);

    for (query &q : queries)
    {
        q.print(MAX_PRINT_LEN);
    }

    knn_search(*table, queries, result, result_count, config);

    {   
        std::stringstream ss;
        auto end = (result_count.size() <= MAX_PRINT_LEN) ? result_count.end() : (result_count.begin()+MAX_PRINT_LEN);
        std::copy(result_count.begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "Results count: %s", ss.str().c_str());
    }
    
    {
        std::stringstream ss;
        auto end = (result.size() <= MAX_PRINT_LEN) ? result.end() : (result.begin() + MAX_PRINT_LEN); 
        std::copy(result.begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "Results: %s", ss.str().c_str());
    }

    // // Decompress all inverted lists
    // unsigned long long time_decompr_start = getTime(), time_decompr_tight_start, time_decompr_tight_stop;
    // double time_decompr_tight = 0.0;
    // for (size_t i = 0; i < rawInvertedLists.size(); i++)
    // {
    //     size_t decompressedsize = rawInvertedLists[i].size();

    //     time_decompr_tight_start = getTime();
    //     codec.decodeArray(
    //         comprInvertedLists[i].data(), comprInvertedLists[i].size(),
    //         rawInvertedLists[i].data(),decompressedsize);
    //     if (manualDelta)
    //         inverseDelta<uint32_t>(static_cast<uint32_t>(0), rawInvertedLists[i].data(),
    //                 rawInvertedLists[i].size());
    //     time_decompr_tight_stop = getTime();

    //     assert(decompressedsize == inv_lists_orig_sizes[i]);
    //     time_decompr_tight += getInterval(time_decompr_tight_start, time_decompr_tight_stop);
    // }
    // unsigned long long time_decompr_stop = getTime();
    // double time_decompr = getInterval(time_decompr_start, time_decompr_stop);

    // std::cout << std::fixed << std::setprecision(3);
    // std::cout << "File: " << dataFile
    //           << ", Compr: " << compression_name
    //           << ", Ratio: "
    //                 << 32.0 * static_cast<double>(compressedsize_total) / static_cast<double>(rawInvertedListsSize)
    //                 << " bpi "
    //           << ", DTime: " << time_decompr
    //           << ", DXTime: " << time_decompr_tight
    //           << std::endl;
    // // }
    // std::cout << "DONE compressing and decompressing inverted lists..." << std::endl;
    // return 0;


    // take a query 

    // uncompress one block of all relevant inverted lists

    // do counting (naively)


    return 0;
}

