#include "hip/hip_runtime.h"
/** Name: test_1.cu
 * Description:
 *   the most basic test
 *   sift data
 *   data is from csv file
 *   query is from csv file, single range
 *
 *
 */


#include "GPUGenie.h"

#include <algorithm>
#include <assert.h>
#include <vector>
#include <iostream>

#include <sstream>
#include <stdio.h>
#include <stdlib.h>

#include "../fastpfor/headers/codecfactory.h"
#include "../fastpfor/headers/deltautil.h"

using namespace std;
using namespace GPUGenie;
using namespace FastPForLib;

int main(int argc, char* argv[])
{

    // We pick a CODEC
 
    IntegerCODEC &codec = *CODECFactory::getFromName("simdfastpfor256");
    // could use others, e.g., "simdbinarypacking", "varintg8iu"
    ////////////
    //
    // create a container with some integers in it
    //
    // for this example, we will not assume that the
    // integers are in sorted order
    //
    // (Note: You don't need to use a vector.)
    //
    size_t N = 10 * 1000;
    std::vector<uint32_t> mydata(N);
    for (uint32_t i = 0; i < N; i += 150)
        mydata[i] = i;
    //
    // the vector mydata could contain anything, really
    //
    ///////////
    //
    // You need some "output" container. You are responsible
    // for allocating enough memory.
    //
    std::vector<uint32_t> compressed_output(N + 1024);
    // N+1024 should be plenty
    //
    //
    size_t compressedsize = compressed_output.size();
    codec.encodeArray(mydata.data(), mydata.size(), compressed_output.data(),
                    compressedsize);
    //
    // if desired, shrink back the array:
    compressed_output.resize(compressedsize);
    compressed_output.shrink_to_fit();
    // display compression rate:
    std::cout << std::setprecision(3);
    std::cout << "You are using "
            << 32.0 * static_cast<double>(compressed_output.size()) /
                   static_cast<double>(mydata.size())
            << " bits per integer. " << std::endl;
    //
    // You are done!... with the compression...
    //

    ///
    // decompressing is also easy:
    //
    std::vector<uint32_t> mydataback(N);
    size_t recoveredsize = mydataback.size();
    //
    codec.decodeArray(compressed_output.data(), compressed_output.size(),
                mydataback.data(), recoveredsize);
    mydataback.resize(recoveredsize);
    //
    // That's it!
    //
    if (mydataback != mydata)
    throw std::runtime_error("bug!");

    // If you need to use differential coding, you can use
    // calls like these to get the deltas and recover the original
    // data from the deltas:
    Delta::deltaSIMD(mydata.data(), mydata.size());
    Delta::inverseDeltaSIMD(mydata.data(), mydata.size());
    // be mindful of CPU caching issues

    string dataFile = "../static/sift_20.csv";
    string queryFile = "../static/sift_20.csv";
    vector<vector<int> > queries;
    vector<vector<int> > data;
    inv_table * table = NULL;
    GPUGenie_Config config;

    config.dim = 5;
    config.count_threshold = 14;
    config.num_of_topk = 5;
    config.hashtable_size = 14*config.num_of_topk*1.5;
    config.query_radius = 0;
    config.use_device = 0;
    config.use_adaptive_range = false;
    config.selectivity = 0.0f;

    config.query_points = &queries;
    config.data_points = &data;

    config.use_load_balance = false;
    config.posting_list_max_length = 6400;
    config.multiplier = 1.5f;
    config.use_multirange = false;

    config.data_type = 0;
    config.search_type = 0;
    config.max_data_size = 0;

    config.num_of_queries = 3;

    assert(config.compression_type == GPUGenie_Config::NO_COMPRESSION);

    read_file(data, dataFile.c_str(), -1);
    read_file(queries, queryFile.c_str(), config.num_of_queries);

    preprocess_for_knn_csv(config, table);

    // check how many tables we have
    assert(table != NULL);
    assert(table->get_total_num_of_table() == 1);

    std::vector<GPUGenie::inv_list> *inv_lists = table->inv_lists();
    std::cout << "inv_lists.size(): " << inv_lists->size() << std::endl;

    // check inverted index of the tables using inv_list class
    for (int attr_index = 0; attr_index < config.dim; attr_index++)
    {
        GPUGenie::inv_list invertedList = (*inv_lists)[attr_index];
        int posting_list_length = invertedList.size();
        int posting_list_min = invertedList.min();
        int posting_list_max = invertedList.max();
        Logger::log(Logger::DEBUG, "attr_index %d, posting_list_length: %d, min: %d, max: %d",
                        attr_index, posting_list_length, posting_list_min, posting_list_max);
        for (int value = posting_list_min; value <= posting_list_max; ++value)
        {
            vector<int> *docIds = invertedList.index(value);
            if (docIds->size())
            {
                std::stringstream strDocIds;
                std::copy(docIds->begin(), docIds->end(), std::ostream_iterator<int>(strDocIds, " "));
                Logger::log(Logger::DEBUG, "  value: %d, docIds: %s", value, strDocIds.str().c_str());
            }
        }
    }

    // check what get_lowerbounf_of_list does exactly
    for (int attr_index = 0; attr_index < config.dim; attr_index++)
    {
        Logger::log(Logger::DEBUG, "table->get_lowerbound_of_list(%d): %d, table->get_upperbound_of_list(%d): %d", attr_index, table->get_lowerbound_of_list(attr_index),
            attr_index, table->get_upperbound_of_list(attr_index));
    }

    std::stringstream ss;

    std::vector<int> *ck = table->ck();
    if (ck)
    {
        auto end = (ck->size() <= 256) ? ck->end() : (ck->begin() + 256); 
        std::copy(ck->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "CK:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }

    std::vector<int> *inv = table->inv();
    if (inv)
    {
        auto end = (inv->size() <= 256) ? inv->end() : (inv->begin() + 256); 
        std::copy(inv->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "INV:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }

    std::vector<int> *inv_index = table->inv_index();
    if (inv_index)
    {
        auto end = (inv_index->size() <= 256) ? inv_index->end() : (inv_index->begin() + 256); 
        std::copy(inv_index->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "INV_INDEX:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }


    std::vector<int> *inv_pos = table->inv_pos();
    if (inv_pos)
    {
        auto end = (inv_pos->size() <= 256) ? inv_pos->end() : (inv_pos->begin() + 256); 
        std::copy(inv_pos->begin(), end, std::ostream_iterator<int>(ss, " "));
        Logger::log(Logger::DEBUG, "INV_POS:\n %s", ss.str().c_str());
        ss.str(std::string());
        ss.clear();
    }

    // check values / print values into a file
    // is there a function that can be used for that?

    /**test for table*/
    vector<int>& _inv = *table[0].inv();
    assert(_inv[0] == 8);
    assert(_inv[1] == 9);
    assert(_inv[2] == 7);
    assert(_inv[3] == 0);
    assert(_inv[4] == 2);
    assert(_inv[5] == 4);

    vector<int> result;
    vector<int> result_count;
    knn_search_after_preprocess(config, table, result, result_count);

    assert(result[0] == 0);
    assert(result_count[0] == 5);

    assert(result[1] == 4);
    assert(result_count[1] == 2);

    assert(result[5] == 1);
    assert(result_count[5] == 5);
    
    assert(result[10] == 2);
    assert(result_count[10] == 5);
    delete[] table;
    return 0;
}

