#include "hip/hip_runtime.h"
/**
 * Name: test_14.cu
 * Description:
 *  Test counting of queries on CPU. Similar to CPU-Idx in GENIE paper.
 */

#include <GPUGenie.h>

#include <algorithm>
#include <assert.h>
#include <vector>
#include <iostream>

#include <sstream>
#include <stdio.h>

#include "codecfactory.h"
#include "intersection.h"

using namespace GPUGenie;
using namespace SIMDCompressionLib;

const std::string DEFAULT_TEST_DATASET = "../static/sift_20.dat";
const std::string DEFAULT_QUERY_DATASET = "../static/sift_20.csv";


/**
 *  Sorts GENIE top-k results for each query independently. The top-k results returned from GENIE are in random order,
 *  and if (top-k > number of resutls with match count greater than 0), then remaining docIds in the result vector are
 *  set to 0, thus the result and count vectors cannot be soreted conventionally. 
 */
void sortGenieResults(GPUGenie::GPUGenie_Config &config, std::vector<int> &gpuResultIdxs,
                            std::vector<int> &gpuResultCounts)
{
    std::vector<int> gpuResultHelper(config.num_of_topk),
                     gpuResultHelperTmp(config.num_of_topk);
    for (int queryIndex = 0; queryIndex < config.num_of_queries; queryIndex++)
    {
        int offsetBegin = queryIndex*config.num_of_topk;
        int offsetEnd = (queryIndex+1)*config.num_of_topk;
        // Fint first zero element
        auto firstZeroIt = std::find(gpuResultCounts.begin()+offsetBegin, gpuResultCounts.begin()+offsetEnd, 0);
        // Only sort elements that have non-zero count. This is because GENIE does not return indexed of elements with
        // zero count
        offsetEnd = std::min(offsetEnd,static_cast<int>(
                                    std::distance(gpuResultCounts.begin(),firstZeroIt)));
        
        // Create helper index from 0 to offsetEnd-offsetBegin
        gpuResultHelper.resize(offsetEnd-offsetBegin);
        gpuResultHelperTmp.resize(offsetEnd-offsetBegin);
        std::iota(gpuResultHelper.begin(), gpuResultHelper.end(),0);

        // Sort the helper index according to gpuResultCounts[...+offsetBegin]
        std::sort(gpuResultHelper.begin(),
                  gpuResultHelper.end(),
                  [&gpuResultCounts,offsetBegin](int lhs, int rhs){
                        return (gpuResultCounts[lhs+offsetBegin] > gpuResultCounts[rhs+offsetBegin]);
                    });

        // Shuffle the gpuResultIdxs according to gpuResultHelper
        for (size_t i = 0; i < gpuResultHelper.size(); i++)
            gpuResultHelperTmp[i] = gpuResultIdxs[gpuResultHelper[i]+offsetBegin];
        // Copy back into gpuResultIndex
        std::copy(gpuResultHelperTmp.begin(), gpuResultHelperTmp.end(), gpuResultIdxs.begin()+offsetBegin);

        // Shuffle the gpuResultCounts according to gpuResultHelper
        for (size_t i = 0; i < gpuResultHelper.size(); i++)
            gpuResultHelperTmp[i] = gpuResultCounts[gpuResultHelper[i]+offsetBegin];
        // Copy back into gpuResultIndex
        std::copy(gpuResultHelperTmp.begin(), gpuResultHelperTmp.end(), gpuResultCounts.begin()+offsetBegin); 
    }
}

int main(int argc, char* argv[])
{
    Logger::log(Logger::INFO, "Available compressions in GENIE (GPUGenie_Config::COMPRESSION_NAMES):");
    for (std::string &compr : GPUGenie_Config::COMPRESSION_NAMES)
        Logger::log(Logger::INFO, "  %s", compr.c_str());


    string dataFile = DEFAULT_TEST_DATASET;
    if (argc == 2)
        dataFile = std::string(argv[1]);
    string queryFile = DEFAULT_QUERY_DATASET;

    vector<vector<int>> queryPoints;
    GPUGenie_Config config;

    config.dim = 5;
    config.count_threshold = 14;
    config.num_of_topk = 10;
    config.hashtable_size = 14*config.num_of_topk*1.5;
    config.query_radius = 0;
    config.use_device = 0;
    config.use_adaptive_range = false;
    config.selectivity = 0.0f;

    config.query_points = &queryPoints;
    config.data_points = NULL;

    config.use_load_balance = false;
    config.posting_list_max_length = 6400;
    config.multiplier = 1.5f;
    config.use_multirange = false;

    config.data_type = 1;
    config.search_type = 0;
    config.max_data_size = 0;

    config.num_of_queries = 3;




    std::cout << "Reading data file " << dataFile << "..." << std::endl;  
    read_file(dataFile.c_str(), &config.data, config.item_num, &config.index, config.row_num);
    assert(config.item_num > 0);
    assert(config.row_num > 0);
    Logger::log(Logger::DEBUG, "config.item_num: %d", config.item_num);
    Logger::log(Logger::DEBUG, "config.row_num: %d", config.row_num);



    std::cout << "--------------------------------------------------------" << std::endl;
    std::cout << "Establishing reference solution on uncompressed table..." << std::endl;

    std::cout << "Preprocessing data (" << config.item_num << " items total)..." << std::endl;  
    inv_table * refTable = NULL;
    preprocess_for_knn_binary(config, refTable);
    assert(refTable != NULL);
    assert(refTable->get_total_num_of_table() == 1); // check how many tables we have

    std::cout << "Examining inverted lists...";
    std::vector<GPUGenie::inv_list> *invLists = refTable->inv_lists();
    // check inverted index of the tables using inv_list class
    for (size_t attr_index = 0; attr_index < invLists->size(); attr_index++)
    {
        GPUGenie::inv_list invertedList = (*invLists)[attr_index];
        int posting_list_length = invertedList.size();
        int posting_list_min = invertedList.min();
        int posting_list_max = invertedList.max();
        Logger::log(Logger::DEBUG, "  attr_index %d, posting_list_length: %d, min: %d, max: %d",
                        attr_index, posting_list_length, posting_list_min, posting_list_max);
        Logger::log(Logger::DEBUG, "    table->get_lowerbound_of_list(%d): %d, table->get_upperbound_of_list(%d): %d", attr_index, refTable->get_lowerbound_of_list(attr_index),
            attr_index, refTable->get_upperbound_of_list(attr_index));
    }

    Logger::logTable(Logger::DEBUG,refTable);

    std::cout << "Loading queries..." << std::endl;
    read_file(*config.query_points, queryFile.c_str(), config.num_of_queries);
    std::vector<query> refQueries;
    load_query(*refTable, refQueries, config);

    std::cout << "Running KNN on GPU..." << std::endl;
    std::vector<int> refResultIdxs;
    std::vector<int> refResultCounts;
    knn_search(*refTable, refQueries, refResultIdxs, refResultCounts, config);
    // Top k results from GENIE don't have to be sorted. In order to compare with CPU implementation, we have to
    // sort the results manually from individual queries => sort subsequence relevant to each query independently
    sortGenieResults(config, refResultIdxs, refResultCounts);
    Logger::log(Logger::DEBUG, "Results from GENIE:");
    Logger::logResults(Logger::DEBUG, refQueries, refResultIdxs, refResultCounts);



    std::cout << "---------------------------" << std::endl;
    std::cout << "Testing compressed table..." << std::endl;

    config.compression = "d1"; // "d1" stands for sequential delta

    std::cout << "Preprocessing data (" << config.item_num << " items total)..." << std::endl;

    inv_table * table = NULL;
    inv_compr_table * comprTable = NULL;
    preprocess_for_knn_binary(config, table); // this returns inv_compr_table if config.compression is set
    assert(table != NULL);
    assert(table->build_status() == inv_table::builded);
    assert(table->get_total_num_of_table() == 1); // check how many tables we have
    comprTable = dynamic_cast<inv_compr_table*>(table);
    assert(config.posting_list_max_length == (int)comprTable->getUncompressedPostingListMaxLength());
    assert(config.compression == comprTable->getCompression()); // check the compression was actually used in the table

    std::cout << "Examining compressed index..." << std::endl;

    std::vector<int> *uncompressedInv = comprTable->uncompressedInv();
    std::vector<int> *uncompressedInvPos = comprTable->uncompressedInvPos();
    std::vector<uint32_t> *compressedInv = comprTable->compressedInv();
    std::vector<int> *compressedInvPos = comprTable->compressedInvPos();
    // the last elm in inv_pos should be the compressed size, which is <= to the original size
    assert(compressedInvPos->size() == uncompressedInvPos->size());
    assert(compressedInvPos->back() == (int)compressedInv->size()); 
    assert(compressedInvPos->back() <= uncompressedInvPos->back()); // compression should not enlarge data
    assert(compressedInv == reinterpret_cast<std::vector<uint32_t>*>(comprTable->inv())); // test alias function
    assert(compressedInvPos == comprTable->inv_pos()); // test alias function

    double avg_inv_list_length = ((double)uncompressedInv->size()) / ((double)uncompressedInvPos->size());
    double avg_compr_inv_list_length = ((double)compressedInv->size()) / ((double)compressedInvPos->size());
    Logger::log(Logger::DEBUG,
            "Uncompressed inverted list length: %d, Inverted lists: %d, Average length of uncompressed inv list: %f",
            uncompressedInv->size(), uncompressedInvPos->size(), avg_inv_list_length);
    Logger::log(Logger::DEBUG, "Compressed size of posting lists array Z: %d bytes", compressedInv->size() * 4);
    Logger::log(Logger::DEBUG, "Uncompressed size of compressedInvPos index: %d bytes", compressedInvPos->size() * 4);
    Logger::log(Logger::DEBUG, "Average size of compressed posting list: %d", avg_compr_inv_list_length);


    std::cout << "Loading queries..." << std::endl;
    std::vector<query> queries;
    load_query(*comprTable, queries, config);

    std::vector<int> resultIdxs;
    std::vector<int> resultCounts;

    std::cout << "Running KNN on GPU (compression, naive counting)" << std::endl;
    std::cout << "KNN_SEARCH_CPU"
              << ", file: " << dataFile << " (" << config.row_num << " rows)" 
              << ", queryFile: " << queryFile << " (" << config.num_of_queries << " queries)"
              << ", topk: " << config.num_of_topk
              << ", compression: " << config.compression
              << ", ";

    knn_search(*comprTable, queries, resultIdxs, resultCounts, config);
    sortGenieResults(config, resultIdxs, resultCounts);

    Logger::log(Logger::DEBUG, "Results from GPU naive decompressed counting:");
    Logger::logResults(Logger::DEBUG, queries, resultIdxs, resultCounts);



    std::cout<< "---------------------------------------------" << std::endl;
    std::cout<< "Comparing reference and compressed results..." << std::endl;

    // Compare the first docId from the GPU and CPU results -- note since we use points from the data file
    // as queries, One of the resutls is a full-dim count match (self match), which is what we compare here.
    assert(refResultIdxs[0 * config.num_of_topk] == resultIdxs[0 * config.num_of_topk]
        && refResultCounts[0 * config.num_of_topk] == resultCounts[0 * config.num_of_topk]);
    assert(refResultIdxs[1 * config.num_of_topk] == resultIdxs[1 * config.num_of_topk]
        && refResultCounts[1 * config.num_of_topk] == resultCounts[1 * config.num_of_topk]);
    assert(refResultIdxs[2 * config.num_of_topk] == resultIdxs[2 * config.num_of_topk]
        && refResultCounts[2 * config.num_of_topk] == resultCounts[2 * config.num_of_topk]);

    return 0;
}

