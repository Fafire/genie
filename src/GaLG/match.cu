#include "hip/hip_runtime.h"
#include "match.h"
#include <cmath>
#include <sys/time.h>
#include <algorithm>

#ifndef GaLG_device_THREADS_PER_BLOCK
#define GaLG_device_THREADS_PER_BLOCK 256
#endif

#define cudaCheckErrors( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{

    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }


    return;
}

#define OFFSETS_TABLE_16 {0u,3949349u,8984219u,9805709u,7732727u,1046459u,9883879u,4889399u,2914183u,3503623u,1734349u,8860463u,1326319u,1613597u,8604269u,9647369u}

#define NULL_AGE 0

#define DEBUG
//#define DEBUG_VERBOSE

typedef u64 T_HASHTABLE;
typedef u32 T_KEY;
typedef u32 T_AGE;

u64 getTime()
{
 struct timeval tv;

 gettimeofday(&tv, NULL);

 u64 ret = tv.tv_usec;

 /* Adds the seconds (10^0) after converting them to milliseconds (10^-3) */
 ret += (tv.tv_sec * 1000 * 1000);

 return ret;
}

float getInterval(u64 start, u64 stop)
{
	return ((float)(stop - start)) / 1000;
}

namespace GaLG
{
  namespace device
  {

     const u32 DEFAULT_GROUP_SIZE            = 192u;
    
     const u32 KEY_TYPE_BITS                 = 32u;
     const u32 KEY_TYPE_MASK                 = u32( u64((1ull) << KEY_TYPE_BITS) - 1u );
     const u32 PACKED_KEY_TYPE_MASK          = u32( u64((1ull) << KEY_TYPE_BITS) - 1u );
     const u32 KEY_TYPE_RANGE                = u32( u64((1ull) << KEY_TYPE_BITS) - 2u );
     const u32 UNDEFINED_KEY                 = u32( u64((1ull) << KEY_TYPE_BITS) - 1u );
     const u32 PACKED_UNDEFINED_KEY          = u32( u64((1ull) << KEY_TYPE_BITS) - 1ul);
    
     const u32 ATTACH_ID_TYPE_BITS           = 28u;
     const u32 ATTACH_ID_TYPE_MASK           = u32( u64((1ull) << ATTACH_ID_TYPE_BITS) - 1ul );
     const u32 UNDEFINED_ATTACH_ID           = u32( u64((1ull) << ATTACH_ID_TYPE_BITS) - 1ul );
     const u32 MAX_ATTACH_ID_TYPE            = u32( u64((1ull) << ATTACH_ID_TYPE_BITS) - 2ul );
    
    const u32 KEY_TYPE_AGE_MASK        = 15u;
    const u32 KEY_TYPE_AGE_BITS        = 4u;
    const u32 KEY_TYPE_INIT_AGE        = 1u;
    const u32 KEY_TYPE_NULL_AGE        = 0u;
    const u32 KEY_TYPE_MAX_AGE         = 16u;
    const u32 KEY_TYPE_MAX_AGE_MASK    = 4u;
    const u32 KEY_TYPE_MAX_AGE_BITS    = 4u;
    
    __device__ __constant__ u32 offsets[16];
    
    __inline__ __host__ __device__
    T_KEY
    get_key_pos(T_HASHTABLE key)
    {
      return key & KEY_TYPE_MASK;
    }
    
    __inline__ __host__ __device__
    T_AGE
    get_key_age(T_HASHTABLE key)
    {
      return ((key) >> (ATTACH_ID_TYPE_BITS + KEY_TYPE_BITS));
    }

    __host__ __inline__ __device__
    u32
    get_key_attach_id(T_HASHTABLE key)
    {
      return ((key) >> (KEY_TYPE_BITS)) & ATTACH_ID_TYPE_MASK;
    }
    __host__ __inline__ __device__
    T_HASHTABLE
    pack_key_pos(T_KEY p)
    {
      return ((p) & KEY_TYPE_MASK);
    }
    __host__ __inline__ __device__
    T_HASHTABLE
    pack_key_pos_and_attach_id_and_age(T_KEY p, u32 i, T_AGE a)
    {
      return u64(((u64(a) << (ATTACH_ID_TYPE_BITS + KEY_TYPE_BITS))) + ((u64(i) & ATTACH_ID_TYPE_MASK) << (KEY_TYPE_BITS)) + u64(p & KEY_TYPE_MASK));
    }
    
    __inline__ __host__ __device__
    u32
    hash(T_KEY key, T_AGE age, int hash_table_size){
      return (offsets[age] + key) % hash_table_size;
    }
    
    
    __inline__ __device__
    void
    access_kernel(u32 id,
                  T_HASHTABLE* htable,
                  int hash_table_size,
                  u32 * index,
                  int * key_found,
                  u32 max_age)
    {
      u32 location;
      T_HASHTABLE out_key;
      T_AGE age = KEY_TYPE_NULL_AGE;
      
      location = hash(id, age, hash_table_size);

#ifdef DEBUG_VERBOSE
        printf(">>> [b%d t%d]Access: hash to %u. id: %u, age: %u.\n", blockIdx.x, threadIdx.x, location, id, age);
#endif

      out_key = htable[location];
      
      if(get_key_pos(out_key) == id
      		&& get_key_age(out_key) != KEY_TYPE_NULL_AGE
      		&& get_key_age(out_key) < max_age){
        * key_found = 1;
        * index = get_key_attach_id(out_key);

#ifdef DEBUG_VERBOSE
        printf(">>> [b%d t%d]Access: Entry found in hash table.\n>>> access_id: %u, index: %u, age: %u, hash: %u\n", blockIdx.x, threadIdx.x, id, index, age, location);
#endif

        return;
      }
      
      //Key at root location is packed with its max age
      // in its hashing sequence.
      max_age = get_key_age(out_key);
      
      //Loop until max_age
      while(age < max_age){
        age ++;
        location = hash(id, age, hash_table_size);
        out_key = htable[location];
        
#ifdef DEBUG_VERBOSE
        printf(">>> [b%d t%d]Access: hash to %u. id: %u, age: %u.\n", blockIdx.x, threadIdx.x, location, id, age);
#endif

        if(get_key_pos(out_key) == id
        		&& get_key_age(out_key) != KEY_TYPE_NULL_AGE
        		&& get_key_age(out_key) < max_age){
          * key_found = 1;
          * index = get_key_attach_id(out_key);
#ifdef DEBUG_VERBOSE
        printf(">>> [b%d t%d]Access: Entry found in hash table.\n>>> access_id: %u, index: %u, age: %u, hash: %u\n", blockIdx.x, threadIdx.x, id, index, age, location);
#endif
          return;
        }
      }
      
      //Entry not found. Return NULL key.
      * key_found = 0;
      * index = (u32)-1;
      
    }
    
    
    __inline__ __device__
    void
    hash_kernel(u32 id,
                T_HASHTABLE* htable,
                T_AGE* max_table,
                int hash_table_size,
                u32* value_index,
                T_AGE max_age,
                u32 * value)
    {

      u32 my_value = atomicAdd(value, 1);

#ifdef DEBUG_VERBOSE
      printf(">>> [b%d t%d]Insertion starts. my_value is %u, id is %d.\n", blockIdx.x, threadIdx.x, my_value, id);
#endif

      *value_index = my_value;
      
      u32 location;
      u32 root_location;
      T_HASHTABLE evicted_key;
      T_AGE age = KEY_TYPE_NULL_AGE;
      T_HASHTABLE key = pack_key_pos_and_attach_id_and_age(id,
                                                   my_value,
                                                   KEY_TYPE_INIT_AGE);
      
      //Loop until max_age
      while(age < max_age){

        //evict key at current age-location
        //Update it if the to-be-inserted key is of a larger age
        location = hash(get_key_pos(key), age, hash_table_size);
        evicted_key = atomicMax(&htable[location], key);
#ifdef DEBUG_VERBOSE
        printf(">>> [b%d t%d]Insertion: hash to %u. id: %u, age: %u, my_value: %u, evicted key %llu.\n", blockIdx.x, threadIdx.x, location, id, age, my_value, evicted_key);
#endif
        if(evicted_key < key){
#ifdef DEBUG_VERBOSE
        printf(">>> [b%d t%d]Insertion: Key id %u evicted at age %u!\n", blockIdx.x, threadIdx.x, get_key_pos(evicted_key), age);
#endif
          root_location = hash(get_key_pos(key), 0u, hash_table_size);
          atomicMax(&max_table[root_location], get_key_age(key));
          
          //If not an empty location, loop again to insert the evicted key.
          if(get_key_age(evicted_key) > 0u)
          {
            key = evicted_key;
            age = get_key_age(evicted_key);
          }
          //If empty location, finish the insertion.
          else
          {
#ifdef DEBUG_VERBOSE
        	printf(">>> [b%d t%d]Insertion finished.\n>>> access_id: %u, my_value: %u.\n", blockIdx.x, threadIdx.x, id, my_value);
#endif
            break;
          }
        }
        else
        {
          //Increase age and try again.
          age++;
          key = pack_key_pos_and_attach_id_and_age(get_key_pos(key), get_key_attach_id(key), age);
        }
      }
    }
    
    __global__
    void
    match(int m_size,
          int i_size,
          int hash_table_size,
          int* d_ck,
          int* d_inv,
          query::dim* d_dims,
          T_HASHTABLE* hash_table_list,
          data_t* data_table_list,
          T_AGE* age_table_list,
          T_AGE max_age,
          u32 * value_idx)
    {
      int query_index =blockIdx.x / m_size;
      query::dim* q = &d_dims[blockIdx.x];
      
      T_HASHTABLE* hash_table = &hash_table_list[query_index*hash_table_size];
      T_AGE* age_table = &age_table_list[query_index*hash_table_size];
      data_t* data_table = &data_table_list[query_index*hash_table_size];
      u32 * my_value_idx = &value_idx[query_index];
      u32 index, access_id;

      int min, max;
      min = q->low;
      max = q->up;
      if (min > max)
        return;

      min < 1 ? min = 0 : min = d_ck[min - 1];
      max = d_ck[max];

      int loop = (max - min) / GaLG_device_THREADS_PER_BLOCK + 1;


      int i;
      for (i = 0; i < loop; i++)
        {
          if (threadIdx.x + i * GaLG_device_THREADS_PER_BLOCK + min < max)
            {
              access_id = d_inv[threadIdx.x + i * GaLG_device_THREADS_PER_BLOCK + min];

              int key_found = 0;
              
              //Try to find the entry in hash tables
              access_kernel(access_id,
                            hash_table,
                            hash_table_size,
                            &index,
                            &key_found,
                            max_age);
              
              if(!key_found)
              {
                //Insert the key into hash table
                //access_id and its location are packed into a packed key
                hash_kernel(access_id,
                            hash_table,
                            age_table,
                            hash_table_size,
                            &index,
                            max_age,
                            my_value_idx);
              }

              data_table[index].id = access_id;
              atomicAdd(&(data_table[index].count), 1u);
              atomicAdd(&(data_table[index].aggregation),q->weight);
            }
        }
    }
  }
}

void
GaLG::match(inv_table& table,
            vector<query>& queries,
            device_vector<data_t>& d_data,
            int& hash_table_size)
throw (int)
{
#ifdef DEBUG
	u64 match_stop, match_elapsed, match_start;
	hipEvent_t kernel_start, kernel_stop;
	float kernel_elapsed;
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_stop);
	match_start = getTime();
	printf("[  0%] Starting matching...\n");
	printf("[ 10%] Fetching and packing data...\n");
#endif

  if (table.build_status() == inv_table::not_builded)
    throw inv_table::not_builded_exception;
  vector<query::dim> dims;
  int i;
  for (i = 0; i < queries.size(); i++)
    {
      if (queries[i].ref_table() != &table)
        throw inv_table::not_matched_exception;
      if (table.build_status() == inv_table::builded)
        queries[i].build();
      else if (table.build_status() == inv_table::builded_compressed)
        queries[i].build_compressed();
      queries[i].dump(dims);
    }
  int total = table.i_size() * queries.size();

#ifdef DEBUG
	printf("[ 20%] Declaring device memory...\n");
#endif

  device_vector<int> d_ck(*table.ck());
  int* d_ck_p = raw_pointer_cast(d_ck.data());

  device_vector<int> d_inv(*table.inv());
  int* d_inv_p = raw_pointer_cast(d_inv.data());

  device_vector<query::dim> d_dims(dims);
  query::dim* d_dims_p = raw_pointer_cast(d_dims.data());
  
  if(hash_table_size <= 0){
	  hash_table_size =(int)sqrt((double)table.i_size())*2;
	  if(hash_table_size < 11) hash_table_size = 11;
  }

  
#ifdef DEBUG
  printf("[ 30%] Allocating device memory to tables...\n");
#endif

  data_t null_data;
  null_data.count = 0u;
  null_data.aggregation = 0.0f;
  null_data.id = 0u;
  std::vector<data_t> h_null_data(queries.size() * hash_table_size, null_data);

#ifdef DEBUG
  for(i = 0; i < h_null_data.size();++i)
  {
	  null_data = h_null_data[i];
	  if(!(null_data.count == 0u && null_data.aggregation == 0.0f && null_data.id == 0u))
	  {
		  printf(">>> Null data table initialization error!\n");
		  h_null_data[i].count = 0u;
		  h_null_data[i].aggregation = 0.0f;
		  h_null_data[i].id = 0u;
	  }
  }
#endif

  std::vector<T_HASHTABLE> h_hash_table(queries.size()*hash_table_size, 0ull);

  T_HASHTABLE* d_hash_table;
  cudaCheckErrors(hipMalloc(&d_hash_table, sizeof(T_HASHTABLE)*queries.size()*hash_table_size));
  hipMemcpy(d_hash_table, &h_hash_table.front(), sizeof(T_HASHTABLE)*queries.size()*hash_table_size, hipMemcpyHostToDevice);
  data_t* d_data_table;
  cudaCheckErrors(hipMalloc(&d_data_table, sizeof(data_t)*queries.size()*hash_table_size));
  hipMemcpy(d_data_table, &h_null_data.front(), sizeof(data_t)*queries.size()*hash_table_size, hipMemcpyHostToDevice);
  T_AGE* d_max_table;
  cudaCheckErrors(hipMalloc(&d_max_table, sizeof(T_AGE)*queries.size()*hash_table_size));
  hipMemset(&d_max_table, 0u,sizeof(T_AGE)*queries.size()*hash_table_size);

  u32 max_age = 16u;
  
#ifdef DEBUG
  printf("[ 33%] Copying memory to symbol...\n");
#endif

  u32 h_offsets[16] = OFFSETS_TABLE_16;
  
  cudaCheckErrors(hipMemcpyToSymbol(HIP_SYMBOL(GaLG::device::offsets), h_offsets, sizeof(u32)*16, 0, hipMemcpyHostToDevice));
  
#ifdef DEBUG
  printf("[ 36%] Creating incremental index variable...\n");
#endif

  u32 * d_value_idx;
  cudaCheckErrors(hipMalloc(&d_value_idx, sizeof(u32) * queries.size()));
  std::vector<u32> h_value_idx(queries.size(), 0u);
  cudaCheckErrors(hipMemcpy(d_value_idx, &h_value_idx.front(), sizeof(u32)*queries.size(), hipMemcpyHostToDevice));
  
#ifdef DEBUG

#endif

#ifdef DEBUG
  printf("[ 40%] Starting match kernels...\n");
  hipEventRecord(kernel_start);
#endif

  device::match<<<dims.size(), GaLG_device_THREADS_PER_BLOCK>>>
  (table.m_size(),
   table.i_size(),
   hash_table_size,
   d_ck_p,
   d_inv_p,
   d_dims_p,
   d_hash_table,
   d_data_table,
   d_max_table,
   max_age,
   d_value_idx);
  
#ifdef DEBUG
  hipEventRecord(kernel_stop);
  printf("[ 90%] Starting memory copy to host...\n");
#endif

  cudaCheckErrors(hipDeviceSynchronize());

  //cudaCheckErrors(hipGetLastError());

  d_data.clear();
  d_data.resize(queries.size()*hash_table_size);
  thrust::copy(d_data_table,
               d_data_table + hash_table_size*queries.size(),
               d_data.begin());

  //cudaCheckErrors(hipGetLastError());

  hipMemcpy(&h_value_idx.front(), d_value_idx, sizeof(u32)*queries.size(),hipMemcpyDeviceToHost);

#ifdef DEBUG
  printf("[ 95%] Cleaning up memory...\n");
#endif

  cudaCheckErrors(hipFree(d_data_table));
  cudaCheckErrors(hipFree(d_hash_table));
  cudaCheckErrors(hipFree(d_max_table));
  cudaCheckErrors(hipFree(d_value_idx));
  
#ifdef DEBUG
  printf("[100%] Matching is done!\n");

  match_stop = getTime();
  match_elapsed = match_stop - match_start;

  hipEventSynchronize(kernel_stop);
  kernel_elapsed = 0.0f;
  hipEventElapsedTime(&kernel_elapsed, kernel_start, kernel_stop);
  printf("[Info] Match function takes %f ms.\n", getInterval(match_start, match_stop));
  printf("[Info] Match kernel takes %f ms.\n", kernel_elapsed);
  printf("Hashed value size:\n");
  for(i = 0; i < queries.size();++i)
  {
	  printf("Query %d: %u.\n", i, h_value_idx[i]);
  }
  printf(">>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>\n");
#endif
}

void
GaLG::match(inv_table& table,
            query& queries,
            device_vector<data_t>& d_data,
            int& hash_table_size)
throw (int)
{
  vector<query> _q;
  _q.push_back(queries);
  match(table, _q, d_data, hash_table_size);
}
