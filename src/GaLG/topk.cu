#include "hip/hip_runtime.h"
#include "topk.h"
#include "GaLG/lib/bucket_topk/bucket_topk.h"

#include <thrust/host_vector.h>
#include <thrust/extrema.h>

struct ValueOfFloat
{
  float max;__host__ __device__ float
  valueOf(float data)
  {
    return (float) max - data;
  }
};

struct ValueOfInt
{
  float max;__host__ __device__ float
  valueOf(int data)
  {
    return (float) max - data;
  }
};

void
GaLG::topk(GaLG::inv_table& table, GaLG::query& queries,
    device_vector<int>& d_top_indexes)
{
  device_vector<float> d_a;
  device_vector<data_t> d_data;
  int hash_table_size;
  vector<query> q;
  q.push_back(queries);
  match(table, q, d_data, hash_table_size);
  topk(d_a, q, d_top_indexes);
}

void
GaLG::topk(GaLG::inv_table& table, vector<GaLG::query>& queries,
    device_vector<int>& d_top_indexes)
{
  device_vector<float> d_a;
  device_vector<data_t> d_data;
  int hash_table_size;
  match(table, queries, d_data, hash_table_size);
  topk(d_a, queries, d_top_indexes);
}

void
GaLG::topk(device_vector<int>& d_search, vector<GaLG::query>& queries,
    device_vector<int>& d_top_indexes)
{
  host_vector<int> h_tops(queries.size());
  int i;
  for (i = 0; i < queries.size(); i++)
    {
      h_tops[i] = queries[i].topk();
    }
  device_vector<int> d_tops(h_tops);
  topk(d_search, d_tops, d_top_indexes);
}

void
GaLG::topk(device_vector<float>& d_search, vector<GaLG::query>& queries,
    device_vector<int>& d_top_indexes)
{
  host_vector<int> h_tops(queries.size());
  int i;
  for (i = 0; i < queries.size(); i++)
    {
      h_tops[i] = queries[i].topk();
    }
  device_vector<int> d_tops(h_tops);
  topk(d_search, d_tops, d_top_indexes);
}

void
GaLG::topk(device_vector<int>& d_search, device_vector<int>& d_tops,
    device_vector<int>& d_top_indexes)
{
  int parts = d_tops.size();
  int total = 0, i, num;
  for (i = 0; i < parts; i++)
    {
      num = d_tops[i];
      total += num;
    }
  thrust::pair<device_vector<int>::iterator, device_vector<int>::iterator> minmax =
      thrust::minmax_element(d_search.begin(), d_search.end());
  host_vector<int> h_end_index(parts);
  device_vector<int> d_end_index(parts);
  int number_of_each = d_search.size() / parts;
  for (i = 0; i < parts; i++)
    {
      h_end_index[i] = (i + 1) * number_of_each;
    }
  d_end_index = h_end_index;
  d_top_indexes.clear(), d_top_indexes.resize(total);

  ValueOfInt val;
  val.max = *minmax.second;
  float min = *minmax.first;
  float max = *minmax.second;
  bucket_topk<int, ValueOfInt>(&d_search, val, min, max, &d_tops, &d_end_index,
      parts, &d_top_indexes);
}

void
GaLG::topk(device_vector<float>& d_search, device_vector<int>& d_tops,
    device_vector<int>& d_top_indexes)
{
  int parts = d_tops.size();
  int total = 0, i, num;
  for (i = 0; i < parts; i++)
    {
      num = d_tops[i];
      total += num;
    }
  thrust::pair<device_vector<float>::iterator, device_vector<float>::iterator> minmax =
      thrust::minmax_element(d_search.begin(), d_search.end());
  host_vector<int> h_end_index(parts);
  device_vector<int> d_end_index(parts);
  int number_of_each = d_search.size() / parts;
  for (i = 0; i < parts; i++)
    {
      h_end_index[i] = (i + 1) * number_of_each;
    }
  d_end_index = h_end_index;
  d_top_indexes.clear(), d_top_indexes.resize(total);

  ValueOfFloat val;
  val.max = *minmax.second;
  bucket_topk<float, ValueOfFloat>(&d_search, val, *minmax.first,
      *minmax.second, &d_tops, &d_end_index, parts, &d_top_indexes);
}
