#include <fstream>
#include <sstream>
#include <vector>
#include <mpi.h>
#include "rapidjson/document.h"

#include "parser.h"
#include "global.h"

const int LOCAL_RANK = atoi(getenv("OMPI_COMM_WORLD_LOCAL_RANK"));

using namespace genie::original;
using namespace rapidjson;
using namespace std;

/*
 * Checks whether all compulsory entries
 * are present in configuration file
 */
static bool ValidateConfiguration(const Document &json_config)
{
	vector<string> string_entries = {"data_file"};
	vector<string> int_entries = {
		"dim",
		"count_threshold",
		"data_type",
		"search_type",
		"max_data_size",
		"num_of_cluster",
		"data_format",
		"radius",
	};
	vector<string> float_entries = {"selectivity"};

	for (auto &&entry : string_entries) {
		if (!json_config.HasMember(entry.c_str())) {
			if (0 == g_mpi_rank)
				cout << "Entry " << entry.c_str() << " is missing" << endl;
			return false;
		}
		if (!json_config[entry.c_str()].IsString()) {
			if (0 == g_mpi_rank)
				cout << "Entry " << entry.c_str() << " has wrong type" << endl;
			return false;
		}	
	}

	for (auto &&entry : int_entries) {
		if (!json_config.HasMember(entry.c_str())) {
			if (0 == g_mpi_rank)
				cout << "Entry " << entry.c_str() << " is missing" << endl;
			return false;
		}
		if (!json_config[entry.c_str()].IsInt()) {
			if (0 == g_mpi_rank)
				cout << "Entry " << entry.c_str() << " has wrong type" << endl;
			return false;
		}	
	}

	for (auto &&entry : float_entries) {
		if (!json_config.HasMember(entry.c_str())) {
			if (0 == g_mpi_rank)
				cout << "Entry " << entry.c_str() << " is missing" << endl;
			return false;
		}
		if (!json_config[entry.c_str()].IsFloat()) {
			if (0 == g_mpi_rank)
				cout << "Entry " << entry.c_str() << " has wrong type" << endl;
			return false;
		}	
	}

	return true;
}

/*
 * Parse configuration file
 */
void distgenie::parser::ParseConfigurationFile(GPUGenie_Config &config, DistGenieConfig &extra_config, const string config_filename)
{
	/* read json configuration from file and parse it */
	ifstream config_file(config_filename);
	string config_file_content((istreambuf_iterator<char>(config_file)), istreambuf_iterator<char>());
	config_file.close();
	Document json_config;
	if (json_config.Parse(config_file_content.c_str()).HasParseError())
		MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);

	/* validate the configuration */
	if (!ValidateConfiguration(json_config))
	{
		if (0 == g_mpi_rank)
			clog << "Configuration file validation failed" << endl;
		MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
	}

	/* set configuration structs accordingly */
	extra_config.data_file = json_config["data_file"].GetString();
	extra_config.num_of_cluster = json_config["num_of_cluster"].GetInt();
	extra_config.data_format = json_config["data_format"].GetInt();

	config.dim = json_config["dim"].GetInt();
	config.count_threshold = json_config["count_threshold"].GetInt();
	config.query_radius = json_config["radius"].GetInt();
	config.use_device = LOCAL_RANK;
	config.selectivity = json_config["selectivity"].GetFloat();
	//config.selectivity = 0.0f;
	if (config.selectivity > 0.0f)
		config.use_adaptive_range = true;
	else
		config.use_adaptive_range = false;
	
	config.use_load_balance = false;
	config.posting_list_max_length = 6400;
	config.multiplier = 1.5f;
	config.use_multirange = false;
	config.save_to_gpu = true;
	
	config.data_type = json_config["data_type"].GetInt();
	config.search_type = json_config["search_type"].GetInt();
	config.max_data_size = json_config["max_data_size"].GetInt();
}

/*
 * Parse query into vector
 */
bool distgenie::parser::ValidateAndParseQuery(GPUGenie_Config &config, DistGenieConfig &extra_config, vector<Cluster> &clusters, const string query)
{
	Document json_query;
	if (json_query.Parse(query.c_str()).HasParseError()) {
		if (0 == g_mpi_rank)
			cout << "Received query is not a valid JSON document" << endl;
		return false;
	}

	/* validation */
	if (!json_query.HasMember("topk")) {
		if (0 == g_mpi_rank)
			cout << "Entry 'topk' is missing" << endl;
		return false;
	}
	if (!json_query["topk"].IsInt()) {
		if (0 == g_mpi_rank)
			cout << "Entry 'topk' should be an interger" << endl;
		return false;
	}
	if (!json_query.HasMember("queries")) {
		if (0 == g_mpi_rank)
			cout << "Entry 'queries' is missing" << endl;
		return false;
	}
	if (!json_query["queries"].IsArray()) {
		if (0 == g_mpi_rank)
			cout << "Entry 'queries' should be an array" << endl;
		return false;
	}
	else
	{
		for (auto &&single_query_json : json_query["queries"].GetArray())
		{
			if (!single_query_json.HasMember("content"))
			{
				if (0 == g_mpi_rank)
					cout << "Some query misses the 'content' section" << endl;
				return false;
			}
			if (!single_query_json["content"].IsArray())
			{
				if (0 == g_mpi_rank)
					cout << "Query's 'content' section should be an array" << endl;
				return false;
			}
			if (!single_query_json.HasMember("clusters"))
			{
				if (0 == g_mpi_rank)
					cout << "Some query misses the 'clusters' section" << endl;
				return false;
			}
			if (!single_query_json["clusters"].IsArray())
			{
				if (0 == g_mpi_rank)
					cout << "Query's 'clusters' section should be an array" << endl;
				return false;
			}
		}
	}

	int topk;
	topk = json_query["topk"].GetInt();
	extra_config.total_queries = json_query["queries"].Size();

	for (auto &&cluster : clusters)
	{
		cluster.m_queries.clear();
		cluster.m_queries_id.clear();
	}
	int id = 0;
	for (auto &&single_query_json : json_query["queries"].GetArray()) {
		vector<int> single_query_content;
		int cluster_id;
		for (auto &&query_value : single_query_json["content"].GetArray()) 
			single_query_content.emplace_back(query_value.GetInt());
		for (auto &&cluster : single_query_json["clusters"].GetArray()) 
		{
			cluster_id = cluster.GetInt();
			clusters.at(cluster_id).m_queries.emplace_back(single_query_content);
			clusters.at(cluster_id).m_queries_id.emplace_back(id);
		}
		++id;
	}

	config.num_of_topk = topk;
	config.hashtable_size = config.num_of_topk * 1.5 * config.count_threshold;

	return true;
}
