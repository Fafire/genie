#include "hip/hip_runtime.h"
#include "scan.h"

#include "DeviceCompositeCodec.h"
#include "DeviceBitPackingCodec.h"
#include "DeviceVarintCodec.h"

using namespace GPUGenie;

// Explicit template instances for working codecs

template void
GPUGenie::decodeArrayParallel<DeviceJustCopyCodec>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);

template void
GPUGenie::decodeArrayParallel<DeviceDeltaCodec>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);

template void
GPUGenie::decodeArrayParallel<DeviceBitPackingCodec>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);

template void
GPUGenie::decodeArrayParallel<DeviceBitPackingPrefixedCodec>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);

template void
GPUGenie::decodeArrayParallel<DeviceVarintCodec>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);

template void
GPUGenie::decodeArrayParallel<DeviceCompositeCodec<DeviceBitPackingCodec,DeviceJustCopyCodec>>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);

template class
GPUGenie::DeviceCompositeCodec<DeviceBitPackingCodec,DeviceJustCopyCodec>;

template void
GPUGenie::decodeArrayParallel<DeviceCompositeCodec<DeviceBitPackingCodec,DeviceVarintCodec>>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);

template class
GPUGenie::DeviceCompositeCodec<DeviceBitPackingCodec,DeviceVarintCodec>;

template <class CODEC> void
GPUGenie::decodeArrayParallel(
        int blocks,
        int threads,
        uint32_t *d_Input,
        size_t arrayLength,
        uint32_t *d_Output,
        size_t capacity,
        size_t *d_decomprLength)
{
    g_decodeArrayParallel<CODEC><<<blocks,threads>>>(d_Input, arrayLength, d_Output, capacity, d_decomprLength);
}

template <class CODEC> __global__ void
GPUGenie::g_decodeArrayParallel(uint32_t *d_Input, size_t arrayLength, uint32_t *d_Output, size_t capacity, size_t *d_decomprLength)
{
    CODEC codec;
    assert(blockDim.x == codec.decodeArrayParallel_lengthPerBlock() / codec.decodeArrayParallel_threadLoad());
    assert(gridDim.x <= codec.decodeArrayParallel_maxBlocks());
    assert(capacity <= gridDim.x * blockDim.x * codec.decodeArrayParallel_threadLoad());

    
    __shared__ uint32_t s_Input[GPUGENIE_SCAN_MAX_SHORT_ARRAY_SIZE];
    __shared__ uint32_t s_Output[GPUGENIE_SCAN_MAX_SHORT_ARRAY_SIZE];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < codec.decodeArrayParallel_lengthPerBlock(); i += codec.decodeArrayParallel_threadsPerBlock())
    {
        s_Input[idx + i] = (idx + i < (int)arrayLength) ? d_Input[idx + i] : 0;
        s_Output[idx + i] = 0;
    }

    __syncthreads();
    codec.decodeArrayParallel(s_Input, arrayLength, s_Output, capacity);
    __syncthreads();

    for (int i = 0; i < codec.decodeArrayParallel_lengthPerBlock(); i += codec.decodeArrayParallel_threadsPerBlock())
    {
        d_Output[idx + i] = s_Output[idx + i];
    }

    if (idx == 0 && d_decomprLength != NULL)
            (*d_decomprLength) = capacity;
}


void
GPUGenie::DeviceDeltaCodec::encodeArray(uint32_t *in, const size_t length, uint32_t *out, size_t &nvalue)
{
    std::memcpy(out, in, sizeof(uint32_t) * length);
    DeviceDeltaHelper<uint32_t>::delta(0, out, length);
    nvalue = length;
}

const uint32_t*
GPUGenie::DeviceDeltaCodec::decodeArray(const uint32_t *in, const size_t length, uint32_t *out, size_t &nvalue)
{
    std::memcpy(out, in, sizeof(uint32_t) * length);
    DeviceDeltaHelper<uint32_t>::inverseDelta(0, out, length);
    nvalue = length;
    return in + length;
}

__device__ const uint32_t*
GPUGenie::DeviceDeltaCodec::decodeArraySequential(const uint32_t *d_in, const size_t length, uint32_t *d_out, size_t &nvalue)
{
    if (length > nvalue){
        // We do not have enough capacity in the decompressed array!
        nvalue = length;
        return d_in;
    }
    for (int i = 0; i < length; i++)
        d_out[i] = d_in[i];
    DeviceDeltaHelper<uint32_t>::inverseDeltaOnGPU(0, d_out, length);
    nvalue = length;
    return d_in + length;
}

__device__ uint32_t*
GPUGenie::DeviceDeltaCodec::decodeArrayParallel(uint32_t *d_in, size_t length, uint32_t *d_out, size_t &nvalue)
{
    assert(length <= gridDim.x * blockDim.x * 4); // one thread can process 4 values
    assert(length <= nvalue); // not enough capacity in the decompressed array!
    assert(blockIdx.x == 0); // currently only support single block

    assert(length > 0 && length <= GPUGENIE_SCAN_MAX_SHORT_ARRAY_SIZE);
    uint pow2arrayLength = GPUGenie::d_pow2ceil_32(length);
    uint arrayLength = (length + 3) / 4;

    // Check supported size range
    // Check parallel model compatibility
    assert(blockDim.x == GPUGENIE_SCAN_THREADBLOCK_SIZE && gridDim.x == 1);

    __syncthreads();
    GPUGenie::d_scanInclusiveShared((uint4 *)d_out, (uint4 *)d_in, arrayLength, pow2arrayLength);
    __syncthreads();
    
    nvalue = length;
    return d_in + length;
}

template <class Codec1, class Codec2> void
GPUGenie::DeviceCompositeCodec<Codec1,Codec2>::encodeArray(uint32_t *in, const size_t length, uint32_t *out, size_t &nvalue)
{
    assert(length > 0);
    assert(nvalue > 0);
    int codec1minEffLength = codec1.decodeArrayParallel_minEffectiveLength();
    size_t codec1Length = (length / codec1minEffLength) * codec1minEffLength;
    size_t codec2Length = length - codec1Length;
    assert (codec1Length + codec2Length == length);
    assert (codec2Length <= length);

    size_t nvalue1 = 0;
    if (codec1Length){
        nvalue1 = nvalue;
        codec1.encodeArray(in, codec1Length, out + 1, nvalue1);
        assert(nvalue >= nvalue1); // Error - compression overflow
    }

    size_t nvalue2 = 0;
    if (codec2Length) {
        nvalue2 = nvalue - nvalue1;
        codec2.encodeArray(in + codec1Length, codec2Length, out + 1 + nvalue1, nvalue2);
        assert(nvalue - nvalue1 >= nvalue2); // Error - compression overflow
    }

    out[0] = nvalue1; // store infromation about compressed length from the first codec

    nvalue = 1 + nvalue1 + nvalue2;
}

template <class Codec1, class Codec2> const uint32_t*
GPUGenie::DeviceCompositeCodec<Codec1,Codec2>::decodeArray(const uint32_t *in, const size_t comprLength, uint32_t *out, size_t &nvalue)
{
    size_t firstCodecComprLength = *in++;

    // Codec1 decompresses as much as it can
    size_t nvalue1 = 0;
    const uint32_t *inForCodec2 = in;

    if (firstCodecComprLength){
        nvalue1 = nvalue; // set capacity for codec1 to overall capacity
        inForCodec2 = codec1.decodeArray(in, firstCodecComprLength, out, nvalue1);

        if (nvalue1 > nvalue){ // Error - Codec1 does not have enough capacity
            nvalue = nvalue1; // Set nvalue to required capacity of codec1
            return in; // Return pointer to the deginning of the compressed array
        }

        if (inForCodec2 == in + comprLength - 1){ // Codec1 decompressed everything
            nvalue = nvalue1;
            return inForCodec2;
        }
    }

    assert(inForCodec2 == in + firstCodecComprLength); // Make sure codec1 returned correct d_in pointer

    // Codec2 decompresses the leftover
    size_t nvalue2 = nvalue - nvalue1; // remaining capacity
    size_t leftoverLength = comprLength - 1 - (inForCodec2 - in);
    const uint32_t *inAfterBothCodecs = codec2.decodeArray(inForCodec2, leftoverLength, out + nvalue1, nvalue2);

    if (nvalue2 > nvalue - nvalue1){ // Error - Codec2 does not have enough capacity
        nvalue = nvalue1 + nvalue2; // Set nvalue to required capacity of codec1 + codec2
        return in; // Return pointer to the deginning of the compressed array
    }

    assert(in + comprLength - 1 == inAfterBothCodecs);
    nvalue = nvalue1 + nvalue2;
    return inAfterBothCodecs;
}


template <class Codec1, class Codec2> __device__ uint32_t*
GPUGenie::DeviceCompositeCodec<Codec1,Codec2>::decodeArraySequential(uint32_t *d_in, size_t length, uint32_t *d_out, size_t &nvalue)
{
    return nullptr;
}



template <class Codec1, class Codec2> __device__ uint32_t*
GPUGenie::DeviceCompositeCodec<Codec1,Codec2>::decodeArrayParallel(
            uint32_t *d_in, size_t comprLength, uint32_t *d_out, size_t &nvalue)
{
    size_t firstCodecComprLength = *d_in++;

    // Codec1 decompresses as much as it can
    size_t nvalue1 = 0;
    uint32_t *d_inForCodec2 = d_in;

    if (firstCodecComprLength){
        nvalue1 = nvalue; // set capacity for codec1 to overall capacity

        d_inForCodec2 = codec1.decodeArrayParallel(d_in, firstCodecComprLength, d_out, nvalue1);
        __syncthreads();

        if (nvalue1 > nvalue){ // Error - Codec1 does not have enough capacity
            nvalue = nvalue1; // Set nvalue to required capacity of codec1
            return d_in; // Return pointer to the deginning of the compressed array
        }

        if (d_inForCodec2 == d_in + comprLength - 1){ // Codec1 decompressed everything
            nvalue = nvalue1;
            return d_inForCodec2;
        }
    }

    assert(d_inForCodec2 == d_in + firstCodecComprLength); // Make sure codec1 returned correct d_in pointer

    // Codec2 decompresses the leftover
    size_t nvalue2 = nvalue - nvalue1; // remaining capacity
    size_t leftoverLength = comprLength - 1 - firstCodecComprLength;
    uint32_t *d_inAfterBothCodecs = codec2.decodeArrayParallel(d_inForCodec2, leftoverLength, d_out + nvalue1, nvalue2);
    __syncthreads();

    if (nvalue2 > nvalue - nvalue1){ // Error - Codec2 does not have enough capacity
        nvalue = nvalue1 + nvalue2; // Set nvalue to required capacity of codec1 + codec2
        return d_in; // Return pointer to the deginning of the compressed array
    }

    assert(d_in + comprLength - 1 == d_inAfterBothCodecs);
    nvalue = nvalue1 + nvalue2;
    return d_inAfterBothCodecs;
}

