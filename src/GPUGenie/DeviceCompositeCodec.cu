#include "hip/hip_runtime.h"
#include "scan.h"

#include "DeviceCompositeCodec.h"
#include "DeviceBitPackingCodec.h"

using namespace GPUGenie;

// Explicit template instances for working codecs

template void
GPUGenie::decodeArrayParallel<DeviceJustCopyCodec>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);

template void
GPUGenie::decodeArrayParallel<DeviceDeltaCodec>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);

template void
GPUGenie::decodeArrayParallel<DeviceBitPackingCodec>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);

template void
GPUGenie::decodeArrayParallel<DeviceBitPackingPrefixedCodec>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);


template <class CODEC> void
GPUGenie::decodeArrayParallel(
        int blocks,
        int threads,
        uint32_t *d_Input,
        size_t arrayLength,
        uint32_t *d_Output,
        size_t capacity,
        size_t *d_decomprLength)
{
    g_decodeArrayParallel<CODEC><<<blocks,threads>>>(d_Input, arrayLength, d_Output, capacity, d_decomprLength);
}

template <class CODEC> __global__ void
GPUGenie::g_decodeArrayParallel(uint32_t *d_Input, size_t arrayLength, uint32_t *d_Output, size_t capacity, size_t *d_decomprLength)
{
    CODEC codec;
    assert(blockDim.x == codec.decodeArrayParallel_lengthPerBlock() / codec.decodeArrayParallel_threadLoad());
    assert(gridDim.x <= codec.decodeArrayParallel_maxBlocks());
    assert(capacity <= gridDim.x * blockDim.x * codec.decodeArrayParallel_threadLoad());

    
    __shared__ uint32_t s_Input[GPUGENIE_SCAN_MAX_SHORT_ARRAY_SIZE];
    __shared__ uint32_t s_Output[GPUGENIE_SCAN_MAX_SHORT_ARRAY_SIZE];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    printf("d_Input[%d] = 0x%08X \n", idx, d_Input[idx]);

    s_Input[idx] = (idx < (int)arrayLength) ? d_Input[idx] : 0;
    s_Output[idx] = 0;

    printf("s_Input[%d] = 0x%08X \n", idx, s_Input[idx]);

    __syncthreads();
    codec.decodeArrayParallel(s_Input, arrayLength, s_Output, capacity);
    __syncthreads();

    printf("s_Output[%d] = 0x%08X \n", idx, s_Output[idx]);

    if (idx < (int)capacity)
        d_Output[idx] = s_Output[idx];

    if (idx == 0 && d_decomprLength != NULL)
            (*d_decomprLength) = capacity;

    printf("d_Output[%d] = 0x%08X \n", idx, s_Output[idx]);
}


void
GPUGenie::DeviceDeltaCodec::encodeArray(uint32_t *in, const size_t length, uint32_t *out, size_t &nvalue)
{
    std::memcpy(out, in, sizeof(uint32_t) * length);
    DeviceDeltaHelper<uint32_t>::delta(0, out, length);
    nvalue = length;
}

const uint32_t*
GPUGenie::DeviceDeltaCodec::decodeArray(const uint32_t *in, const size_t length, uint32_t *out, size_t &nvalue)
{
    std::memcpy(out, in, sizeof(uint32_t) * length);
    DeviceDeltaHelper<uint32_t>::inverseDelta(0, out, length);
    nvalue = length;
    return in + length;
}

__device__ const uint32_t*
GPUGenie::DeviceDeltaCodec::decodeArraySequential(const uint32_t *d_in, const size_t length, uint32_t *d_out, size_t &nvalue)
{
    if (length > nvalue){
        // We do not have enough capacity in the decompressed array!
        nvalue = length;
        return d_in;
    }
    for (int i = 0; i < length; i++)
        d_out[i] = d_in[i];
    DeviceDeltaHelper<uint32_t>::inverseDeltaOnGPU(0, d_out, length);
    nvalue = length;
    return d_in + length;
}

__device__ uint32_t*
GPUGenie::DeviceDeltaCodec::decodeArrayParallel(uint32_t *d_in, size_t length, uint32_t *d_out, size_t &nvalue)
{
    assert(length <= gridDim.x * blockDim.x * 4); // one thread can process 4 values
    assert(length <= nvalue); // not enough capacity in the decompressed array!
    assert(blockIdx.x == 0); // currently only support single block

    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[idx] = 0; // d_out should be shared memory

    assert(length > 0 && length <= GPUGENIE_SCAN_MAX_SHORT_ARRAY_SIZE);
    uint pow2arrayLength = GPUGenie::d_pow2ceil_32(length);
    uint arrayLength = (length + 3) / 4;


    // Check supported size range
    // Check parallel model compatibility
    assert(blockDim.x == GPUGENIE_SCAN_THREADBLOCK_SIZE && gridDim.x == 1);

    __syncthreads();
    GPUGenie::d_scanExclusiveShared((uint4 *)d_out, (uint4 *)d_in, arrayLength, pow2arrayLength);
    __syncthreads();
    
    if (idx == 0)
        assert(d_out[idx] == 0);
    else if (idx < arrayLength)
        assert(d_out[idx] >= d_out[idx-1]);

    // turn it into inclusive scan
    uint32_t inc = 0;
    if (idx < length)
        inc = d_out[idx+1];
    __syncthreads();
    if (idx < length)
        d_out[idx] = inc;

    nvalue = length;
    return d_in + length;
}

