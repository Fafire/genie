#include "hip/hip_runtime.h"
#include "inv_compr_table.h"

void
GPUGenie::inv_compr_table::build(u64 max_length, bool use_load_balance)
{
    inv_table::build(max_length, use_load_balance);


    //  u64 table_start = getTime();
    // _ck.clear();
    // _inv.clear();
    // _inv_index.clear();
    // _inv_pos.clear();
    // if(!use_load_balance)
    // {
    //     max_length = (u64)0 - (u64)1;
    // }
    // unsigned int last;
    // int key, dim, value;
    // for (unsigned int i = 0; i < _inv_lists.size(); i++)
    // {
    //     dim = i << _shifter;
    //     for (value = _inv_lists[i].min(); value <= _inv_lists[i].max(); value++)
    //     {
    //         key = dim + value - _inv_lists[i].min();
            
    //         vector<int>* _index;
            
    //         _index = _inv_lists[i].index(value);
    
    //         vector<int> index;
    //         index.clear();
    //         if(_index != NULL)
    //             index = *_index;
    //         if(_inv_lists.size() <= 1)//used int subsequence search
    //             shift_bits_subsequence = _inv_lists[i]._shift_bits_subsequence();

    //         if (_ck.size() <= (unsigned int) key)
    //         {
    //             last = _ck.size();
    //             _ck.resize(key + 1);
    //             _inv_index.resize(key + 1);
    //             for (; last < _ck.size(); ++last)
    //             {
    //                 _ck[last] = _inv.size();
    //                 _inv_index[last] = _inv_pos.size();
    //             }
    //         }
    //         for (unsigned int j = 0; j < index.size(); ++j)
    //         {
    //             if (j % max_length == 0)
    //             {
    //                 _inv_pos.push_back(_inv.size());
    //             }
    //             _inv.push_back(index[j]);
    //             _ck[key] = _inv.size();
    //         }

    //     }

    // }
    // _inv_index.push_back(_inv_pos.size());
    // _inv_pos.push_back(_inv.size());

    // _build_status = builded;
    //     u64 table_end = getTime();
    // cout<<"build table time = "<<getInterval(table_start, table_end)<<"ms."<<endl;
    //     //Logger::log(Logger::DEBUG, "inv_index size %d:", _inv_index.size());
    // //Logger::log(Logger::DEBUG, "inv_pos size %d:", _inv_pos.size());
    // //Logger::log(Logger::DEBUG, "inv size %d:", _inv.size());
    //     Logger::log(Logger::INFO, "inv_index size %d:", _inv_index.size());
    // Logger::log(Logger::INFO, "inv_pos size %d:", _inv_pos.size());
    // Logger::log(Logger::INFO, "inv size %d:", _inv.size());

}

void GPUGenie::inv_compr_table::~inv_compr_table() {}


const std::string&
GPUGenie::inv_compr_table::getCompression() const
{
    return m_compression;
}

void
GPUGenie::inv_compr_table::setCompression(const std::string &compression)
{
    if (_build_status == builded)
    {
        Logger::log(Logger::ALERT, "ERROR: Attempting to change compression type on already built table!");
        return;
    }
    m_compression = compression;
}

std::vector<GPUGenie::inv_compr_list>*
GPUGenie::inv_compr_table::compressedInvLists()
{
    return &m_comprInvLists;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInv()
{
    return m_comprInv;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInvPos()
{
    return m_comprInvPos;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInvIndex()
{
    return _inv_index;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedCK()
{
    return _ck;
}


