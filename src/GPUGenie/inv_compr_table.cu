#include "hip/hip_runtime.h"
#include <iostream>

#include "Logger.h"
#include "genie_errors.h"
#include "Timing.h"

#include "inv_compr_table.h"


void
GPUGenie::inv_compr_table::build(u64 max_length, bool use_load_balance)
{
    Logger::log(Logger::DEBUG, "Bulding uncompressed inv_table...");
    inv_table::build(max_length, use_load_balance);


    Logger::log(Logger::DEBUG, "Bulding compressed inv_table...");

    std::vector<int> &inv = *(inv_table::inv());
    std::vector<int> &invPos = *(inv_table::inv_pos());
    std::vector<uint32_t> &compressedInv = m_comprInv;
    std::vector<int> &compressedInvPos = m_comprInvPos;

    uint64_t compressionStartTime = getTime();

    shared_ptr<DeviceIntegerCODEC> codec;
    switch (this->m_compression){
        case "copy":
            codec = std::shared_ptr<IntegerCODEC>(
                        new DeviceJustCopy());
            break;
        case "d1":
            codec = std::shared_ptr<IntegerCODEC>(
                        new DeviceDeltaCodec());
            break;
        case "d1-bp32":
            codec = std::shared_ptr<IntegerCODEC>(
                        new DeviceCompositeCodec<DeviceBinaryPacking,DeviceJustCopy>());
            break;
        case "d1-varint":
        case "d1-bp32-varint":
        default:
            Logger::log(Logger::ALERT, "Unsupported inverted table compression %s", this->m_compression);
    }

    // make uint32_t copy of uncompressed inv array
    std::vector<uint32_t> inv32(inv.begin(), inv.end());

    compressedInv.resize(inv.size());
    compressedInvPos.resize(invPos.size());
    compressedInvPos.push_back(0);

    
    int compressedInvCurrentPos = 0;
    int compressedInvSize = 0;
    int compressedInvCapacity = compressedInv.size();

    uint32_t *out = compressedInv.data();
    for (int pos = 0; pos < invPos.size(); pos++)
    {
        int invStart = invPos[pos];
        int invEnd = invPos[pos+1];
        size_t invLength = invEnd - invStart;
        assert(invEnd > invStart);

        uint32_t *in = inv32.data() + sizeof(uint32_t) * invStart; // compression input
        size_t nvalue = compressedInvCapacity; // nvalue is the compressed size

        codec->encodeArray(in, invLength, out, nvalue);

        out += sizeof(int) * nvalue; // shift compression output pointer
        compressedInvCapacity -= nvalue;
        compressedInvSize += nvalue;

        compressedInvPos.push_back(compressedInvSize);
    }

    compressedInv.resize(compressedInvSize); // shrink to used space only
    assert(compressedInvSize == compressedInvPos.back());

    uint64_t compressionEndTime = getTime();

    Logger::log(Logger::DEBUG, "Done bulding compressed inv_compr_table in time %f",
        getInterval(compressionStartTime, compressionEndTime));

    Logger::log(Logger::INFO, "Compression %s, codec: %s, compression ratio: %f", m_compression.c_str(),
        codec->name(), 32.0 * static_cast<double>(compressedInv.size()) / static_cast<double>(inv.size()));
}


void GPUGenie::inv_compr_table::bp32(uint32_t *in, const size_t length, uint32_t *out, size_t &nvalue) {
    checkifdivisibleby(length, BlockSize);
    const uint32_t *const initout(out);
    *out++ = static_cast<uint32_t>(length);
    uint32_t Bs[HowManyMiniBlocks];
    uint32_t init = 0;
    const uint32_t *const final = in + length;
    for (; in + HowManyMiniBlocks * MiniBlockSize <= final;
         in += HowManyMiniBlocks * MiniBlockSize) {
      uint32_t tmpinit = init;
      for (uint32_t i = 0; i < HowManyMiniBlocks; ++i) {
        Bs[i] = BlockPacker::maxbits(in + i * MiniBlockSize, tmpinit);
      }
      *out++ = (Bs[0] << 24) | (Bs[1] << 16) | (Bs[2] << 8) | Bs[3];
      for (uint32_t i = 0; i < HowManyMiniBlocks; ++i) {
        BlockPacker::packblockwithoutmask(in + i * MiniBlockSize, out, Bs[i],
                                          init);
        out += Bs[i];
      }
    }
    if (in < final) {
      size_t howmany = (final - in) / MiniBlockSize;
      uint32_t tmpinit = init;
      memset(&Bs[0], 0, HowManyMiniBlocks * sizeof(uint32_t));
      for (uint32_t i = 0; i < howmany; ++i) {
        Bs[i] = BlockPacker::maxbits(in + i * MiniBlockSize, tmpinit);
      }
      *out++ = (Bs[0] << 24) | (Bs[1] << 16) | (Bs[2] << 8) | Bs[3];
      for (uint32_t i = 0; i < howmany; ++i) {
        BlockPacker::packblockwithoutmask(in + i * MiniBlockSize, out, Bs[i],
                                          init);
        out += Bs[i];
      }
    }
    nvalue = out - initout;
  }

GPUGenie::inv_compr_table::~inv_compr_table()
{
    clear_gpu_mem();
}


const std::string&
GPUGenie::inv_compr_table::getCompression() const
{
    return m_compression;
}

void
GPUGenie::inv_compr_table::setCompression(const std::string &compression)
{
    if (this->build_status() == builded)
    {
        Logger::log(Logger::ALERT, "ERROR: Attempting to change compression type on already built table!");
        return;
    }
    m_compression = compression;
}

size_t
GPUGenie::inv_compr_table::getUncompressedPostingListMaxLength() const
{
    return m_uncompressedInvListsMaxLength;
}

void
GPUGenie::inv_compr_table::setUncompressedPostingListMaxLength(size_t length)
{
    this->m_uncompressedInvListsMaxLength = length;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInv()
{
    return &m_comprInv;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInvPos()
{
    return &m_comprInvPos;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInvIndex()
{
    return this->inv_index();
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedCK()
{
    return this->ck();
}

int*
GPUGenie::inv_compr_table::deviceCompressedInv() const
{
    return m_d_compr_inv_p;
}

bool GPUGenie::inv_compr_table::cpy_data_to_gpu()
{
    try{
        if(m_d_compr_inv_p == NULL)
            cudaCheckErrors(hipMalloc(&m_d_compr_inv_p, sizeof(int) * m_comprInv.size()));
        u64 t = getTime();
        cudaCheckErrors(hipMemcpy(m_d_compr_inv_p, &m_comprInv[0], sizeof(int) * m_comprInv.size(),
                hipMemcpyHostToDevice));
        u64 tt = getTime();
        std::cout<<"The compressed inverted list(all data) transfer time = " << getInterval(t,tt) << "ms" <<std::endl;

    } catch(std::bad_alloc &e){
        throw(GPUGenie::gpu_bad_alloc(e.what()));
    }

    return true;
}

void GPUGenie::inv_compr_table::clear()
{
    inv_table::clear();

    ck()->clear();
    m_comprInv.clear();
    m_comprInvPos.clear();
}

void GPUGenie::inv_compr_table::clear_gpu_mem()
{
    if (m_d_compr_inv_p == NULL)
        return;

    std::cout << "cudaFreeTime: " ;
    u64 t1 = getTime();
    cudaCheckErrors(hipFree(m_d_compr_inv_p));
    u64 t2 = getTime();
    std::cout << getInterval(t1, t2) << " ms."<< std::endl;

}

