#include "hip/hip_runtime.h"
#include <iostream>

#include "Logger.h"
#include "genie_errors.h"
#include "Timing.h"

#include "inv_compr_table.h"


void
GPUGenie::inv_compr_table::build(u64 max_length, bool use_load_balance)
{
    Logger::log(Logger::DEBUG, "Bulding uncompressed inv_table...");
    inv_table::build(max_length, use_load_balance);


    Logger::log(Logger::DEBUG, "Bulding compressed inv_table...");

    std::vector<int> &inv = *(inv_table::inv());
    std::vector<int> &invPos = *(inv_table::inv_pos());
    std::vector<int> &compressedInv = m_comprInv;
    std::vector<int> &compressedInvPos = m_comprInvPos;

    compressedInv.clear();
    compressedInvPos.clear();

    uint64_t compressionStart = getTime();

    switch (this->m_compression){
        case "copy":
            copy(inv,invPod,compressedInv,compressedInvPos);
            break;
        case "d1":
            delta1Encode(inv,invPod,compressedInv,compressedInvPos);
            break;
        case "d1-bp128":
            delta1Encode(inv,invPod,compressedInv,compressedInvPos);
            bp128Encode(inv,invPod,compressedInv,compressedInvPos);
            break;
        default:
            Logger::log(Logger::ALERT, "Unsupported inverted table compression %s", this->m_compression);
    }

    uint64_t compressionEnd = getTime();

    //  u64 table_start = getTime();
    // _ck.clear();
    // _inv.clear();
    // _inv_index.clear();
    // _inv_pos.clear();
    // if(!use_load_balance)
    // {
    //     max_length = (u64)0 - (u64)1;
    // }
    // unsigned int last;
    // int key, dim, value;
    // for (unsigned int i = 0; i < _inv_lists.size(); i++)
    // {
    //     dim = i << _shifter;
    //     for (value = _inv_lists[i].min(); value <= _inv_lists[i].max(); value++)
    //     {
    //         key = dim + value - _inv_lists[i].min();
            
    //         vector<int>* _index;
            
    //         _index = _inv_lists[i].index(value);
    
    //         vector<int> index;
    //         index.clear();
    //         if(_index != NULL)
    //             index = *_index;
    //         if(_inv_lists.size() <= 1)//used int subsequence search
    //             shift_bits_subsequence = _inv_lists[i]._shift_bits_subsequence();

    //         if (_ck.size() <= (unsigned int) key)
    //         {
    //             last = _ck.size();
    //             _ck.resize(key + 1);
    //             _inv_index.resize(key + 1);
    //             for (; last < _ck.size(); ++last)
    //             {
    //                 _ck[last] = _inv.size();
    //                 _inv_index[last] = _inv_pos.size();
    //             }
    //         }
    //         for (unsigned int j = 0; j < index.size(); ++j)
    //         {
    //             if (j % max_length == 0)
    //             {
    //                 _inv_pos.push_back(_inv.size());
    //             }
    //             _inv.push_back(index[j]);
    //             _ck[key] = _inv.size();
    //         }

    //     }

    // }
    // _inv_index.push_back(_inv_pos.size());
    // _inv_pos.push_back(_inv.size());

    // _build_status = builded;
    //     u64 table_end = getTime();
    // std::cout<<"build table time = "<<getInterval(table_start, table_end)<<"ms."<<std::endl;
    //     //Logger::log(Logger::DEBUG, "inv_index size %d:", _inv_index.size());
    // //Logger::log(Logger::DEBUG, "inv_pos size %d:", _inv_pos.size());
    // //Logger::log(Logger::DEBUG, "inv size %d:", _inv.size());
    //     Logger::log(Logger::INFO, "inv_index size %d:", _inv_index.size());
    // Logger::log(Logger::INFO, "inv_pos size %d:", _inv_pos.size());
    // Logger::log(Logger::INFO, "inv size %d:", _inv.size());

}


void bp32(uint32_t *in, const size_t length, uint32_t *out,
                   size_t &nvalue) {
    checkifdivisibleby(length, BlockSize);
    const uint32_t *const initout(out);
    *out++ = static_cast<uint32_t>(length);
    uint32_t Bs[HowManyMiniBlocks];
    uint32_t init = 0;
    const uint32_t *const final = in + length;
    for (; in + HowManyMiniBlocks * MiniBlockSize <= final;
         in += HowManyMiniBlocks * MiniBlockSize) {
      uint32_t tmpinit = init;
      for (uint32_t i = 0; i < HowManyMiniBlocks; ++i) {
        Bs[i] = BlockPacker::maxbits(in + i * MiniBlockSize, tmpinit);
      }
      *out++ = (Bs[0] << 24) | (Bs[1] << 16) | (Bs[2] << 8) | Bs[3];
      for (uint32_t i = 0; i < HowManyMiniBlocks; ++i) {
        BlockPacker::packblockwithoutmask(in + i * MiniBlockSize, out, Bs[i],
                                          init);
        out += Bs[i];
      }
    }
    if (in < final) {
      size_t howmany = (final - in) / MiniBlockSize;
      uint32_t tmpinit = init;
      memset(&Bs[0], 0, HowManyMiniBlocks * sizeof(uint32_t));
      for (uint32_t i = 0; i < howmany; ++i) {
        Bs[i] = BlockPacker::maxbits(in + i * MiniBlockSize, tmpinit);
      }
      *out++ = (Bs[0] << 24) | (Bs[1] << 16) | (Bs[2] << 8) | Bs[3];
      for (uint32_t i = 0; i < howmany; ++i) {
        BlockPacker::packblockwithoutmask(in + i * MiniBlockSize, out, Bs[i],
                                          init);
        out += Bs[i];
      }
    }
    nvalue = out - initout;
  }

GPUGenie::inv_compr_table::~inv_compr_table()
{
    clear_gpu_mem();
}


const std::string&
GPUGenie::inv_compr_table::getCompression() const
{
    return m_compression;
}

void
GPUGenie::inv_compr_table::setCompression(const std::string &compression)
{
    if (this->build_status() == builded)
    {
        Logger::log(Logger::ALERT, "ERROR: Attempting to change compression type on already built table!");
        return;
    }
    m_compression = compression;
}

size_t
GPUGenie::inv_compr_table::getUncompressedPostingListMaxLength() const
{
    return m_uncompressedInvListsMaxLength;
}

void
GPUGenie::inv_compr_table::setUncompressedPostingListMaxLength(size_t length)
{
    this->m_uncompressedInvListsMaxLength = length;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInv()
{
    return &m_comprInv;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInvPos()
{
    return &m_comprInvPos;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInvIndex()
{
    return this->inv_index();
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedCK()
{
    return this->ck();
}

int*
GPUGenie::inv_compr_table::deviceCompressedInv() const
{
    return m_d_compr_inv_p;
}

bool GPUGenie::inv_compr_table::cpy_data_to_gpu()
{
    try{
        if(m_d_compr_inv_p == NULL)
            cudaCheckErrors(hipMalloc(&m_d_compr_inv_p, sizeof(int) * m_comprInv.size()));
        u64 t = getTime();
        cudaCheckErrors(hipMemcpy(m_d_compr_inv_p, &m_comprInv[0], sizeof(int) * m_comprInv.size(),
                hipMemcpyHostToDevice));
        u64 tt = getTime();
        std::cout<<"The compressed inverted list(all data) transfer time = " << getInterval(t,tt) << "ms" <<std::endl;

    } catch(std::bad_alloc &e){
        throw(GPUGenie::gpu_bad_alloc(e.what()));
    }

    return true;
}

void GPUGenie::inv_compr_table::clear()
{
    inv_table::clear();

    ck()->clear();
    m_comprInv.clear();
    m_comprInvPos.clear();
}

void GPUGenie::inv_compr_table::clear_gpu_mem()
{
    if (m_d_compr_inv_p == NULL)
        return;

    std::cout << "cudaFreeTime: " ;
    u64 t1 = getTime();
    cudaCheckErrors(hipFree(m_d_compr_inv_p));
    u64 t2 = getTime();
    std::cout << getInterval(t1, t2) << " ms."<< std::endl;

}

