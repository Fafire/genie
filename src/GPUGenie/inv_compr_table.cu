#include "hip/hip_runtime.h"
#include <iostream>

#include "Logger.h"
#include "genie_errors.h"
#include "Timing.h"

#include "inv_compr_table.h"


void
GPUGenie::inv_compr_table::build(u64 max_length, bool use_load_balance)
{
    Logger::log(Logger::DEBUG, "Bulding uncompressed inv_table...");
    inv_table::build(max_length, use_load_balance);


    Logger::log(Logger::DEBUG, "Bulding compressed inv_table...");

    std::vector<int> &inv = *(inv_table::inv());
    std::vector<int> &invPos = *(inv_table::inv_pos());
    std::vector<uint32_t> &compressedInv = m_comprInv;
    std::vector<int> &compressedInvPos = m_comprInvPos;

    uint64_t compressionStartTime = getTime();

    shared_ptr<DeviceIntegerCODEC> codec;
    switch (this->m_compression){
        case "copy":
            codec = std::shared_ptr<IntegerCODEC>(
                        new DeviceJustCopyCodec());
            break;
        case "d1":
            codec = std::shared_ptr<IntegerCODEC>(
                        new DeviceDeltaCodec());
            break;
        case "d1-bp32":
            codec = std::shared_ptr<IntegerCODEC>(
                        new DeviceCompositeCodec<DeviceBitPackingCODEC,DeviceJustCopyCodec>());
            break;
        case "d1-varint":
        case "d1-bp32-varint":
        default:
            Logger::log(Logger::ALERT, "Unsupported inverted table compression %s", this->m_compression);
    }

    // make uint32_t copy of uncompressed inv array
    std::vector<uint32_t> inv32(inv.begin(), inv.end());

    compressedInv.resize(inv.size());
    compressedInvPos.resize(invPos.size());
    compressedInvPos.push_back(0);

    
    int compressedInvCurrentPos = 0;
    int compressedInvSize = 0;
    int compressedInvCapacity = compressedInv.size();

    uint32_t *out = compressedInv.data();
    for (int pos = 0; pos < invPos.size(); pos++)
    {
        int invStart = invPos[pos];
        int invEnd = invPos[pos+1];
        size_t invLength = invEnd - invStart;
        assert(invEnd > invStart);

        uint32_t *in = inv32.data() + sizeof(uint32_t) * invStart; // compression input
        size_t nvalue = compressedInvCapacity; // nvalue is the compressed size

        codec->encodeArray(in, invLength, out, nvalue);

        out += sizeof(int) * nvalue; // shift compression output pointer
        compressedInvCapacity -= nvalue;
        compressedInvSize += nvalue;

        compressedInvPos.push_back(compressedInvSize);
    }

    compressedInv.resize(compressedInvSize); // shrink to used space only
    assert(compressedInvSize == compressedInvPos.back());

    uint64_t compressionEndTime = getTime();

    Logger::log(Logger::DEBUG, "Done bulding compressed inv_compr_table in time %f",
        getInterval(compressionStartTime, compressionEndTime));

    Logger::log(Logger::INFO, "Compression %s, codec: %s, compression ratio: %f", m_compression.c_str(),
        codec->name(), 32.0 * static_cast<double>(compressedInv.size()) / static_cast<double>(inv.size()));
}


GPUGenie::inv_compr_table::~inv_compr_table()
{
    clear_gpu_mem();
}


const std::string&
GPUGenie::inv_compr_table::getCompression() const
{
    return m_compression;
}

void
GPUGenie::inv_compr_table::setCompression(const std::string &compression)
{
    if (this->build_status() == builded)
    {
        Logger::log(Logger::ALERT, "ERROR: Attempting to change compression type on already built table!");
        return;
    }
    m_compression = compression;
}

size_t
GPUGenie::inv_compr_table::getUncompressedPostingListMaxLength() const
{
    return m_uncompressedInvListsMaxLength;
}

void
GPUGenie::inv_compr_table::setUncompressedPostingListMaxLength(size_t length)
{
    this->m_uncompressedInvListsMaxLength = length;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInv()
{
    return &m_comprInv;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInvPos()
{
    return &m_comprInvPos;
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedInvIndex()
{
    return this->inv_index();
}

std::vector<int>*
GPUGenie::inv_compr_table::compressedCK()
{
    return this->ck();
}

int*
GPUGenie::inv_compr_table::deviceCompressedInv() const
{
    return m_d_compr_inv_p;
}

bool GPUGenie::inv_compr_table::cpy_data_to_gpu()
{
    try{
        if(m_d_compr_inv_p == NULL)
            cudaCheckErrors(hipMalloc(&m_d_compr_inv_p, sizeof(int) * m_comprInv.size()));
        u64 t = getTime();
        cudaCheckErrors(hipMemcpy(m_d_compr_inv_p, &m_comprInv[0], sizeof(int) * m_comprInv.size(),
                hipMemcpyHostToDevice));
        u64 tt = getTime();
        std::cout<<"The compressed inverted list(all data) transfer time = " << getInterval(t,tt) << "ms" <<std::endl;

    } catch(std::bad_alloc &e){
        throw(GPUGenie::gpu_bad_alloc(e.what()));
    }

    return true;
}

void GPUGenie::inv_compr_table::clear()
{
    inv_table::clear();

    ck()->clear();
    m_comprInv.clear();
    m_comprInvPos.clear();
}

void GPUGenie::inv_compr_table::clear_gpu_mem()
{
    if (m_d_compr_inv_p == NULL)
        return;

    std::cout << "cudaFreeTime: " ;
    u64 t1 = getTime();
    cudaCheckErrors(hipFree(m_d_compr_inv_p));
    u64 t2 = getTime();
    std::cout << getInterval(t1, t2) << " ms."<< std::endl;

}

