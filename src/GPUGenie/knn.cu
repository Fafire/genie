#include "hip/hip_runtime.h"
#include "knn.h"
#include <math.h>
#include <assert.h>
#include <thrust/copy.h>

bool GPUGENIE_ERROR = false;
unsigned long long GPUGENIE_TIME = 0ull;

#ifndef GPUGenie_knn_THREADS_PER_BLOCK
#define GPUGenie_knn_THREADS_PER_BLOCK 1024
#endif

#ifndef GPUGenie_knn_DEFAULT_HASH_TABLE_SIZE
#define GPUGenie_knn_DEFAULT_HASH_TABLE_SIZE 1
#endif

#ifndef GPUGenie_knn_DEFAULT_BITMAP_BITS
#define GPUGenie_knn_DEFAULT_BITMAP_BITS 2
#endif

#ifndef GPUGenie_knn_DEFAULT_DATA_PER_THREAD
#define GPUGenie_knn_DEFAULT_DATA_PER_THREAD 256
#endif

__global__
void
convert_data(float * dd, data_t * od, int size)
{
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if(tId >= size) return;
	dd[tId] = od[tId].aggregation;
}

__global__
void
extract_index(int * id, data_t * od, int size)
{
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if(tId >= size) return;
	//if(od[id[tId]].aggregation != 0.0f)
		id[tId] = od[id[tId]].id;
	//else id[tId] = -1;
}

__global__
void
augment_bitmap(u32 * augmented,
			   u32 * selected,
			   u32 selected_size,
			   u32 augmented_size,
			   u32 num_per_u32)
{
	u32 id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id >= selected_size) return;
	u32 begin = id * GPUGenie_knn_DEFAULT_DATA_PER_THREAD;
	u32 data;
	u32 aug_id;
	u32 offset;
	for(u32 i = begin; i < begin+GPUGenie_knn_DEFAULT_DATA_PER_THREAD; ++i)
	{
		data = selected[i];
		aug_id = i * num_per_u32;
		if(aug_id >= augmented_size) return;
		for(u32 j = aug_id; j < aug_id + num_per_u32 && j < augmented_size; ++j)
		{
			offset = (32u/num_per_u32) * (j - aug_id);
			augmented[j] = (data >> offset) & ((1u << (32u/num_per_u32)) - 1u);
		}
	}
}

__global__
void
correct_index(int * index, int size, int k, int offset)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= size) return;

	index[id] -= offset * (id / k);
}
int
GPUGenie::calculate_bits_per_data(int bitmap_bits)
{
	  float logresult = log2((float) bitmap_bits);
	  bitmap_bits = (int) logresult;
	  if(logresult - bitmap_bits > 0)
	  {
		 bitmap_bits += 1;
	  }
	  logresult = log2((float)bitmap_bits);
	  bitmap_bits = (int) logresult;
	  if(logresult - bitmap_bits > 0)
	  {
		 bitmap_bits += 1;
	  }
	  bitmap_bits = pow(2, bitmap_bits);
	  return bitmap_bits;
}

void
GPUGenie::knn(GPUGenie::inv_table& table,
		   vector<GPUGenie::query>& queries,
		   device_vector<int>& d_top_indexes,
		   int max_load)
{
	int hash_table_size = GPUGenie_knn_DEFAULT_HASH_TABLE_SIZE * table.i_size() + 1;
	knn(table, queries, d_top_indexes, hash_table_size, max_load,GPUGenie_knn_DEFAULT_BITMAP_BITS);
}

void
GPUGenie::knn(GPUGenie::inv_table& table, vector<GPUGenie::query>& queries,
    device_vector<int>& d_top_indexes, int hash_table_size, int max_load,int bitmap_bits)
{
	knn(table, queries, d_top_indexes, hash_table_size,max_load, bitmap_bits, table.m_size(), 0, 0);
}


void
GPUGenie::knn(GPUGenie::inv_table& table,
		   vector<GPUGenie::query>& queries,
		   device_vector<int>& d_top_indexes,
		   int hash_table_size,
		   int max_load,
		   int bitmap_bits,
		   int dim)
{
	knn(table, queries, d_top_indexes, hash_table_size, max_load,bitmap_bits, dim, 0,0);
}
void
GPUGenie::knn_tweets(GPUGenie::inv_table& table,
		   vector<GPUGenie::query>& queries,
		   device_vector<int>& d_top_indexes,
		   int hash_table_size,
		   int max_load,
		   int bitmap_bits,
		   int dim,
		   int num_of_hot_dims,
		   int hot_dim_threshold)
{
  int qmax = 0;

  for(int i = 0; i < queries.size(); ++i)
  {
	 int count = queries[i].count_ranges();
	  if(count > qmax)
		  qmax = count;
  }
#ifdef GPUGENIE_DEBUG
  u64 start = getTime();
#endif
  knn(table, queries, d_top_indexes, hash_table_size,max_load, bitmap_bits,
		  	  float(qmax+1), num_of_hot_dims, hot_dim_threshold);
#ifdef GPUGENIE_DEBUG
  u64 end = getTime();
  double elapsed = getInterval(start, end);
  printf(">>>>>>> knn takes %fms <<<<<< \n", elapsed);
#endif
}
void
GPUGenie::knn(GPUGenie::inv_table& table,
		   vector<GPUGenie::query>& queries,
		   device_vector<int>& d_top_indexes,
		   int hash_table_size,
		   int max_load,
		   int bitmap_bits,
		   int dim,
		   int num_of_hot_dims,
		   int hot_dim_threshold)
{
#ifdef GPUGENIE_DEBUG
  printf("Parameters: %d,%d,%d,%d,%d\n", hash_table_size, bitmap_bits, dim, num_of_hot_dims, hot_dim_threshold);
#endif
  //for improve
//  int qmax = 0;
//  for(int i = 0; i < queries.size(); ++i)
//  {
//	 int count = queries[i].count_ranges();
//	  if(count > qmax)
//		  qmax = count;
//  }
  //end for improve
  dim = 2;
#ifdef GPUGENIE_DEBUG  //for improve
  u64 startKnn = getTime();
#endif

#ifdef GPUGENIE_DEBUG  //for improve
  u64 end3Knn = getTime();
  printf(">>>>> knn() before match() %f ms <<<<<\n", getInterval(startKnn, end3Knn));
#endif

  int bitmap_threshold = bitmap_bits;
  device_vector<data_t> d_data;
  device_vector<u32> d_bitmap;
  //device_vector<u32> d_selected_bitmap;//for improve
  //device_vector<u32> d_augmented_bitmap;//for improve

 // std::vector<int> selected_query_index;//for improve
  //device_vector<int> d_selected_top_indexes;//for improve
#ifdef GPUGENIE_DEBUG  //for improve
  u64 end2Knn = getTime();
  printf(">>>>> knn() before match() %f ms <<<<<\n", getInterval(startKnn, end2Knn));
#endif
  device_vector<u32> d_num_of_items_in_hashtable(queries.size());
  printf("[knn] max_load is %d.\n", max_load);
  match(table, queries, d_data, d_bitmap, hash_table_size,max_load, bitmap_bits, num_of_hot_dims, hot_dim_threshold, d_num_of_items_in_hashtable);
#ifdef GPUGENIE_DEBUG  //for improve
  u64 end1Knn = getTime();
  printf(">>>>> knn() after match() %f ms <<<<<\n", getInterval(startKnn, end1Knn));
#endif
  //cudaCheckErrors(hipDeviceSynchronize());

  //host_vector<u32> h_num_of_items_in_hashtable(d_num_of_items_in_hashtable);//for improve
  /**   Debug Section  **/
  // printf("The item count in hash map is:\n");
  // for(int i = 0; i < h_num_of_items_in_hashtable.size(); ++i)
  // {
	 //  printf("%d, ", h_num_of_items_in_hashtable[i]);
  // }
  // printf("\n");
  /** End of Debug Section **/

  //If no bitmap, then no need to collect topk in bitmap
  if(bitmap_bits > 1){
	  device_vector<u32> d_augmented_bitmap;//for improve
	  device_vector<u32> d_selected_bitmap;//for improve
	  std::vector<int> selected_query_index;//for improve
	  device_vector<int> d_selected_top_indexes;//for improve
	  host_vector<u32> h_num_of_items_in_hashtable(d_num_of_items_in_hashtable);//for improve
#ifdef GPUGENIE_DEBUG
	  u64 start = getTime();
#endif

	  //Calculate how many bits a data point is assigned to
	  bitmap_bits = calculate_bits_per_data(bitmap_bits);
	  u32 bitmap_size = table.i_size() / (32/bitmap_bits) + 1;
	  u32 num_per_u32 = 32u / bitmap_bits;

	  assert(32u % bitmap_bits == 0);

	  //Collect bitmap of queries with no enough topk result
	  for(u32 i = 0; i < h_num_of_items_in_hashtable.size(); ++i)
	  {
		  if(h_num_of_items_in_hashtable[i] < queries[i].topk())
		  {
			  selected_query_index.push_back(i);
			  d_selected_bitmap.insert(d_selected_bitmap.end(),
									   d_bitmap.begin()+i*bitmap_size,
									   d_bitmap.begin()+(i+1)*bitmap_size);
		  }
	  }
	  d_bitmap.clear();
	  device_vector<u32>().swap(d_bitmap);

#ifdef GPUGENIE_DEBUG
	  u64 end = getTime();
	  printf("size of selected bitmap: %d\n", d_selected_bitmap.size());
	  printf(">>>>> extract selected bitmaps takes %fms <<<<<\n", getInterval(start, end));
#endif

	  if(selected_query_index.size() > 0)
	  {
#ifdef GPUGENIE_DEBUG
		  start = getTime();
#endif
		  //Augment the selected bitmap
		  u32 augmented_size = selected_query_index.size() * table.i_size();
		  d_augmented_bitmap.resize(augmented_size);
		  u32 * d_selected_bitmap_p = thrust::raw_pointer_cast(d_selected_bitmap.data());
		  u32 * d_augmented_bitmap_p = thrust::raw_pointer_cast(d_augmented_bitmap.data());
		  assert(GPUGenie_knn_DEFAULT_DATA_PER_THREAD % num_per_u32 == 0);
		  u32 total_threads =  d_selected_bitmap.size() / (GPUGenie_knn_DEFAULT_DATA_PER_THREAD / num_per_u32) + 1;

		  augment_bitmap<<<total_threads/GPUGenie_knn_THREADS_PER_BLOCK + 1,GPUGenie_knn_THREADS_PER_BLOCK>>>
				  (d_augmented_bitmap_p, d_selected_bitmap_p, d_selected_bitmap.size(), augmented_size, num_per_u32);
		  //cudaCheckErrors(hipDeviceSynchronize());
#ifdef GPUGENIE_DEBUG
		  end = getTime();
		  printf(">>>>> augment bitmap takes %fms <<<<<\n", getInterval(start, end));
		  start = getTime();
#endif
		  /**   Debug Section  **/
		  // host_vector<u32> h_augmented_bitmap(d_augmented_bitmap);
		  // printf("The first 2048 items in augmented bitmap are:\n");
		  // for(int i = 0; i < 2048; ++i)
		  // {
			 //  printf("%d, ", h_augmented_bitmap[i]);
		  // }
		  // printf("\n");
		  // host_vector<u32> h_selected_bitmap(d_selected_bitmap);
		  // printf("The first 8 items in selected bitmap are:\n");
		  // for(int i = 0; i < 2; ++i)
		  // {
			 //  printf("%d, ", h_selected_bitmap[i]);
		  // }
		  // printf("\n");
		  /** End of Debug Section **/

		  d_selected_bitmap.clear();
		  device_vector<u32>().swap(d_selected_bitmap);

		  std::vector<GPUGenie::query> selected_queries;

		  //In case queries have different topk settings
		  u32 index_size = 0u;
		  for(u32 i = 0; i < selected_query_index.size(); ++i)
		  {
			selected_queries.push_back(queries[selected_query_index[i]]);
			index_size += selected_queries[i].topk();
		  }

		  topk(d_augmented_bitmap, selected_queries, d_selected_top_indexes, u32(bitmap_threshold));
		  //cudaCheckErrors(hipDeviceSynchronize());

		  d_augmented_bitmap.clear();
		  device_vector<u32>().swap(d_augmented_bitmap);
		  /**   Debug Section  **/
		  // printf("The items in d_selected_top_indexes are:\n");
		  // for(u32 i = 0; i < selected_query_index.size(); ++i)
		  // {
		  // 	  printf("Query %d:\n", selected_query_index[i]);
		  // 	  for(u32 j = 0; j < selected_queries[i].topk(); ++j)
		  // 	  {
				// printf("%d, ", h_selected_top_indexes[i * selected_queries[i].topk() + j] - i * table.i_size());
		  // 	  }
			 //  printf("\n");
		  // }
		  /** End of Debug Section **/

 //		  for(u32 i = 0; i < selected_query_index.size(); ++i)
 //		  {
 //			  for(u32 j = 0; j < selected_queries[i].topk(); ++j)
 //			  {
 //				 h_selected_top_indexes[i * selected_queries[i].topk() + j] -= table.i_size()*i;
 //			  }
 //		  }
#ifdef GPUGENIE_DEBUG
		  end = getTime();
		  printf(">>>>> selected bitmap topk takes %fms <<<<<\n", getInterval(start, end));
		  start =getTime();
#endif
		  int * d_selected_top_indexes_p = thrust::raw_pointer_cast(d_selected_top_indexes.data());
		  correct_index<<<d_selected_top_indexes.size() / GPUGenie_knn_THREADS_PER_BLOCK + 1, GPUGenie_knn_THREADS_PER_BLOCK>>>
					   (d_selected_top_indexes_p, d_selected_top_indexes.size(), queries[0].topk(), table.i_size());
		  //cudaCheckErrors(hipDeviceSynchronize());
#ifdef GPUGENIE_DEBUG
		  end = getTime();
		  printf(">>>>>correct selected bitmap index takes %fms <<<<<\n", getInterval(start, end));
#endif
		  /**   Debug Section  **/
 //		   printf("The items in d_selected_top_indexes are:\n");
 //		   for(u32 i = 0; i < selected_query_index.size(); ++i)
 //		   {
 //		   	  printf("Query %d:\n", selected_query_index[i]);
 //		   	  for(u32 j = 0; j < selected_queries[i].topk(); ++j)
 //		   	  {
 //				 printf("%d, ", h_selected_top_indexes[i * selected_queries[i].topk() + j]);
 //		   	  }
 //			   printf("\n");
 //		   }
		  /** End of Debug Section **/
	  }

  }

#ifdef GPUGENIE_DEBUG  //for improve
  u64 endKnn = getTime();
  printf(">>>>> knn() before topk and extractIndex %f ms <<<<<\n", getInterval(startKnn, endKnn));
#endif

#ifdef GPUGENIE_DEBUG
  printf("Start topk....\n");
  u64 start = getTime();
#endif

  topk(d_data, queries, d_top_indexes, float(dim));
  //cudaCheckErrors(hipDeviceSynchronize());

#ifdef GPUGENIE_DEBUG
  u64 end = getTime();
  printf("Topk Finished! \n");
  printf(">>>>> main topk takes %fms <<<<<\n", getInterval(start, end));
  start=getTime();
#endif

  extract_index<<<d_top_indexes.size() / GPUGenie_knn_THREADS_PER_BLOCK + 1, GPUGenie_knn_THREADS_PER_BLOCK>>>
			   (thrust::raw_pointer_cast(d_top_indexes.data()), thrust::raw_pointer_cast(d_data.data()), d_top_indexes.size());
  //cudaCheckErrors(hipDeviceSynchronize());

  // If has selected topk results, then overwrite the main vector d_top_indexes with the selected results
//  if(d_selected_top_indexes.size() != 0 && selected_query_index.size() != 0)//for improve
//  {
//	  int qid;
//	  for(u32 i = 0; i < selected_query_index.size(); ++i)
//	  {
//		  qid = selected_query_index[i];
//		  thrust::copy(d_selected_top_indexes.begin()+i*queries[qid].topk(),
//					   d_selected_top_indexes.begin()+(i+1)*queries[qid].topk(),
//					   d_top_indexes.begin()+qid*queries[qid].topk());
//	  }
//  }

#ifdef GPUGENIE_DEBUG
  end=getTime();
  printf("Finish topk search!\n");
  printf(">>>>> extract index and copy selected topk results takes %fms <<<<<\n", getInterval(start, end));
#endif
}
