#include "hip/hip_runtime.h"
#include "knn.h"
#include <math.h>
#include <assert.h>
#include <thrust/copy.h>

bool GPUGENIE_ERROR = false;
unsigned long long GPUGENIE_TIME = 0ull;

#ifndef GPUGenie_knn_THREADS_PER_BLOCK
#define GPUGenie_knn_THREADS_PER_BLOCK 1024
#endif

#ifndef GPUGenie_knn_DEFAULT_HASH_TABLE_SIZE
#define GPUGenie_knn_DEFAULT_HASH_TABLE_SIZE 1
#endif

#ifndef GPUGenie_knn_DEFAULT_BITMAP_BITS
#define GPUGenie_knn_DEFAULT_BITMAP_BITS 2
#endif

#ifndef GPUGenie_knn_DEFAULT_DATA_PER_THREAD
#define GPUGenie_knn_DEFAULT_DATA_PER_THREAD 256
#endif

__global__
void
extract_index_and_count(int * id, int * count, data_t * od, int size)
{
	int tId = threadIdx.x + blockIdx.x * blockDim.x;
	if(tId >= size) return;
	int topk_id = id[tId];
	id[tId] = od[topk_id].id;
	count[tId] = (int) od[topk_id].aggregation;
}

void
GPUGenie::knn_bijectMap(GPUGenie::inv_table& table,
		   vector<GPUGenie::query>& queries,
		   device_vector<int>& d_top_indexes,
		   device_vector<int>& d_top_count,
		   int hash_table_size,
		   int max_load,
		   int bitmap_bits,
		   int dim,
		   int num_of_hot_dims,
		   int hot_dim_threshold)
{
  int qmax = 0;

  for(int i = 0; i < queries.size(); ++i)
  {
	 int count = queries[i].count_ranges();
	  if(count > qmax)
		  qmax = count;
  }
#ifdef GPUGENIE_DEBUG
  u64 start = getTime();
#endif
  knn(table, queries, d_top_indexes,d_top_count, hash_table_size,max_load, bitmap_bits,
		  	  float(qmax+1), num_of_hot_dims, hot_dim_threshold);
#ifdef GPUGENIE_DEBUG
  u64 end = getTime();
  double elapsed = getInterval(start, end);
  printf(">>>>>>> knn takes %fms <<<<<< \n", elapsed);
#endif
}
void
GPUGenie::knn(GPUGenie::inv_table& table,
		   vector<GPUGenie::query>& queries,
		   device_vector<int>& d_top_indexes,
		   device_vector<int>& d_top_count,
		   int hash_table_size,
		   int max_load,
		   int bitmap_bits,
		   int dim,
		   int num_of_hot_dims,
		   int hot_dim_threshold)
{
#ifdef GPUGENIE_DEBUG
  printf("Parameters: %d,%d,%d,%d,%d\n", hash_table_size, bitmap_bits, dim, num_of_hot_dims, hot_dim_threshold);
#endif
  //for improve
//  int qmax = 0;
//  for(int i = 0; i < queries.size(); ++i)
//  {
//	 int count = queries[i].count_ranges();
//	  if(count > qmax)
//		  qmax = count;
//  }
  //end for improve
  dim = 2;
#ifdef GPUGENIE_DEBUG  //for improve
  u64 startKnn = getTime();
#endif

#ifdef GPUGENIE_DEBUG  //for improve
  u64 end3Knn = getTime();
  printf(">>>>> knn() before match() %f ms <<<<<\n", getInterval(startKnn, end3Knn));
#endif

  device_vector<data_t> d_data;
  device_vector<u32> d_bitmap;

#ifdef GPUGENIE_DEBUG  //for improve
  u64 end2Knn = getTime();
  printf(">>>>> knn() before match() %f ms <<<<<\n", getInterval(startKnn, end2Knn));
#endif
  device_vector<u32> d_num_of_items_in_hashtable(queries.size());
  printf("[knn] max_load is %d.\n", max_load);
  match(table, queries, d_data, d_bitmap, hash_table_size,max_load, bitmap_bits, num_of_hot_dims, hot_dim_threshold, d_num_of_items_in_hashtable);
#ifdef GPUGENIE_DEBUG  //for improve
  u64 end1Knn = getTime();
  printf(">>>>> knn() after match() %f ms <<<<<\n", getInterval(startKnn, end1Knn));
#endif

#ifdef GPUGENIE_DEBUG  //for improve
  u64 endKnn = getTime();
  printf(">>>>> knn() before topk and extractIndex %f ms <<<<<\n", getInterval(startKnn, endKnn));
#endif

#ifdef GPUGENIE_DEBUG
  printf("Start topk....\n");
  u64 start = getTime();
#endif

  topk(d_data, queries, d_top_indexes, float(dim));

#ifdef GPUGENIE_DEBUG
  u64 end = getTime();
  printf("Topk Finished! \n");
  printf(">>>>> main topk takes %fms <<<<<\n", getInterval(start, end));
  start=getTime();
#endif
  d_top_count.resize(d_top_indexes.size());
  extract_index_and_count<<<d_top_indexes.size() / GPUGenie_knn_THREADS_PER_BLOCK + 1, GPUGenie_knn_THREADS_PER_BLOCK>>>
			   (thrust::raw_pointer_cast(d_top_indexes.data()),
			    thrust::raw_pointer_cast(d_top_count.data()),
			    thrust::raw_pointer_cast(d_data.data()),
			    d_top_indexes.size());

#ifdef GPUGENIE_DEBUG
  end=getTime();
  printf("Finish topk search!\n");
  printf(">>>>> extract index and copy selected topk results takes %fms <<<<<\n", getInterval(start, end));
#endif
}
