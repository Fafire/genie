#include "hip/hip_runtime.h"

#include <algorithm>
#include <iostream>
#include <stdlib.h>
#include <string>
#include <sstream>
#include <math.h>

#include <thrust/copy.h>
#include <thrust/device_vector.h>

#include "match.h"
#include "Logger.h"
#include "Timing.h"
#include "genie_errors.h"
#include "DeviceCompositeCodec.h"
#include "DeviceBitPackingCodec.h"
#include "DeviceVarintCodec.h"

#include "match_integrated.h"

const size_t MATCH_THREADS_PER_BLOCK = 256;

const size_t DECOMPR_BLOCKS = 1;
const size_t DECOMPR_THREADS_PER_BLOCK = 256;
const size_t DECOMPR_BATCH = DECOMPR_BLOCKS * DECOMPR_THREADS_PER_BLOCK;

#define OFFSETS_TABLE_16 {0u,       3949349u, 8984219u, 9805709u,\
                          7732727u, 1046459u, 9883879u, 4889399u,\
                          2914183u, 3503623u, 1734349u, 8860463u,\
                          1326319u, 1613597u, 8604269u, 9647369u}

#define DEBUG_KERNELS

typedef u64 T_HASHTABLE;
typedef u32 T_KEY;
typedef u32 T_AGE;


namespace GPUGenie
{

std::map<std::string, IntegratedKernelPtr> initIntegratedKernels()
{

    std::map<std::string, IntegratedKernelPtr> kernels;

    kernels["copy"] = match_integrated<DeviceJustCopyCodec>;
    kernels["d1"] = match_integrated<DeviceDeltaCodec>;
    kernels["bp32"] = match_integrated<DeviceBitPackingCodec>;
    kernels["varint"] = match_integrated<DeviceVarintCodec>;
    kernels["bp32-copy"] = match_integrated<DeviceCompositeCodec<DeviceBitPackingCodec,DeviceJustCopyCodec>>;
    kernels["bp32-varint"] = match_integrated<DeviceCompositeCodec<DeviceBitPackingCodec,DeviceVarintCodec>>;

    return kernels;
}

std::map<std::string, IntegratedKernelPtr> integratedKernels = initIntegratedKernels();

template <class Codec> __global__ void
match_adaptiveThreshold_integrated(
        int m_size, // number of dimensions, i.e. inv_table::m_size()
        int i_size, // number of instances, i.e. inv_table::m_size() * (1u<<shift_bits_subsequence)
        int hash_table_size, // hash table size
        int* d_compr_inv, // d_uncompr_inv_p points to the start location of uncompr posting list array in GPU memory
        query::dim* d_dims, // compiled queries (dim structure) with locations into d_uncompr_inv
        size_t decomprDimsOffset, // offset for d_dims to be used
        T_HASHTABLE* hash_table_list, // data_t struct (id, aggregation) array of size queries.size() * hash_table_size
        u32 * bitmap_list, // of bitmap_size
        int bitmap_bits,
        u32* d_topks, // d_topks set to max_topk for all queries
        u32* d_threshold, //initialized as 1, and increase gradually
        u32* d_passCount, //initialized as 0, count the number of items passing one d_threshold
        u32 num_of_max_count, //number of maximum count per query
        u32 * noiih, // number of integers in a hash table; set to 0 for all queries
        bool * overflow,
        unsigned int shift_bits_subsequence)
{
    if (m_size == 0 || i_size == 0)
        return;
    query::dim& myb_query = d_dims[blockIdx.x + decomprDimsOffset];
    int query_index = myb_query.query;
    u32* my_noiih = &noiih[query_index];
    u32* my_threshold = &d_threshold[query_index];
    u32* my_passCount = &d_passCount[query_index * num_of_max_count];         //
    u32 my_topk = d_topks[query_index];                //for AT

    T_HASHTABLE* hash_table = &hash_table_list[query_index * hash_table_size];
    u32 * bitmap;
    if (bitmap_bits)
        bitmap = &bitmap_list[query_index * (i_size / (32 / bitmap_bits) + 1)];
    u32 access_id;
    int min, max, order;
    if(myb_query.start_pos >= myb_query.end_pos)
        return;

    min = myb_query.start_pos;
    max = myb_query.end_pos;
    order = myb_query.order;
    bool key_eligible;                //
    bool pass_threshold;    //to determine whether pass the check of my_theshold

    // Iterate the posting lists array between q.start_pos and q.end_pos in blocks of MATCH_THREADS_PER_BLOCK
    // docsIDs, where each thread reads one docID at a time
    for (int i = 0; i < (max - min - 1) / MATCH_THREADS_PER_BLOCK + 1; ++i)
    {
        // index to read from the posting posts array
        int tmp_id = threadIdx.x + i * MATCH_THREADS_PER_BLOCK + min;
        if (tmp_id < max)
        {
            u32 count = 0;                //for AT
            access_id = d_uncompr_inv[tmp_id]; // retrieved docID from posting lists array

            if(shift_bits_subsequence != 0)
            {
                int __offset = access_id & (((unsigned int)1<<shift_bits_subsequence) - 1);
                int __new_offset = __offset - order;
                if(__new_offset >= 0)
                {
                    access_id = access_id - __offset + __new_offset;
                }
                else
                    continue;
            }

            u32 thread_threshold = *my_threshold;
            if (bitmap_bits)
            {

                key_eligible = false;
                //all count are store in the bitmap, and access the count
                count = bitmap_kernel_AT(access_id, bitmap, bitmap_bits,
                        thread_threshold, &key_eligible);

                if (!key_eligible)
                    continue;                //i.e. count< thread_threshold
            }

            key_eligible = false;
            if (count < *my_threshold)
            {
                continue;      //threshold has been increased, no need to insert
            }

            //Try to find the entry in hash tables
            access_kernel_AT(
                    access_id,               
                    hash_table, hash_table_size, myb_query, count, &key_eligible,
                    my_threshold, &pass_threshold);

            if (key_eligible)
            {
                if (pass_threshold)
                {
                    updateThreshold(my_passCount, my_threshold, my_topk, count);
                }

                continue;
            }

            if (!key_eligible)
            {
                //Insert the key into hash table
                //access_id and its location are packed into a packed key

                if (count < *my_threshold)
                {
                    continue;//threshold has been increased, no need to insert
                }

                hash_kernel_AT(access_id, hash_table, hash_table_size, myb_query, count,
                        my_threshold, my_noiih, overflow, &pass_threshold);
                if (*overflow)
                {

                    return;
                }
                if (pass_threshold)
                {
                    updateThreshold(my_passCount, my_threshold, my_topk, count);
                }
            }

        }
    }
}

template <class Codec> void
match_integrated(
            inv_compr_table& table,
            vector<query>& queries,
            device_vector<data_t>& d_data,
            device_vector<u32>& d_bitmap,
            int hash_table_size,
            int bitmap_bits, //or for AT: for adaptiveThreshold, if bitmap_bits<0, use adaptive threshold, the absolute value of bitmap_bits is count value stored in the bitmap
            device_vector<u32>& d_noiih,
            device_vector<u32>& d_threshold,
            device_vector<u32>& d_passCount)
{
    try{
        Logger::log(Logger::DEBUG, "Started match()");
        Logger::log(Logger::DEBUG, "hash table size: %d.", hash_table_size);

        u32 shift_bits_subsequence = table._shift_bits_subsequence();

        if (table.build_status() == inv_table::not_builded)
            throw GPUGenie::cpu_runtime_error("table not built!");
        
        // Time measuring events
        hipEvent_t kernel_start, kernel_stop;
        hipEvent_t startMatching, stopMatching, startConvert, stopConvert;
        hipEventCreate(&startMatching);
        hipEventCreate(&stopMatching);
        hipEventCreate(&startConvert);
        hipEventCreate(&stopConvert);
        hipEventCreate(&kernel_start);
        hipEventCreate(&kernel_stop);
        float matchDecomprTime = 0.0, convertTime, kernelsTime = 0.0;
        u64 match_stop, match_start;
        match_start = getTime();

        Logger::log(Logger::INFO, "[  0%] Starting matching...");
        
        d_noiih.resize(queries.size(), 0);
        u32 * d_noiih_p = thrust::raw_pointer_cast(d_noiih.data());

        vector<query::dim> dims;        
        //number of maximum count per query
        u32 num_of_max_count = build_queries(queries, table, dims, table.getUncompressedPostingListMaxLength());

        Logger::log(Logger::DEBUG, "num_of_max_count: %d", num_of_max_count);
        

        //for AT: for adaptiveThreshold, enable adaptiveThreshold
        if (bitmap_bits < 0)
            bitmap_bits = -bitmap_bits; //for hash_table_size, still let it determine by users currently

        Logger::log(Logger::DEBUG, "[info] bitmap_bits:%d.", bitmap_bits);
        //end for AT

        int threshold = bitmap_bits - 1, bitmap_size = 0;
        if (bitmap_bits > 1)
        {
            float logresult = std::log2((float) bitmap_bits);
            bitmap_bits = (int) logresult;
            if (logresult - bitmap_bits > 0)
            {
                bitmap_bits += 1;
            }
            logresult = std::log2((float) bitmap_bits);
            bitmap_bits = (int) logresult;
            if (logresult - bitmap_bits > 0)
            {
                bitmap_bits += 1;
            }
            bitmap_bits = pow(2, bitmap_bits);
            bitmap_size = ((((unsigned int)1<<shift_bits_subsequence) * table.i_size()) / (32 / bitmap_bits) + 1)
                    * queries.size();
        }
        else
        {
            bitmap_bits = threshold = 0;
        }

        Logger::log(Logger::DEBUG, "Bitmap bits: %d, threshold:%d, shift_bits_subsequence: %d",
            bitmap_bits, threshold, shift_bits_subsequence);


        Logger::log(Logger::INFO, "[ 20%] Declaring device memory...");


        u64 query_start, query_end;
        query_start = getTime();
        thrust::device_vector<query::dim> d_dims(dims);
        query::dim* d_dims_p = thrust::raw_pointer_cast(d_dims.data());
        query_end  = getTime();
        Logger::log(Logger::DEBUG, "query_transfer time: %d",getInterval(query_start, query_end));
        
        if (table.get_total_num_of_table() > 1 || !table.is_stored_in_gpu)
            table.cpy_data_to_gpu();

        d_bitmap.resize(bitmap_size);
        if (bitmap_size)
            thrust::fill(d_bitmap.begin(), d_bitmap.end(), 0u);

        u32 * d_bitmap_p = raw_pointer_cast(d_bitmap.data());



        Logger::log(Logger::INFO, "[ 30%] Allocating device memory to tables...");

        data_t nulldata;
        nulldata.id = 0u;
        nulldata.aggregation = 0.0f;
        T_HASHTABLE* d_hash_table;
        data_t* d_data_table;
        d_data.clear();

        d_data.resize(queries.size() * hash_table_size, nulldata);
        d_data_table = thrust::raw_pointer_cast(d_data.data());
        d_hash_table = reinterpret_cast<T_HASHTABLE*>(d_data_table);


        Logger::log(Logger::INFO, "[ 32%] Allocating device memory for uncompressed posting lists...");

        size_t d_uncompr_inv_size = DECOMPR_BATCH * table.getUncompressedPostingListMaxLength();
        thrust::device_vector<int> d_uncompr_inv(d_uncompr_inv_size);
        int *d_uncompr_inv_p = thrust::raw_pointer_cast(d_uncompr_inv.data());


        Logger::log(Logger::INFO, "[ 33%] Copying memory to symbol...");

        u32 h_offsets[16] = OFFSETS_TABLE_16;
        cudaCheckErrors(hipMemcpyToSymbol(HIP_SYMBOL(GPUGenie::device::offsets), h_offsets, sizeof(u32)*16, 0, hipMemcpyHostToDevice));


        Logger::log(Logger::INFO,"[ 40%] Starting decompression & match kernels...");

        hipEventRecord(kernel_start);

        bool h_overflow[1] = {false};
        bool * d_overflow;
        cudaCheckErrors(hipMalloc((void**) &d_overflow, sizeof(bool)));

        u32 loop_count = 1u;
        do
        {
            // Set overflow to false
            h_overflow[0] = false;
            cudaCheckErrors(hipMemcpy(d_overflow, h_overflow, sizeof(bool), hipMemcpyHostToDevice));

            // Set threshold to 1 for all queries
            d_threshold.resize(queries.size());
            thrust::fill(d_threshold.begin(), d_threshold.end(), 1);
            u32 * d_threshold_p = thrust::raw_pointer_cast(d_threshold.data());
            
            // Set d_passCount to 0 for all queries and all num_of_max_count
            d_passCount.resize(queries.size()*num_of_max_count);
            thrust::fill(d_passCount.begin(), d_passCount.end(), 0u);
            u32 * d_passCount_p = thrust::raw_pointer_cast(d_passCount.data());

            // Set d_topks to 0 for all queries
            u32 max_topk = cal_max_topk(queries);
            device_vector<u32> d_topks;
            d_topks.resize(queries.size());
            thrust::fill(d_topks.begin(), d_topks.end(), max_topk);
            u32 * d_topks_p = thrust::raw_pointer_cast(d_topks.data());

            
            // Call matching kernel, where each BLOCK does matching of one compiled query, only matching for the
            // next DECOMPR_BATCH compiled queries is done in one invocation of the kernel -- this corresponds to
            // the number of decompressed invereted lists
            hipEventRecord(startMatching);

            throw std::logic_error("No integrated kernel available!");

            hipEventRecord(startMatching);
            match_adaptiveThreshold_queryPerBlock_compressed<Codec><<<batch,MATCH_THREADS_PER_BLOCK>>>
                   (table.m_size(),
                    table.i_size() * ((unsigned int)1<<shift_bits_subsequence),
                    hash_table_size, // hash table size
                    // d_compr_inv points to the start location of compressed posting list array in GPU mem
                    d_compr_inv, 
                    // compiled queries (dim structure)
                    d_dims_p,
                    // offset into compiled queries
                    dimsOffset,
                    d_hash_table, // data_t struct (id, aggregation) array of size queries.size() * hash_table_size
                    d_bitmap_p, // of bitmap_size
                    bitmap_bits,
                    d_topks_p, // d_topks set to max_topk for all queries
                    d_threshold_p,//initialized as 1, and increase gradually
                    d_passCount_p,//initialized as 0, count the number of items passing one d_threshold
                    num_of_max_count,//number of maximum count per query
                    d_noiih_p, // number of integers in a hash table set to 0 for all queries
                    d_overflow, // bool
                    shift_bits_subsequence);
            hipEventRecord(stopMatching);
            hipEventSynchronize(stopMatching);
            hipEventElapsedTime(&matchDecomprTime, startMatching, stopMatching);

            cudaCheckErrors(hipDeviceSynchronize());
            

            // Increase hash table size in case there was an overflow
            cudaCheckErrors(hipMemcpy(h_overflow, d_overflow, sizeof(bool), hipMemcpyDeviceToHost));
            if(h_overflow[0])
            {
                hash_table_size += num_of_max_count*max_topk;
                if(hash_table_size > table.i_size())
                {
                    hash_table_size = table.i_size();
                }
                thrust::fill(d_noiih.begin(), d_noiih.end(), 0u);
                if(bitmap_size)
                {
                    thrust::fill(d_bitmap.begin(), d_bitmap.end(), 0u);
                }
                d_data.resize(queries.size()*hash_table_size);
                thrust::fill(d_data.begin(), d_data.end(), nulldata);
                d_data_table = thrust::raw_pointer_cast(d_data.data());
                d_hash_table = reinterpret_cast<T_HASHTABLE*>(d_data_table);
            }

            // Log failed matching attempt
            if (loop_count>1 || (loop_count == 1 && h_overflow[0]))
                Logger::log(Logger::INFO,"%d time trying to launch match kernel: %s!",
                    loop_count, h_overflow[0]?"failed":"succeeded");
            loop_count ++;

        }while(h_overflow[0]);

        cudaCheckErrors(hipFree(d_overflow));

        hipEventRecord(kernel_stop);
        Logger::log(Logger::INFO,"[ 90%] Starting data converting......");

        hipEventRecord(startConvert);
        device::convert_to_data<<<hash_table_size*queries.size() / 1024 + 1,1024>>>(
            d_hash_table,(u32)hash_table_size*queries.size());
        hipEventRecord(stopConvert);

        hipEventSynchronize(stopConvert
            );
        Logger::log(Logger::INFO, "[100%] Matching is done!");

        match_stop = getTime();

        hipEventElapsedTime(&kernelsTime, kernel_start, kernel_stop);
        hipEventElapsedTime(&convertTime, startConvert, stopConvert);


        Logger::log(Logger::INFO,
                ">>>>[time profiling]: Decompresison kernels take %f ms. (GPU only) ",
                decomprTime);
        Logger::log(Logger::INFO,
                ">>>>[time profiling]: Match kernels take %f ms. (GPU only) ",
                matchTime);
        Logger::log(Logger::INFO,
                ">>>>[time profiling]: Decompression and match kernels take %f ms. (GPU only) ",
                kernelsTime);
        Logger::log(Logger::INFO,
                ">>>>[time profiling]: Conversion kernel takes %f ms. (GPU only) ",
                convertTime);
        Logger::log(Logger::INFO,
                ">>>>[time profiling]: Match function takes %f ms. (GPU+CPU)",
                getInterval(match_start, match_stop));
        Logger::log(Logger::VERBOSE, ">>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>");

    } catch(std::bad_alloc &e){
        throw GPUGenie::gpu_bad_alloc(e.what());
    }
}

}
