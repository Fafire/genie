#include "hip/hip_runtime.h"

#include <algorithm>
#include <iostream>
#include <stdlib.h>
#include <string>
#include <sstream>
#include <math.h>

#include <thrust/copy.h>
#include <thrust/device_vector.h>

#include "match.h"
#include "Logger.h"
#include "PerfLogger.hpp"
#include "Timing.h"
#include "genie_errors.h"
#include "DeviceCompositeCodec.h"
#include "DeviceBitPackingCodec.h"
#include "DeviceVarintCodec.h"

#include "match_integrated.h"

const size_t MATCH_THREADS_PER_BLOCK = 256;

#define OFFSETS_TABLE_16 {0u,       3949349u, 8984219u, 9805709u,\
                          7732727u, 1046459u, 9883879u, 4889399u,\
                          2914183u, 3503623u, 1734349u, 8860463u,\
                          1326319u, 1613597u, 8604269u, 9647369u}

/**
 * Maximal length the codecs are able to decompress into.
 *
 * GENIE uses fixed 256 threads in its kernels. This implies that a Codec has to have a thread load at least 4 (one
 * thread decompressed into 4 values), otherwise such codec will fail.
 */
#define GPUGENIE_INTEGRATED_KERNEL_SM_SIZE (1024)

typedef u64 T_HASHTABLE;
typedef u32 T_KEY;
typedef u32 T_AGE;


namespace GPUGenie
{

template void match_integrated<DeviceCopyCodec>(inv_compr_table&, std::vector<query>&,
thrust::device_vector<data_t>&, thrust::device_vector<u32>&, int, int, thrust::device_vector<u32>&,
thrust::device_vector<u32>&, thrust::device_vector<u32>&);

template void match_integrated<DeviceDeltaCodec>(inv_compr_table&, std::vector<query>&, thrust::device_vector<data_t>&,
thrust::device_vector<u32>&, int, int, thrust::device_vector<u32>&, thrust::device_vector<u32>&,
thrust::device_vector<u32>&);

template void match_integrated<DeviceBitPackingCodec>(inv_compr_table&, std::vector<query>&,
thrust::device_vector<data_t>&, thrust::device_vector<u32>&, int, int, thrust::device_vector<u32>&,
thrust::device_vector<u32>&, thrust::device_vector<u32>&);

template void match_integrated<DeviceVarintCodec>(inv_compr_table&, std::vector<query>&, thrust::device_vector<data_t>&,
thrust::device_vector<u32>&, int, int, thrust::device_vector<u32>&, thrust::device_vector<u32>&,
thrust::device_vector<u32>&);

template void match_integrated<DeviceCompositeCodec<DeviceBitPackingCodec,DeviceCopyCodec>>(inv_compr_table&,
std::vector<query>&, thrust::device_vector<data_t>&, thrust::device_vector<u32>&, int, int, thrust::device_vector<u32>&,
thrust::device_vector<u32>&, thrust::device_vector<u32>&);

template void match_integrated<DeviceCompositeCodec<DeviceBitPackingCodec,DeviceVarintCodec>>(inv_compr_table&,
std::vector<query>&, thrust::device_vector<data_t>&, thrust::device_vector<u32>&, int, int, thrust::device_vector<u32>&,
thrust::device_vector<u32>&, thrust::device_vector<u32>&);


std::map<std::string, IntegratedKernelPtr> initIntegratedKernels()
{

    std::map<std::string, IntegratedKernelPtr> kernels;

    kernels["copy"] = match_integrated<DeviceCopyCodec>;
    kernels["d1"] = match_integrated<DeviceDeltaCodec>;
    kernels["bp32"] = match_integrated<DeviceBitPackingCodec>;
    kernels["varint"] = match_integrated<DeviceVarintCodec>;
    kernels["bp32-copy"] = match_integrated<DeviceCompositeCodec<DeviceBitPackingCodec,DeviceCopyCodec>>;
    kernels["bp32-varint"] = match_integrated<DeviceCompositeCodec<DeviceBitPackingCodec,DeviceVarintCodec>>;

    return kernels;
}

std::map<std::string, IntegratedKernelPtr> integratedKernels = initIntegratedKernels();



template <class Codec> __global__ void
match_adaptiveThreshold_integrated(
        int m_size, // number of dimensions, i.e. inv_table::m_size()
        int i_size, // number of instances, i.e. inv_table::m_size() * (1u<<shift_bits_subsequence)
        int hash_table_size, // hash table size
        uint32_t* d_compr_inv, // d_uncompr_inv_p points to the start location of uncompr posting list array in GPU memory
        query::dim* d_dims, // compiled queries (dim structure) with locations into d_uncompr_inv
        T_HASHTABLE* hash_table_list, // data_t struct (id, aggregation) array of size queries.size() * hash_table_size
        u32 * bitmap_list, // of bitmap_size
        int bitmap_bits,
        u32* d_topks, // d_topks set to max_topk for all queries
        u32* d_threshold, //initialized as 1, and increase gradually
        u32* d_passCount, //initialized as 0, count the number of items passing one d_threshold
        u32 num_of_max_count, //number of maximum count per query
        u32 * noiih, // number of integers in a hash table; set to 0 for all queries
        bool * overflow,
        unsigned int shift_bits_subsequence)
{
    assert(MATCH_THREADS_PER_BLOCK == blockDim.x);

    assert(m_size != 0 && i_size != 0);

    query::dim& myb_query = d_dims[blockIdx.x];
    int query_index = myb_query.query;
    u32* my_noiih = &noiih[query_index];
    u32* my_threshold = &d_threshold[query_index];
    u32* my_passCount = &d_passCount[query_index * num_of_max_count];         //
    u32 my_topk = d_topks[query_index];                //for AT

    T_HASHTABLE* hash_table = &hash_table_list[query_index * hash_table_size];
    u32 * bitmap;
    if (bitmap_bits)
        bitmap = &bitmap_list[query_index * (i_size / (32 / bitmap_bits) + 1)];

    assert(myb_query.start_pos < myb_query.end_pos);

    int min = myb_query.start_pos;
    int max = myb_query.end_pos;
    size_t comprLength = max - min;
    int order = myb_query.order;

    Codec codec;
    // check if Codec is compatible with the current list
    assert(max - min <= codec.decodeArrayParallel_maxBlocks() * codec.decodeArrayParallel_lengthPerBlock());
    assert(max - min <= gridDim.x * blockDim.x * codec.decodeArrayParallel_threadLoad());
    assert(blockDim.x == codec.decodeArrayParallel_lengthPerBlock() / codec.decodeArrayParallel_threadLoad());

    __shared__ uint32_t s_comprInv[GPUGENIE_INTEGRATED_KERNEL_SM_SIZE];
    __shared__ uint32_t s_decomprInv[GPUGENIE_INTEGRATED_KERNEL_SM_SIZE];

    int idx = threadIdx.x;
    // Copy the compressed list from global memory into shared memory
    // TODO change to more coalesced access (each thread accesses consecutive 128b value)
    for (int i = 0; i < codec.decodeArrayParallel_lengthPerBlock(); i += codec.decodeArrayParallel_threadsPerBlock())
    {
        s_comprInv[idx + i] = (idx + i < (int)comprLength) ? d_compr_inv[idx + i + min] : 0;
        s_decomprInv[idx + i] = 0;
    }
    // set uncompressed length to maximal length, decomprLength also acts as capacity for the codec
    size_t decomprLength = GPUGENIE_INTEGRATED_KERNEL_SM_SIZE;
    __syncthreads();
    codec.decodeArrayParallel(s_comprInv, comprLength, s_decomprInv, decomprLength);
    __syncthreads();

    // if (idx == 0)
    //     printf("Block %d, query %d, start_pos %d, end_pos %d, comprLength %d, decomprLength %d,\n    compr values [0x%08x,0x%08x,0x%08x,0x%08x,0x%08x,0x%08x,0x%08x,0x%08x,0x%08x,0x%08x],\n    decompr values [%d,%d,%d,%d,%d,%d,%d,%d,%d,%d] \n",
    //         blockIdx.x, query_index, min, max, (int)comprLength, (int)decomprLength,
    //         s_comprInv[0], s_comprInv[1], s_comprInv[2], s_comprInv[3], s_comprInv[4], 
    //         s_comprInv[5], s_comprInv[6], s_comprInv[7], s_comprInv[8], s_comprInv[9],
    //         s_decomprInv[0], s_decomprInv[1], s_decomprInv[2], s_decomprInv[3], s_decomprInv[4], 
    //         s_decomprInv[5], s_decomprInv[6], s_decomprInv[7], s_decomprInv[8], s_decomprInv[9]);

    assert(decomprLength != 0);

    // Iterate the decompressed posting lists array s_decomprIOnv in blocks of MATCH_THREADS_PER_BLOCK
    // docsIDs, where each thread processes one docID at a time
    for (int i = 0; i < ((int)decomprLength - 1) / MATCH_THREADS_PER_BLOCK + 1; ++i)
    {
        if (idx + i * MATCH_THREADS_PER_BLOCK < (int)decomprLength)
        {
            u32 count = 0; //for AT
            u32 access_id = s_decomprInv[idx + i * MATCH_THREADS_PER_BLOCK];// retrieved docID from posting lists array

            if(shift_bits_subsequence != 0)
            {
                int __offset = access_id & (((unsigned int)1<<shift_bits_subsequence) - 1);
                int __new_offset = __offset - order;
                if(__new_offset >= 0)
                {
                    access_id = access_id - __offset + __new_offset;
                }
                else
                    continue;
            }

            u32 thread_threshold = *my_threshold;
            bool key_eligible;                //
            if (bitmap_bits)
            {

                key_eligible = false;
                //all count are store in the bitmap, and access the count
                count = bitmap_kernel_AT(access_id, bitmap, bitmap_bits,
                        thread_threshold, &key_eligible);

                if (!key_eligible)
                    continue;                //i.e. count< thread_threshold
            }

            key_eligible = false;
            if (count < *my_threshold)
            {
                continue;      //threshold has been increased, no need to insert
            }

            //Try to find the entry in hash tables
            bool pass_threshold;    //to determine whether pass the check of my_theshold
            access_kernel_AT(
                    access_id,               
                    hash_table, hash_table_size, myb_query, count, &key_eligible,
                    my_threshold, &pass_threshold);

            if (key_eligible)
            {
                if (pass_threshold)
                {
                    updateThreshold(my_passCount, my_threshold, my_topk, count);
                }

                continue;
            }

            if (!key_eligible)
            {
                //Insert the key into hash table
                //access_id and its location are packed into a packed key

                if (count < *my_threshold)
                {
                    continue;//threshold has been increased, no need to insert
                }

                hash_kernel_AT(access_id, hash_table, hash_table_size, myb_query, count,
                        my_threshold, my_noiih, overflow, &pass_threshold);
                if (*overflow)
                {

                    return;
                }
                if (pass_threshold)
                {
                    updateThreshold(my_passCount, my_threshold, my_topk, count);
                }
            }

        }
    }
}

template <class Codec> void
match_integrated(
            inv_compr_table& table,
            vector<query>& queries,
            device_vector<data_t>& d_data,
            device_vector<u32>& d_bitmap,
            int hash_table_size,
            int bitmap_bits, //or for AT: for adaptiveThreshold, if bitmap_bits<0, use adaptive threshold, the absolute value of bitmap_bits is count value stored in the bitmap
            device_vector<u32>& d_noiih,
            device_vector<u32>& d_threshold,
            device_vector<u32>& d_passCount)
{
    try{
        Logger::log(Logger::DEBUG, "Started match()");
        Logger::log(Logger::DEBUG, "hash table size: %d.", hash_table_size);

        u32 shift_bits_subsequence = table._shift_bits_subsequence();

        if (table.build_status() == inv_table::not_builded)
            throw GPUGenie::cpu_runtime_error("table not built!");
        
        // Time measuring events
        hipEvent_t kernel_start, kernel_stop;
        hipEvent_t startMatching, stopMatching, startConvert, stopConvert;
        hipEventCreate(&startMatching);
        hipEventCreate(&stopMatching);
        hipEventCreate(&startConvert);
        hipEventCreate(&stopConvert);
        hipEventCreate(&kernel_start);
        hipEventCreate(&kernel_stop);
        float matchDecomprTime, convertTime;
        u64 match_stop, match_start;
        match_start = getTime();

        Logger::log(Logger::INFO, "[  0%] Starting matching...");
        
        d_noiih.resize(queries.size(), 0);
        u32 * d_noiih_p = thrust::raw_pointer_cast(d_noiih.data());

        vector<query::dim> dims;        
        //number of maximum count per query
        u32 num_of_max_count = build_queries(queries, table, dims, table.getUncompressedPostingListMaxLength());

        Logger::log(Logger::DEBUG, "num_of_max_count: %d", num_of_max_count);
        

        //for AT: for adaptiveThreshold, enable adaptiveThreshold
        if (bitmap_bits < 0)
            bitmap_bits = -bitmap_bits; //for hash_table_size, still let it determine by users currently

        Logger::log(Logger::DEBUG, "[info] bitmap_bits:%d.", bitmap_bits);
        //end for AT

        int threshold = bitmap_bits - 1, bitmap_size = 0;
        if (bitmap_bits > 1)
        {
            float logresult = std::log2((float) bitmap_bits);
            bitmap_bits = (int) logresult;
            if (logresult - bitmap_bits > 0)
            {
                bitmap_bits += 1;
            }
            logresult = std::log2((float) bitmap_bits);
            bitmap_bits = (int) logresult;
            if (logresult - bitmap_bits > 0)
            {
                bitmap_bits += 1;
            }
            bitmap_bits = pow(2, bitmap_bits);
            bitmap_size = ((((unsigned int)1<<shift_bits_subsequence) * table.i_size()) / (32 / bitmap_bits) + 1)
                    * queries.size();
        }
        else
        {
            bitmap_bits = threshold = 0;
        }

        Logger::log(Logger::DEBUG, "Bitmap bits: %d, threshold:%d, shift_bits_subsequence: %d",
            bitmap_bits, threshold, shift_bits_subsequence);


        Logger::log(Logger::INFO, "[ 20%] Declaring device memory...");


        u64 query_start, query_end;
        query_start = getTime();
        thrust::device_vector<query::dim> d_dims(dims);
        query::dim* d_dims_p = thrust::raw_pointer_cast(d_dims.data());
        query_end  = getTime();
        Logger::log(Logger::DEBUG, "query_transfer time: %d",getInterval(query_start, query_end));
        
        // Make sure if we decompress a single lists from the table, we can fit it into shared memory
        assert(table.getUncompressedPostingListMaxLength() <= GPUGENIE_INTEGRATED_KERNEL_SM_SIZE);
        if (table.get_total_num_of_table() > 1 || !table.is_stored_in_gpu)
            table.cpy_data_to_gpu();

        d_bitmap.resize(bitmap_size);
        if (bitmap_size)
            thrust::fill(d_bitmap.begin(), d_bitmap.end(), 0u);

        u32 * d_bitmap_p = raw_pointer_cast(d_bitmap.data());



        Logger::log(Logger::INFO, "[ 30%] Allocating device memory to tables...");

        data_t nulldata;
        nulldata.id = 0u;
        nulldata.aggregation = 0.0f;
        T_HASHTABLE* d_hash_table;
        data_t* d_data_table;
        d_data.clear();

        d_data.resize(queries.size() * hash_table_size, nulldata);
        d_data_table = thrust::raw_pointer_cast(d_data.data());
        d_hash_table = reinterpret_cast<T_HASHTABLE*>(d_data_table);


        Logger::log(Logger::INFO, "[ 33%] Copying memory to symbol...");

        u32 h_offsets[16] = OFFSETS_TABLE_16;
        cudaCheckErrors(hipMemcpyToSymbol(HIP_SYMBOL(GPUGenie::offsets), h_offsets, sizeof(u32)*16, 0, hipMemcpyHostToDevice));


        Logger::log(Logger::INFO,"[ 40%] Starting decompression & match kernels...");

        hipEventRecord(kernel_start);

        bool h_overflow[1] = {false};
        bool * d_overflow;
        cudaCheckErrors(hipMalloc((void**) &d_overflow, sizeof(bool)));

        u32 loop_count = 1u;
        do
        {
            // Set overflow to false
            h_overflow[0] = false;
            cudaCheckErrors(hipMemcpy(d_overflow, h_overflow, sizeof(bool), hipMemcpyHostToDevice));

            // Set threshold to 1 for all queries
            d_threshold.resize(queries.size());
            thrust::fill(d_threshold.begin(), d_threshold.end(), 1);
            u32 * d_threshold_p = thrust::raw_pointer_cast(d_threshold.data());
            
            // Set d_passCount to 0 for all queries and all num_of_max_count
            d_passCount.resize(queries.size()*num_of_max_count);
            thrust::fill(d_passCount.begin(), d_passCount.end(), 0u);
            u32 * d_passCount_p = thrust::raw_pointer_cast(d_passCount.data());

            // Set d_topks to 0 for all queries
            u32 max_topk = cal_max_topk(queries);
            device_vector<u32> d_topks;
            d_topks.resize(queries.size());
            thrust::fill(d_topks.begin(), d_topks.end(), max_topk);
            u32 * d_topks_p = thrust::raw_pointer_cast(d_topks.data());

            
            // Call matching kernel, where each BLOCK does matching of one compiled query, only matching for the
            // next DECOMPR_BATCH compiled queries is done in one invocation of the kernel -- this corresponds to
            // the number of decompressed invereted lists
            hipEventRecord(startMatching);
            match_adaptiveThreshold_integrated<Codec><<<dims.size(),MATCH_THREADS_PER_BLOCK>>>
                   (table.m_size(),
                    (table.i_size() * ((unsigned int)1<<shift_bits_subsequence)),
                    hash_table_size, // hash table size
                    // d_compr_inv points to the start location of compressed posting list array in GPU mem
                    table.deviceCompressedInv(),
                    // compiled queries (dim structure)
                    d_dims_p,
                    d_hash_table, // data_t struct (id, aggregation) array of size queries.size() * hash_table_size
                    d_bitmap_p, // of bitmap_size
                    bitmap_bits,
                    d_topks_p, // d_topks set to max_topk for all queries
                    d_threshold_p,//initialized as 1, and increase gradually
                    d_passCount_p,//initialized as 0, count the number of items passing one d_threshold
                    num_of_max_count,//number of maximum count per query
                    d_noiih_p, // number of integers in a hash table set to 0 for all queries
                    d_overflow, // bool
                    shift_bits_subsequence);
            hipEventRecord(stopMatching);
            hipEventSynchronize(stopMatching);

            cudaCheckErrors(hipDeviceSynchronize());
            

            // Increase hash table size in case there was an overflow
            cudaCheckErrors(hipMemcpy(h_overflow, d_overflow, sizeof(bool), hipMemcpyDeviceToHost));
            if(h_overflow[0])
            {
                hash_table_size += num_of_max_count*max_topk;
                if(hash_table_size > table.i_size())
                {
                    hash_table_size = table.i_size();
                }
                thrust::fill(d_noiih.begin(), d_noiih.end(), 0u);
                if(bitmap_size)
                {
                    thrust::fill(d_bitmap.begin(), d_bitmap.end(), 0u);
                }
                d_data.resize(queries.size()*hash_table_size);
                thrust::fill(d_data.begin(), d_data.end(), nulldata);
                d_data_table = thrust::raw_pointer_cast(d_data.data());
                d_hash_table = reinterpret_cast<T_HASHTABLE*>(d_data_table);
            }

            // Log failed matching attempt
            if (loop_count>1 || (loop_count == 1 && h_overflow[0]))
                Logger::log(Logger::INFO,"%d time trying to launch match kernel: %s!",
                    loop_count, h_overflow[0]?"failed":"succeeded");
            loop_count ++;

        }while(h_overflow[0]);

        cudaCheckErrors(hipFree(d_overflow));

        hipEventRecord(kernel_stop);
        Logger::log(Logger::INFO,"[ 90%] Starting data converting......");

        hipEventRecord(startConvert);
        convert_to_data<<<hash_table_size*queries.size() / 1024 + 1,1024>>>(
            d_hash_table,(u32)hash_table_size*queries.size());
        hipEventRecord(stopConvert);

        hipEventSynchronize(stopConvert
            );
        Logger::log(Logger::INFO, "[100%] Matching is done!");

        match_stop = getTime();


        hipEventElapsedTime(&matchDecomprTime, startMatching, stopMatching);
        hipEventElapsedTime(&convertTime, startConvert, stopConvert);

        Logger::log(Logger::INFO,
                ">>>>[time profiling]: Match + decompression kernel takes %f ms. (GPU only) ",
                matchDecomprTime);
        Logger::log(Logger::INFO,
                ">>>>[time profiling]: Conversion kernel takes %f ms. (GPU only) ",
                convertTime);
        Logger::log(Logger::INFO,
                ">>>>[time profiling]: Total CPU+GPU match function takes %f ms. (GPU+CPU)",
                getInterval(match_start, match_stop));
        Logger::log(Logger::VERBOSE, ">>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>>");

        Codec c;
        PerfLogger::get().ofs()
            << c.name() << ","
            << std::fixed << std::setprecision(3) << matchDecomprTime << ","
            << std::fixed << std::setprecision(3) << convertTime << ","
            << std::fixed << std::setprecision(3) << getInterval(match_start, match_stop) << std::endl;

    } catch(std::bad_alloc &e){
        throw GPUGenie::gpu_bad_alloc(e.what());
    }
}

}
