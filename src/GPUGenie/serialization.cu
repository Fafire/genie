#include "hip/hip_runtime.h"
#include <fstream>
#include <memory>

#include <boost/archive/text_oarchive.hpp>
#include <boost/archive/text_iarchive.hpp>

#include "serialization.h"

void
genie::util::SaveTable(const std::string &filename, const GPUGenie::inv_table* table)
{
    // Cannot save more than one table using this function
    if (table->get_table_index() != 0 || table->get_total_num_of_table() != 1)
        throw GPUGenie::genie_error("Saving multiple tables not supported");

    std::ofstream ofs(filename.c_str(), std::ios::binary | std::ios::trunc);
    boost::archive::text_oarchive oa(ofs);
    oa << *table;
}

std::shared_ptr<GPUGenie::inv_table>
genie::util::LoadTable(const std::string &filename)
{
    std::shared_ptr<GPUGenie::inv_table> loaded_table = make_shared<GPUGenie::inv_table>();
    std::ifstream ifs(filename.c_str(), std::ios::binary);
    boost::archive::text_iarchive ia(ifs);
    ia >> *loaded_table;
    return loaded_table;
}

