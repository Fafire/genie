#include "interface.h"

using namespace std;
using namespace genie;

shared_ptr<GPUGenie::inv_table> genie::table::BuildTable(TableData& table_data)
{
	GPUGenie::GPUGenie_Config config;
	config.data_points = &table_data;
	GPUGenie::inv_table* table_ptr = nullptr;
	GPUGenie::preprocess_for_knn_csv(config, table_ptr);
	// Note: force array deleter to be used
	shared_ptr<GPUGenie::inv_table> table(table_ptr, [](GPUGenie::inv_table* p) { delete[] p; });

	return table;
}
