#include "hip/hip_runtime.h"
#include "inv_table.h"
#include "stdio.h"
#include "inv_list.h"
#include "Logger.h"

using namespace GPUGenie;

void
GPUGenie::inv_table::init()
{
    _shifter = 16;
    _size = -1;
    _build_status =  not_builded;
    _inv_lists.clear();
    _ck.clear();
    _inv.clear();
    _inv_index.clear();
}




bool
GPUGenie::inv_table::cpy_data_to_gpu()
{
         hipMalloc(&d_ck_p, sizeof(int)*_ck.size());
         hipMemcpy(d_ck_p, &_ck[0], sizeof(int)*_ck.size(), hipMemcpyHostToDevice);

         hipMalloc(&d_inv_p, sizeof(int)*_inv.size());
         hipMemcpy(d_inv_p, &_inv[0], sizeof(int)*_inv.size(), hipMemcpyHostToDevice);

         hipMalloc(&d_inv_index_p, sizeof(int)*_inv_index.size());
         hipMemcpy(d_inv_index_p, &_inv_index[0], sizeof(int)*_inv_index.size(), hipMemcpyHostToDevice);

         hipMalloc(&d_inv_pos_p, sizeof(int)*_inv_pos.size());
         hipMemcpy(d_inv_pos_p, &_inv_pos[0], sizeof(int)*_inv_pos.size(), hipMemcpyHostToDevice);

         return true;
}

void
GPUGenie::inv_table::clear()
{
  _build_status = not_builded;
  _inv_lists.clear();
  _ck.clear();
  _inv.clear();
  _ck_map.clear();
  clear_gpu_mem();
}

GPUGenie::inv_table::~inv_table()
{
    if(is_stored_in_gpu == true)
    {
        hipFree(d_inv_p);
        hipFree(d_inv_index_p);
        hipFree(d_inv_pos_p);
        hipFree(d_ck_p);
    }
}

void
GPUGenie::inv_table::clear_gpu_mem()
{
    if(is_stored_in_gpu == false)
        return;

    hipFree(d_inv_p);
    hipFree(d_inv_index_p);
    hipFree(d_inv_pos_p);
    hipFree(d_ck_p);
    is_stored_in_gpu = false;

}

bool
GPUGenie::inv_table::empty()
{
  return _size == -1;
}

int
GPUGenie::inv_table::m_size()
{
  return _inv_lists.size();
}

int
GPUGenie::inv_table::i_size()
{
  return _size <= -1 ? 0 : _size;
}

int
GPUGenie::inv_table::shifter()
{
  return _shifter;
}

void
GPUGenie::inv_table::append(inv_list& inv)
{
  if (_size == -1 || _size == inv.size())
    {
      _build_status = not_builded;
      _size = inv.size();
      _inv_lists.push_back(inv);
    }
}

void
GPUGenie::inv_table::append(inv_list* inv)
{
  if (inv != NULL)
    {
      append(*inv);
    }
}

GPUGenie::inv_table::status
GPUGenie::inv_table::build_status()
{
  return _build_status;
}

vector<inv_list>*
GPUGenie::inv_table::inv_lists()
{
  return &_inv_lists;
}

vector<int>*
GPUGenie::inv_table::ck()
{
  return &_ck;
}

vector<int>*
GPUGenie::inv_table::inv()
{
  return &_inv;
}

vector<int>*
GPUGenie::inv_table::inv_index()
{
	return &_inv_index;
}

vector<int>*
GPUGenie::inv_table::inv_pos()
{
	return &_inv_pos;
}

map<int, int>*
GPUGenie::inv_table::ck_map()
{
  return &_ck_map;
}

void
GPUGenie::inv_table::build(u64 max_length)
{
  _ck.clear(), _inv.clear();
  _inv_index.clear(); _inv_pos.clear();
  int i, j, key, dim, value, last;
  for (i = 0; i < _inv_lists.size(); i++)
    {
      dim = i << _shifter;
      for (value = _inv_lists[i].min(); value <= _inv_lists[i].max(); value++)
        {
          key = dim + value - _inv_lists[i].min();
          vector<int>& index = *_inv_lists[i].index(value);

          if (_ck.size() <= key)
            {
              last = _ck.size();
              _ck.resize(key + 1);
              _inv_index.resize(key+1);
              for (; last < _ck.size(); last++)
              {
            	  _ck[last] = _inv.size();
            	  _inv_index[last] = _inv_pos.size();
              }
            }
          for (j = 0; j < index.size(); j++)
            {
        	  if(j % max_length == 0){
        		  _inv_pos.push_back(_inv.size());
        	  }
              _inv.push_back(index[j]);
              _ck[key] = _inv.size();
            }

        }

    }
  _inv_index.push_back(_inv_pos.size());
  _inv_pos.push_back(_inv.size());

  _build_status = builded;
  Logger::log(Logger::DEBUG, "inv_index size %d:", _inv_index.size());
  Logger::log(Logger::DEBUG, "inv_pos size %d:", _inv_pos.size());
  Logger::log(Logger::DEBUG, "inv size %d:", _inv.size());
}

void
GPUGenie::inv_table::build_compressed()
{
  _ck.clear(), _inv.clear(), _ck_map.clear();
  int i, j, key, dim, value;
  for (i = 0; i < _inv_lists.size(); i++)
    {
      dim = i << _shifter;
      for (value = _inv_lists[i].min(); value <= _inv_lists[i].max(); value++)
        {
          key = dim + value - _inv_lists[i].min();
          vector<int>* indexes = _inv_lists[i].index(value);

          for (j = 0; j < indexes->size(); j++)
            {
              _inv.push_back((*indexes)[j]);
              _ck_map[key] = _ck.size();
            }
          if (indexes->size() > 0)
            {
              _ck.push_back(_inv.size());
            }
        }
    }
  _build_status = builded_compressed;
}
/*

void
GPUGenie::inv_table::serialize_to_file(const char* filename)
{
    ofsream ofs(filename, ios::trunc|ios::binary);
    if(!ofs.is_open())
    {
        return;
    }
    int status_int = _build_status;
    ofs.write((char*)&status_int, sizeof(int));
    ofs.write((char*)&_shifter, sizeof(int));
    ofs.write((char*)&_size, sizeof(int));


}

void
GPUGenie::inv_table::deserialize_from_file(const char* filename)
{
     
}
*/
