#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fstream>
#include <boost/serialization/map.hpp>
#include <boost/archive/binary_iarchive.hpp>
#include <boost/archive/binary_oarchive.hpp>
#include <exception>
#include "raw_data.h"
#include "Logger.h"

#include "inv_table.h"

using namespace GPUGenie;

void GPUGenie::inv_table::init()
{
	_shifter = 16;
	_size = -1;
	_build_status = not_builded;
	_inv_lists.clear();
	_ck.clear();
	_inv.clear();
	_inv_index.clear();
}

bool GPUGenie::inv_table::cpy_data_to_gpu()
{
	hipMalloc(&d_ck_p, sizeof(int) * _ck.size());
	hipMemcpy(d_ck_p, &_ck[0], sizeof(int) * _ck.size(),
			hipMemcpyHostToDevice);

	hipMalloc(&d_inv_p, sizeof(int) * _inv.size());
	hipMemcpy(d_inv_p, &_inv[0], sizeof(int) * _inv.size(),
			hipMemcpyHostToDevice);

	hipMalloc(&d_inv_index_p, sizeof(int) * _inv_index.size());
	hipMemcpy(d_inv_index_p, &_inv_index[0], sizeof(int) * _inv_index.size(),
			hipMemcpyHostToDevice);

	hipMalloc(&d_inv_pos_p, sizeof(int) * _inv_pos.size());
	hipMemcpy(d_inv_pos_p, &_inv_pos[0], sizeof(int) * _inv_pos.size(),
			hipMemcpyHostToDevice);

	return true;
}

void GPUGenie::inv_table::clear()
{
	_build_status = not_builded;
	_inv_lists.clear();
	_ck.clear();
	_inv.clear();
	_ck_map.clear();
	clear_gpu_mem();
}

GPUGenie::inv_table::~inv_table()
{
	if (is_stored_in_gpu == true)
	{
		hipFree(d_inv_p);
		hipFree(d_inv_index_p);
		hipFree(d_inv_pos_p);
		hipFree(d_ck_p);
	}
}

void GPUGenie::inv_table::clear_gpu_mem()
{
	if (is_stored_in_gpu == false)
		return;

	hipFree(d_inv_p);
	hipFree(d_inv_index_p);
	hipFree(d_inv_pos_p);
	hipFree(d_ck_p);
	is_stored_in_gpu = false;

}

bool GPUGenie::inv_table::empty()
{
	return _size == -1;
}

int GPUGenie::inv_table::m_size()
{
    return _dim_size;
	//return _inv_lists.size();
}

int GPUGenie::inv_table::i_size()
{
	return _size <= -1 ? 0 : _size;
}

int GPUGenie::inv_table::shifter()
{
	return _shifter;
}

void GPUGenie::inv_table::append(inv_list& inv)
{
	if (_size == -1 || _size == inv.size())
	{
		_build_status = not_builded;
		_size = inv.size();
		_inv_lists.push_back(inv);

        _dim_size = _inv_lists.size();
        inv_list_upperbound.push_back(inv.max());
        inv_list_lowerbound.push_back(inv.min());
	}
}

void GPUGenie::inv_table::append(inv_list* inv)
{
	if (inv != NULL)
	{
		append(*inv);
	}
}

int
GPUGenie::inv_table::get_upperbound_of_list(int index)
{
    if(index < inv_list_upperbound.size())
        return inv_list_upperbound[index];
    else
        return -1;
}

int
GPUGenie::inv_table::get_lowerbound_of_list(int index)
{
    if(index < inv_list_lowerbound.size())
        return inv_list_lowerbound[index];
    else
        return -1;
}


GPUGenie::inv_table::status GPUGenie::inv_table::build_status()
{
	return _build_status;
}

vector<inv_list>*
GPUGenie::inv_table::inv_lists()
{
	return &_inv_lists;
}

vector<int>*
GPUGenie::inv_table::ck()
{
	return &_ck;
}

vector<int>*
GPUGenie::inv_table::inv()
{
	return &_inv;
}

vector<int>*
GPUGenie::inv_table::inv_index()
{
	return &_inv_index;
}

vector<int>*
GPUGenie::inv_table::inv_pos()
{
	return &_inv_pos;
}

map<int, int>*
GPUGenie::inv_table::ck_map()
{
	return &_ck_map;
}

void
GPUGenie::inv_table::build(u64 max_length)
{
	_ck.clear(), _inv.clear();
	_inv_index.clear();
	_inv_pos.clear();
	unsigned int last;
	int key, dim, value;
	for (unsigned int i = 0; i < _inv_lists.size(); i++)
	{
		dim = i << _shifter;
		for (value = _inv_lists[i].min(); value <= _inv_lists[i].max(); value++)
		{
			key = dim + value - _inv_lists[i].min();
			vector<int>& index = *_inv_lists[i].index(value);

			if (_ck.size() <= (unsigned int) key)
			{
				last = _ck.size();
				_ck.resize(key + 1);
				_inv_index.resize(key + 1);
				for (; last < _ck.size(); last++)
				{
					_ck[last] = _inv.size();
					_inv_index[last] = _inv_pos.size();
				}
			}
			for (unsigned int j = 0; j < index.size(); j++)
			{
				if (j % max_length == 0)
				{
					_inv_pos.push_back(_inv.size());
				}
				_inv.push_back(index[j]);
				_ck[key] = _inv.size();
			}

		}

	}
	_inv_index.push_back(_inv_pos.size());
	_inv_pos.push_back(_inv.size());

	_build_status = builded;
	Logger::log(Logger::DEBUG, "inv_index size %d:", _inv_index.size());
	Logger::log(Logger::DEBUG, "inv_pos size %d:", _inv_pos.size());
	Logger::log(Logger::DEBUG, "inv size %d:", _inv.size());
}

void
GPUGenie::inv_table::build_compressed()
{
	_ck.clear(), _inv.clear(), _ck_map.clear();
	int key, dim, value;
	for (unsigned int i = 0; i < _inv_lists.size(); i++)
	{
		dim = i << _shifter;
		for (value = _inv_lists[i].min(); value <= _inv_lists[i].max(); value++)
		{
			key = dim + value - _inv_lists[i].min();
			vector<int>* indexes = _inv_lists[i].index(value);

			for (unsigned int j = 0; j < indexes->size(); j++)
			{
				_inv.push_back((*indexes)[j]);
				_ck_map[key] = _ck.size();
			}
			if (indexes->size() > 0)
			{
				_ck.push_back(_inv.size());
			}
		}
	}
	_build_status = builded_compressed;
}



void
GPUGenie::inv_table::write_to_file(const char* filename)
{
    if(_build_status == not_builded)
        return;
    ofstream ofs(filename, ios::binary|ios::trunc|ios::out);
    if(!ofs.is_open())
        return;

    ofs.write((char*)&_shifter, sizeof(int));
    ofs.write((char*)&_size, sizeof(int));
    ofs.write((char*)&_dim_size, sizeof(int));
    int temp_status = _build_status;
    ofs.write((char*)&temp_status, sizeof(int));

    unsigned int _ck_size = _ck.size();
    unsigned int _inv_size = _inv.size();
    unsigned int _inv_index_size = _inv_index.size();
    unsigned int _inv_pos_size = _inv_pos.size();
    
    ofs.write((char*)&_ck_size, sizeof(unsigned int));
    ofs.write((char*)&_inv_size, sizeof(unsigned int));
    ofs.write((char*)&_inv_index_size, sizeof(unsigned int));
    ofs.write((char*)&_inv_pos_size, sizeof(unsigned int));

    ofs.write((char*)&_ck[0], _ck_size*sizeof(int));
    ofs.write((char*)&_inv[0], _inv_size*sizeof(int));
    ofs.write((char*)&_inv_index[0],_inv_index_size*sizeof(int));
    ofs.write((char*)&_inv_pos[0], _inv_pos_size*sizeof(int));

    unsigned int _list_upperbound_size = inv_list_upperbound.size();
    unsigned int _list_lowerbound_size = inv_list_lowerbound.size();

    ofs.write((char*)&_list_upperbound_size, sizeof(unsigned int));
    ofs.write((char*)&_list_lowerbound_size, sizeof(unsigned int));

    ofs.write((char*)&inv_list_upperbound[0], _list_upperbound_size*sizeof(int));
    ofs.write((char*)&inv_list_upperbound[0], _list_upperbound_size*sizeof(int));


    if(_build_status == builded_compressed)
    {
        boost::archive::binary_oarchive oarch(ofs);
        oarch << _ck_map;
    }

    ofs.close();
    return;
}


void
GPUGenie::inv_table::read_from_file(const char* filename)
{


    ifstream ifs(filename, ios::binary|ios::in);
    if(!ifs.is_open())
        return;
    ifs.read((char*)&_shifter, sizeof(int));
    ifs.read((char*)&_size, sizeof(int));
    ifs.read((char*)&_dim_size, sizeof(int));
    int temp_status;
    ifs.read((char*)&temp_status, sizeof(int));
    _build_status = static_cast<status>(temp_status);


    unsigned int _ck_size;
    unsigned int _inv_size;
    unsigned int _inv_index_size;
    unsigned int _inv_pos_size;

    ifs.read((char*)&_ck_size, sizeof(unsigned int));
    ifs.read((char*)&_inv_size, sizeof(unsigned int));
    ifs.read((char*)&_inv_index_size, sizeof(unsigned int));
    ifs.read((char*)&_inv_pos_size, sizeof(unsigned int));

    _ck.resize(_ck_size);
    _inv.resize(_inv_size);
    _inv_index.resize(_inv_index_size);
    _inv_pos.resize(_inv_pos_size);

    ifs.read((char*)&_ck[0], _ck_size*sizeof(int));
    ifs.read((char*)&_inv[0], _inv_size*sizeof(int));
    ifs.read((char*)&_inv_index[0],_inv_index_size*sizeof(int));
    ifs.read((char*)&_inv_pos[0], _inv_pos_size*sizeof(int));
    
    unsigned int _list_upperbound_size;
    unsigned int _list_lowerbound_size;

    ifs.read((char*)&_list_upperbound_size, sizeof(unsigned int));
    ifs.read((char*)&_list_lowerbound_size, sizeof(unsigned int));

    inv_list_upperbound.resize(_list_upperbound_size);
    inv_list_lowerbound.resize(_list_lowerbound_size);
    ifs.read((char*)&inv_list_upperbound[0], _list_upperbound_size*sizeof(int));
    ifs.read((char*)&inv_list_upperbound[0], _list_upperbound_size*sizeof(int));


    if(_build_status == builded_compressed)
    {
        boost::archive::binary_iarchive iarch(ifs);
        iarch >> _ck_map;
    }

    ifs.close();
    return;
}


