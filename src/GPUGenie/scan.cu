#include "hip/hip_runtime.h"
/*
 * This module contains source code provided by NVIDIA Corporation.
 */

#include <assert.h>
#include <GPUGenie/genie_errors.h>

#include "scan.h"

const uint THREADBLOCK_SIZE = GPUGENIE_SCAN_THREADBLOCK_SIZE;

const uint GPUGenie::SCAN_THREADBLOCK_SIZE     = GPUGENIE_SCAN_THREADBLOCK_SIZE;
const uint GPUGenie::SCAN_MIN_SHORT_ARRAY_SIZE = GPUGENIE_SCAN_MIN_SHORT_ARRAY_SIZE;
const uint GPUGenie::SCAN_MAX_SHORT_ARRAY_SIZE = GPUGENIE_SCAN_MAX_SHORT_ARRAY_SIZE;
const uint GPUGenie::SCAN_MIN_LARGE_ARRAY_SIZE = GPUGENIE_SCAN_MIN_LARGE_ARRAY_SIZE;
const uint GPUGenie::SCAN_MAX_LARGE_ARRAY_SIZE = GPUGENIE_SCAN_MAX_LARGE_ARRAY_SIZE;

// Naive inclusive scan: O(N * log2(N)) operations
// Allocate 2 * 'size' local memory, initialize the first half with 'size' zeros avoiding if(pos >= offset) condition
// evaluation and saving instructions
inline __device__ uint scan1Inclusive(uint idata, volatile uint *s_Data, uint size)
{
    uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
    s_Data[pos] = 0;
    pos += size;
    s_Data[pos] = idata;

    for (uint offset = 1; offset < size; offset <<= 1)
    {
        __syncthreads();
        uint t = s_Data[pos] + s_Data[pos - offset];
        __syncthreads();
        s_Data[pos] = t;
    }

    return s_Data[pos];
}

inline __device__ uint scan1Exclusive(uint idata, volatile uint *s_Data, uint size)
{
    return scan1Inclusive(idata, s_Data, size) - idata;
}


inline __device__ uint4 scan4Inclusive(uint4 idata4, volatile uint *s_Data, uint size)
{
    //Level-0 inclusive scan
    idata4.y += idata4.x;
    idata4.z += idata4.y;
    idata4.w += idata4.z;

    //Level-1 exclusive scan
    uint oval = scan1Exclusive(idata4.w, s_Data, size / 4);

    idata4.x += oval;
    idata4.y += oval;
    idata4.z += oval;
    idata4.w += oval;

    return idata4;
}

// Exclusive vector scan: the array to be scanned is stored in local thread memory scope as uint4
inline __device__ uint4 scan4Exclusive(uint4 idata4, volatile uint *s_Data, uint size)
{
    uint4 odata4 = scan4Inclusive(idata4, s_Data, size);
    odata4.x -= idata4.x;
    odata4.y -= idata4.y;
    odata4.z -= idata4.z;
    odata4.w -= idata4.w;
    return odata4;
}

__global__ void GPUGenie::g_scanExclusiveShared(
    uint4 *d_Dst,
    uint4 *d_Src,
    uint activeThreads,
    uint pow2size)
{
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load data
    uint4 idata4 = (pos < activeThreads) ? d_Src[pos] : uint4{0,0,0,0};

    //Calculate exclusive scan
    uint4 odata4 = scan4Exclusive(idata4, s_Data, pow2size);

    //Write back
    if (pos < activeThreads)
        d_Dst[pos] = odata4;
}

__device__ void GPUGenie::d_scanExclusivePerBlockShared(
    uint4 *d_Dst,
    uint4 *d_Src,
    uint activeThreads,
    uint pow2size)
{
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    uint pos = threadIdx.x;

    //Load data
    uint4 idata4 = (pos < activeThreads) ? d_Src[pos] : uint4{0,0,0,0};

    //Calculate exclusive scan
    uint4 odata4 = scan4Exclusive(idata4, s_Data, pow2size);

    //Write back
    if (pos < activeThreads)
        d_Dst[pos] = odata4;
}

__device__ void GPUGenie::d_scanExclusiveShared(
    uint4 *d_Dst,
    uint4 *d_Src,
    uint activeThreads,
    uint pow2size)
{
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load data
    uint4 idata4 = (pos < activeThreads) ? d_Src[pos] : uint4{0,0,0,0};

    //Calculate exclusive scan
    uint4 odata4 = scan4Exclusive(idata4, s_Data, pow2size);

    //Write back
    if (pos < activeThreads)
        d_Dst[pos] = odata4;
}

__global__ void GPUGenie::g_scanInclusiveShared(
    uint4 *d_Dst,
    uint4 *d_Src,
    uint activeThreads,
    uint pow2size)
{
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load data
    uint4 idata4 = (pos < activeThreads) ? d_Src[pos] : uint4{0,0,0,0};

    //Calculate exclusive scan
    uint4 odata4 = scan4Inclusive(idata4, s_Data, pow2size);

    //Write back
    if (pos < activeThreads)
        d_Dst[pos] = odata4;
}

__device__ void GPUGenie::d_scanInclusivePerBlockShared(
    uint4 *d_Dst,
    uint4 *d_Src,
    uint activeThreads,
    uint pow2size)
{
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    uint pos = threadIdx.x;

    //Load data
    uint4 idata4 = (pos < activeThreads) ? d_Src[pos] : uint4{0,0,0,0};

    //Calculate exclusive scan
    uint4 odata4 = scan4Inclusive(idata4, s_Data, pow2size);

    //Write back
    if (pos < activeThreads)
        d_Dst[pos] = odata4;
}

__device__ void GPUGenie::d_scanInclusiveShared(
    uint4 *d_Dst,
    uint4 *d_Src,
    uint activeThreads,
    uint pow2size)
{
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load data
    uint4 idata4 = (pos < activeThreads) ? d_Src[pos] : uint4{0,0,0,0};

    //Calculate exclusive scan
    uint4 odata4 = scan4Inclusive(idata4, s_Data, pow2size);

    //Write back
    if (pos < activeThreads)
        d_Dst[pos] = odata4;
}

//Exclusive scan of top elements of bottom-level scans (4 * THREADBLOCK_SIZE)
__global__ void scanExclusiveShared2(
    uint *d_Buf,
    uint *d_Dst,
    uint *d_Src,
    uint arrayLength,
    uint blocks)
{
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    //Skip loads and stores for inactive threads of last threadblock (pos >= blocks)
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load top elements
    //Convert results of bottom-level scan back to inclusive
    uint idata = 0;

    uint sumLocation;
    if (pos < blocks - 1)
        sumLocation = (4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos;
    else
        sumLocation = arrayLength;

    idata = 
        d_Dst[sumLocation] + d_Src[sumLocation];

    //Compute
    uint odata = scan1Exclusive(idata, s_Data, blocks);

    //Avoid out-of-bound access
    if (pos < blocks)
    {
        d_Buf[pos] = odata;
    }
}

//Final step of large-array scan: combine basic inclusive scan with exclusive scan of top elements of input arrays
__global__ void uniformUpdate(
    uint4 *d_Data,
    uint *d_Buffer,
    uint arrayLength)
{
    __shared__ uint buf;
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIdx.x == 0)
    {
        buf = d_Buffer[blockIdx.x];
    }

    __syncthreads();

    if (pos < arrayLength)
    {   
        uint4 data4 = d_Data[pos];
        data4.x += buf;
        data4.y += buf;
        data4.z += buf;
        data4.w += buf;
        d_Data[pos] = data4;
    }
}


//Internal exclusive scan buffer
static uint *d_Buf;

void GPUGenie::initScan(void)
{
    cudaCheckErrors(hipMalloc((void **)&d_Buf, THREADBLOCK_SIZE * sizeof(uint)));
}

void GPUGenie::closeScan(void)
{
    cudaCheckErrors(hipFree(d_Buf));
}


// Returns the first power of two greater or equal to x
__device__ uint GPUGenie::d_pow2ceil_32 (uint x)
{
    if (x == 0)
        return 0;
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return x+1;
}

// Returns the first power of two greater or equal to x
uint GPUGenie::h_pow2ceil_32 (uint x)
{
    if (x == 0)
        return 0;
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return x+1;
}

static uint iDivUp(uint dividend, uint divisor)
{
    return ((dividend % divisor) == 0) ? (dividend / divisor) : (dividend / divisor + 1);
}

size_t GPUGenie::scanExclusiveShort(
    uint *d_Dst,
    uint *d_Src,
    uint arrayLength)
{
    // Check the array length is a mutliple of 4. This is because we use uint4 processed by a single thread.
    assert(arrayLength % 4 == 0);

    //Check power-of-two factorization
    uint pow2arrayLength = h_pow2ceil_32(arrayLength);
    // printf("power of two size: %u\n", pow2arrayLength);
    assert(pow2arrayLength >= arrayLength);

    // Check supported size range
    assert((pow2arrayLength >= SCAN_MIN_SHORT_ARRAY_SIZE) && (pow2arrayLength <= SCAN_MAX_SHORT_ARRAY_SIZE));

    // printf("running scanExclusiveShort on %d blocks each of %d threads, total active threads: %d\n",
    //     (pow2arrayLength+(4*THREADBLOCK_SIZE)-1)/(4*THREADBLOCK_SIZE),THREADBLOCK_SIZE, arrayLength/4);

    g_scanExclusiveShared<<<(pow2arrayLength+(4*THREADBLOCK_SIZE)-1)/(4*THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint4 *)d_Src,
        arrayLength / 4,
        pow2arrayLength
    );
    CUDA_LAST_ERROR();

    return THREADBLOCK_SIZE;
}

size_t GPUGenie::scanExclusiveLarge(
    uint *d_Dst,
    uint *d_Src,
    uint arrayLength)
{
    // Check the array length is a mutliple of 4. This is because we use uint4 processed by a single thread.
    assert(arrayLength % 4 == 0);

    //Check power-of-two factorization
    uint pow2arrayLength = h_pow2ceil_32(arrayLength);
    // printf("power of two size: %u\n", pow2arrayLength);
    assert(pow2arrayLength >= (arrayLength));

    //Check supported size range
    assert((pow2arrayLength >= SCAN_MIN_LARGE_ARRAY_SIZE) && (pow2arrayLength <= SCAN_MAX_LARGE_ARRAY_SIZE));

    // printf("running scanExclusiveLong on %d blocks each of %d threads\n",
        // (pow2arrayLength + (4 * THREADBLOCK_SIZE) - 1) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE);

    g_scanExclusiveShared<<<(pow2arrayLength + (4 * THREADBLOCK_SIZE) - 1) / (4 * THREADBLOCK_SIZE),
                           THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint4 *)d_Src,
        arrayLength / 4,
        4 * THREADBLOCK_SIZE
    );
    CUDA_LAST_ERROR();

    //Not all threadblocks need to be packed with input data:
    //inactive threads of highest threadblock just don't do global reads and writes
    const uint blockCount2 = iDivUp(pow2arrayLength / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE);
    scanExclusiveShared2<<< blockCount2, THREADBLOCK_SIZE>>>(
        (uint *)d_Buf,
        (uint *)d_Dst,
        (uint *)d_Src,
        arrayLength, // uses the original arrayLength for uint array, unlike scanExclusiveShared and uniformUpdate
        pow2arrayLength / (4 * THREADBLOCK_SIZE)
    );
    CUDA_LAST_ERROR();

    uniformUpdate<<<pow2arrayLength / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint *)d_Buf,
        (arrayLength + 3) / 4
    );
    CUDA_LAST_ERROR();

    return THREADBLOCK_SIZE;
}

void GPUGenie::scanExclusiveHost(
    uint *dst,
    uint *src,
    uint arrayLength)
{
    dst[0] = 0;

    for (uint j = 1; j < arrayLength; j++)
        dst[j] = src[j - 1] + dst[j - 1];
}



