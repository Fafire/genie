#include "hip/hip_runtime.h"
#include <genie/utility/scan.h>

#include "DeviceBitPackingCodec.h"
#include "DeviceVarintCodec.h"

#include "DeviceCompositeCodec.h"

#include "DeviceCodecTemplatesImpl.hpp"

using namespace GPUGenie;

// Explicit template instances for Composite Codecs

template class
GPUGenie::DeviceCompositeCodec<DeviceBitPackingCodec,DeviceCopyCodec>;
template class
GPUGenie::DeviceCompositeCodec<DeviceBitPackingCodec,DeviceVarintCodec>;

// Explicit template instances for CPU decoding wrapper function for Composite Codecs
// NOTE: This is intentionally separated into mutliple codec implementation files in order to facilitiate separate
// compilation units, as opposed to defining all these templates in one place
template void
GPUGenie::decodeArrayParallel<DeviceCompositeCodec<DeviceBitPackingCodec,DeviceCopyCodec>>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);
template void
GPUGenie::decodeArrayParallel<DeviceCompositeCodec<DeviceBitPackingCodec,DeviceVarintCodec>>(int, int, uint32_t*, size_t, uint32_t*, size_t, size_t*);


template <class Codec1, class Codec2> void
GPUGenie::DeviceCompositeCodec<Codec1,Codec2>::encodeArray(uint32_t *in, const size_t length, uint32_t *out, size_t &nvalue)
{
    assert(length > 0);
    assert(nvalue > 0);
    int codec1minEffLength = codec1.decodeArrayParallel_minEffectiveLength();
    size_t codec1Length = (length / codec1minEffLength) * codec1minEffLength;
    size_t codec2Length = length - codec1Length;
    assert (codec1Length + codec2Length == length);
    assert (codec2Length <= length);

    size_t nvalue1 = 0;
    if (codec1Length){
        nvalue1 = nvalue;
        codec1.encodeArray(in, codec1Length, out + 1, nvalue1);
        assert(nvalue >= nvalue1); // Error - compression overflow
    }

    size_t nvalue2 = 0;
    if (codec2Length) {
        nvalue2 = nvalue - nvalue1;
        codec2.encodeArray(in + codec1Length, codec2Length, out + 1 + nvalue1, nvalue2);
        assert(nvalue - nvalue1 >= nvalue2); // Error - compression overflow
    }

    out[0] = nvalue1; // store infromation about compressed length from the first codec

    nvalue = 1 + nvalue1 + nvalue2;
}

template <class Codec1, class Codec2> const uint32_t*
GPUGenie::DeviceCompositeCodec<Codec1,Codec2>::decodeArray(const uint32_t *in, const size_t comprLength, uint32_t *out, size_t &nvalue)
{
    size_t firstCodecComprLength = *in++;

    // Codec1 decompresses as much as it can
    size_t nvalue1 = 0;
    const uint32_t *inForCodec2 = in;

    if (firstCodecComprLength){
        nvalue1 = nvalue; // set capacity for codec1 to overall capacity
        inForCodec2 = codec1.decodeArray(in, firstCodecComprLength, out, nvalue1);

        if (nvalue1 > nvalue){ // Error - Codec1 does not have enough capacity
            nvalue = nvalue1; // Set nvalue to required capacity of codec1
            return in; // Return pointer to the deginning of the compressed array
        }

        if (inForCodec2 == in + comprLength - 1){ // Codec1 decompressed everything
            nvalue = nvalue1;
            return inForCodec2;
        }
    }

    assert(inForCodec2 == in + firstCodecComprLength); // Make sure codec1 returned correct d_in pointer

    // Codec2 decompresses the leftover
    size_t nvalue2 = nvalue - nvalue1; // remaining capacity
    size_t leftoverLength = comprLength - 1 - (inForCodec2 - in);
    const uint32_t *inAfterBothCodecs = codec2.decodeArray(inForCodec2, leftoverLength, out + nvalue1, nvalue2);

    if (nvalue2 > nvalue - nvalue1){ // Error - Codec2 does not have enough capacity
        nvalue = nvalue1 + nvalue2; // Set nvalue to required capacity of codec1 + codec2
        return in; // Return pointer to the deginning of the compressed array
    }

    assert(in + comprLength - 1 == inAfterBothCodecs);
    nvalue = nvalue1 + nvalue2;
    return inAfterBothCodecs;
}


template <class Codec1, class Codec2> __device__ uint32_t*
GPUGenie::DeviceCompositeCodec<Codec1,Codec2>::decodeArraySequential(uint32_t *d_in, size_t length, uint32_t *d_out, size_t &nvalue)
{
    return nullptr;
}



template <class Codec1, class Codec2> __device__ uint32_t*
GPUGenie::DeviceCompositeCodec<Codec1,Codec2>::decodeArrayParallel(
            uint32_t *d_in, size_t comprLength, uint32_t *d_out, size_t &nvalue)
{
    size_t firstCodecComprLength = *d_in++;

    // Codec1 decompresses as much as it can
    size_t nvalue1 = 0;
    uint32_t *d_inForCodec2 = d_in;

    if (firstCodecComprLength){
        nvalue1 = nvalue; // set capacity for codec1 to overall capacity

        d_inForCodec2 = codec1.decodeArrayParallel(d_in, firstCodecComprLength, d_out, nvalue1);
        __syncthreads();

        if (nvalue1 > nvalue){ // Error - Codec1 does not have enough capacity
            nvalue = nvalue1; // Set nvalue to required capacity of codec1
            return d_in; // Return pointer to the deginning of the compressed array
        }

        if (d_inForCodec2 == d_in + comprLength - 1){ // Codec1 decompressed everything
            nvalue = nvalue1;
            return d_inForCodec2;
        }
    }

    assert(d_inForCodec2 == d_in + firstCodecComprLength); // Make sure codec1 returned correct d_in pointer

    // Codec2 decompresses the leftover
    size_t nvalue2 = nvalue - nvalue1; // remaining capacity
    size_t leftoverLength = comprLength - 1 - firstCodecComprLength;
    uint32_t *d_inAfterBothCodecs = codec2.decodeArrayParallel(d_inForCodec2, leftoverLength, d_out + nvalue1, nvalue2);
    __syncthreads();

    if (nvalue2 > nvalue - nvalue1){ // Error - Codec2 does not have enough capacity
        nvalue = nvalue1 + nvalue2; // Set nvalue to required capacity of codec1 + codec2
        return d_in; // Return pointer to the deginning of the compressed array
    }

    assert(d_in + comprLength - 1 == d_inAfterBothCodecs);
    nvalue = nvalue1 + nvalue2;
    return d_inAfterBothCodecs;
}

